#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/utils.cuh"

namespace TNN_NS {

//template<typename T>
//__device__ float get_float_value(T value) { return value; }
template<> __device__ float get_float_value<__half>(__half value) { return __half2float(value); }

//template<typename T>
//__device__ T convert_float_value(float value) { return T(value); }
template<> __device__ __half convert_float_value<__half>(float value) { return __float2half(value); }

}  //  namespace TNN_NS;

