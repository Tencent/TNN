#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/acc/cuda_layer_acc.h"
#include "tnn/utils/dims_utils.h"

namespace TNN_NS {

DECLARE_CUDA_ACC(ScatterND, LAYER_SCATTER_ND);

__global__ void scatter_nd_kernel(int offset_count, const int* indice, const float* update, float* output,
        int last_indice_dimension, int element_to_copy, int* element_counts) {
    CUDA_KERNEL_LOOP(index, offset_count) {
        int offset = 0;
        for (int j = 0; j < last_indice_dimension; j++) {
            offset += indice[index * last_indice_dimension + j] * element_counts[j];
        }
        for (int j = 0; j < element_to_copy; j++) {
            output[offset] = update[index * element_to_copy + j];
        }
    }
}

Status CudaScatterNDLayerAcc::Init(Context *context, LayerParam *param, LayerResource *resource,
        const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    Status ret = CudaLayerAcc::Init(context, param, resource, inputs, outputs);
    auto res = dynamic_cast<ScatterNDLayerResource *>(resource);
    if (ret != TNN_OK) {
        return ret;
    }

    if (inputs.size() < 3) {
        auto count = res->indices.GetDataCount();
        CreateTempBuf(count * sizeof(int));
    }

    auto input_dims = inputs[0]->GetBlobDesc().dims;
    CreateTempBuf(input_dims.size() * sizeof(int));

    return TNN_OK;
}

Status CudaScatterNDLayerAcc::Reshape(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    auto resource = dynamic_cast<ScatterNDLayerResource *>(resource_);
    DimsVector indices_dims;
    if (inputs.size() < 3) {
        indices_dims = resource->indices.GetBufferDims();
    } else {
        indices_dims = inputs[1]->GetBlobDesc().dims;
    }

    auto input_dims = inputs[0]->GetBlobDesc().dims;
    auto indice_rank = indices_dims.size();
    auto last_indice_dimension = indices_dims[indice_rank - 1];
    std::vector<int> element_counts(last_indice_dimension, 0);
    for (int i = 0; i < last_indice_dimension; ++i) {
        element_counts[i] = DimsVectorUtils::Count(input_dims, i + 1);
    }

    if (inputs.size() < 3) {
        int* indice = resource->indices.force_to<int*>();
        int count = resource->indices.GetDataCount();
        hipMemcpyAsync(tempbufs_[0].ptr, indice, count * sizeof(int), hipMemcpyHostToDevice, context_->GetStream());
        hipMemcpyAsync(tempbufs_[1].ptr, element_counts.data(), last_indice_dimension * sizeof(int),
            hipMemcpyHostToDevice, context_->GetStream());
    } else {
        hipMemcpyAsync(tempbufs_[0].ptr, element_counts.data(), last_indice_dimension * sizeof(int),
            hipMemcpyHostToDevice, context_->GetStream());
    }

    return TNN_OK;
}

Status CudaScatterNDLayerAcc::Forward(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    Blob *input_blob  = inputs[0];
    Blob *update_blob = inputs.size() < 3 ? inputs[1] : inputs[2];
    Blob *output_blob = outputs[0];
    auto input_dims = inputs[0]->GetBlobDesc().dims;
    float* input_data = reinterpret_cast<float*>(input_blob->GetHandle().base);
    float* update_data = reinterpret_cast<float*>(update_blob->GetHandle().base);
    float* output_data = reinterpret_cast<float*>(output_blob->GetHandle().base);
    int* indice_data = nullptr;
    int* element_counts = nullptr;

    auto resource = dynamic_cast<ScatterNDLayerResource *>(resource_);
    DimsVector indices_dims;
    if (inputs.size() < 3) {
        indices_dims = resource->indices.GetBufferDims();
        indice_data = (int*)tempbufs_[0].ptr;
        element_counts = (int*)tempbufs_[1].ptr;
    } else {
        indices_dims = inputs[1]->GetBlobDesc().dims;
        indice_data = reinterpret_cast<int*>(inputs[1]->GetHandle().base);
        element_counts = (int*)tempbufs_[0].ptr;
    }

    auto indice_rank = indices_dims.size();
    auto last_indice_dimension = indices_dims[indice_rank - 1];
    int element_to_copy = DimsVectorUtils::Count(input_dims, last_indice_dimension);
    int offset_count = DimsVectorUtils::Count(indices_dims, 0, indice_rank - 1);

    hipMemcpyAsync(output_data, input_data, DimsVectorUtils::Count(input_dims) * sizeof(float),
        hipMemcpyDeviceToDevice, context_->GetStream());

    scatter_nd_kernel<<<TNN_CUDA_GET_BLOCKS(offset_count), TNN_CUDA_NUM_THREADS, 0, context_->GetStream()>>>(
        offset_count, indice_data, update_data, output_data, last_indice_dimension, element_to_copy, element_counts);

    return TNN_OK;
}

REGISTER_CUDA_ACC(ScatterND, LAYER_SCATTER_ND);

}  // namespace TNN_NS
