#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/acc/cuda_layer_acc.h"
#include "tnn/utils/dims_utils.h"

namespace TNN_NS {

DECLARE_CUDA_ACC(ReduceLogSumExp, LAYER_REDUCE_LOG_SUM_EXP);

template <int blockSize, typename T>
__global__ void reduce_log_sum_exp_kernel(const int num, const int channels,
        const int spatial_dim, const T* input, T* output) {
    int n = blockIdx.x / spatial_dim;
    int s = blockIdx.x % spatial_dim;

    __shared__ float smax[blockSize/32];
    __shared__ float ssum[blockSize/32];

    int tid = threadIdx.x;
    float max_value = -FLT_MAX;
    for (int c = tid; c < channels; c += blockDim.x) {
        float value = get_float_value<T>(input[(n * channels + c) * spatial_dim + s]);
        max_value = fmaxf(value, max_value);
    }

    float tmp = __shfl_down_sync(0xffffffff, max_value, 16, 32);
    max_value = fmaxf(max_value, tmp);
    tmp = __shfl_down_sync(0xffffffff, max_value, 16, 32);
    max_value = fmaxf(max_value, tmp);
    tmp = __shfl_down_sync(0x0000ffff, max_value, 8, 16);
    max_value = fmaxf(max_value, tmp);
    tmp = __shfl_down_sync(0x000000ff, max_value, 4, 8);
    max_value = fmaxf(max_value, tmp);
    tmp = __shfl_down_sync(0x0000000f, max_value, 2, 4);
    max_value = fmaxf(max_value, tmp);
    tmp = __shfl_down_sync(0x00000003, max_value, 1, 2);
    max_value = fmaxf(max_value, tmp);

    if (tid % 32 == 0) {
        smax[tid / 32] = max_value;
    }
    __syncthreads();

    if (tid < blockDim.x / 32) {
        max_value = smax[tid];
    } else {
        max_value = -FLT_MAX;
    }

    tmp = __shfl_down_sync(0x0000000f, max_value, 2, 4);
    max_value = fmaxf(max_value, tmp);
    tmp = __shfl_down_sync(0x00000003, max_value, 1, 2);
    max_value = fmaxf(max_value, tmp);

    if (tid == 0) {
        smax[0] = max_value;
    }
    __syncthreads();

    float thread_sum = 0;
    for (int c = tid; c < channels; c += blockDim.x) {
        float value = get_float_value<T>(input[(n * channels + c) * spatial_dim + s]);
        thread_sum += exp(value - smax[0]);
    }

    thread_sum += __shfl_down_sync(0xffffffff, thread_sum, 16, 32);
    thread_sum += __shfl_down_sync(0x0000ffff, thread_sum, 8, 16);
    thread_sum += __shfl_down_sync(0x000000ff, thread_sum, 4, 8);
    thread_sum += __shfl_down_sync(0x0000000f, thread_sum, 2, 4);
    thread_sum += __shfl_down_sync(0x00000003, thread_sum, 1, 2);

    if (tid % 32 == 0) {
        ssum[tid / 32] = thread_sum;
    }
    __syncthreads();

    if (tid < blockDim.x / 32) {
        thread_sum = ssum[tid];
    } else {
        thread_sum = 0;
    }

    thread_sum += __shfl_down_sync(0x0000000f, thread_sum, 2, 4);
    thread_sum += __shfl_down_sync(0x00000003, thread_sum, 1, 2);

    if (tid == 0) {
        output[n * spatial_dim + s] = convert_float_value<T>(log(thread_sum) + smax[0]);
    }
}

Status CudaReduceLogSumExpLayerAcc::Init(Context *context, LayerParam *param, LayerResource *resource,
        const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return CudaLayerAcc::Init(context, param, resource, inputs, outputs);
}

Status CudaReduceLogSumExpLayerAcc::Reshape(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

Status CudaReduceLogSumExpLayerAcc::Forward(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    auto params = dynamic_cast<ReduceLayerParam *>(param_);
    if (!params) {
        LOGE("Error: layer param is null\n");
        return Status(TNNERR_MODEL_ERR, "Error: layer param is null");
    }

    Blob *input_blob  = inputs[0];
    Blob *output_blob = outputs[0];

    int channels = 1;
    int first_axis = 4;
    int last_axis = 0;
    // remove duplicate axes
    auto axis = params->axis;
    std::sort(axis.begin(), axis.end());
    axis.erase(std::unique(axis.begin(), axis.end() ), axis.end());
    for (int i = 0; i < axis.size(); i++) {
        channels *= input_blob->GetBlobDesc().dims[axis[i]];
        first_axis = std::min(axis[i], first_axis);
        last_axis = std::max(axis[i], last_axis);
    }

    for(int i=first_axis; i<=last_axis; ++i) {
        if (std::find(axis.begin(), axis.end(), i) == axis.end()) {
            LOGE("Error: discontinuous reduce axes!");
            return Status(TNNERR_PARAM_ERR, "Error: discontinuous reduce axes!"); 
        }
    }

    int outer_dim = DimsVectorUtils::Count(input_blob->GetBlobDesc().dims, 0, first_axis);
    int inner_dim = DimsVectorUtils::Count(input_blob->GetBlobDesc().dims, last_axis+1);
    int count = DimsVectorUtils::Count(output_blob->GetBlobDesc().dims);
    const int BLOCKSIZE = 128;
    if (input_blob->GetBlobDesc().data_type == DATA_TYPE_FLOAT) {
        float* input_data = static_cast<float*>(input_blob->GetHandle().base);
        float* output_data = static_cast<float*>(output_blob->GetHandle().base);
        reduce_log_sum_exp_kernel<BLOCKSIZE, float><<<count, BLOCKSIZE, BLOCKSIZE*sizeof(float), context_->GetStream()>>>(
            outer_dim, channels, inner_dim, input_data, output_data);
    } else if (input_blob->GetBlobDesc().data_type == DATA_TYPE_HALF) {
        __half* input_data = static_cast<__half*>(input_blob->GetHandle().base);
        __half* output_data = static_cast<__half*>(output_blob->GetHandle().base);
        reduce_log_sum_exp_kernel<BLOCKSIZE, __half><<<count, BLOCKSIZE, BLOCKSIZE*sizeof(float), context_->GetStream()>>>(
            outer_dim, channels, inner_dim, input_data, output_data);
    } else {
        LOGE("Error: layer acc dont support datatype: %d\n", input_blob->GetBlobDesc().data_type);
        return Status(TNNERR_MODEL_ERR, "Error: layer acc don't support datatype");
    }
    return TNN_OK;
}

REGISTER_CUDA_ACC(ReduceLogSumExp, LAYER_REDUCE_LOG_SUM_EXP);

}  // namespace TNN_NS
