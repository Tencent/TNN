// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/acc/cuda_layer_acc.h"
#include "tnn/utils/dims_vector_utils.h"

namespace TNN_NS {

DECLARE_CUDA_ACC(Acos, LAYER_ACOS);

Status CudaAcosLayerAcc::Init(Context *context, LayerParam *param, LayerResource *resource,
        const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

Status CudaAcosLayerAcc::Reshape(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

Status CudaAcosLayerAcc::Forward(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

REGISTER_CUDA_ACC(Acos, LAYER_ACOS);

}  // namespace TNN_NS