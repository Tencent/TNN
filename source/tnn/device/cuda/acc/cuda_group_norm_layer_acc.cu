#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/acc/cuda_layer_acc.h"
#include "tnn/utils/dims_vector_utils.h"

namespace TNN_NS {

DECLARE_CUDA_ACC(GroupNorm, LAYER_GROUP_NORM);

namespace {

inline static int getThreads(int count) {
    if (count <= 0) return 0;
    if (count <= 32) return 32;
    if (count > 256) return 512;
    count -= 1;
    count |= (count >> 1);
    count |= (count >> 2);
    count |= (count >> 4);
    return count + 1;
}

template<typename T>
struct Tuple2 {
    T v1; T v2;
    __device__ __host__ inline Tuple2<T>(const T a, const T b) : v1(a), v2(b) {}
    __device__ __host__ inline Tuple2<T>() : v1(0.), v2(0.) {}
    __device__ __host__ inline Tuple2<T>(const T& other): v1(other), v2(other) {}
    __device__ __host__ inline Tuple2<T> operator+(const Tuple2<T> &other) { return {v1 + other.v1, v2 + other.v2}; }
    __device__ __host__ inline Tuple2<T> &operator+=(const Tuple2<T> &other) { v1 += other.v1; v2 += other.v2; return *this; }
};

template<typename T> struct GNAccType {using type = T; };
template<> struct GNAccType<__half> {using type = float; };
template<> struct GNAccType<float> {using type = float; };

__device__ inline static Tuple2<float> __shfl_down_sync(unsigned mask, Tuple2<float> var, unsigned int delta, int width) {
    auto ret = ::__shfl_down_sync(mask, *(double *)&var, delta, width);
    return *(Tuple2<float>*)&ret;
}
// __device__ inline static Tuple2<__half> __shfl_down_sync(unsigned mask, Tuple2<__half> var, unsigned int delta, int width) {
//     auto ret = __shfl_down_sync(mask, *(float*)&var, delta, width);
//     return *(Tuple2<__half>*)&ret;
// }

template<typename T, int WARP_SIZE>
struct WarpReducer { __device__ inline static T reduce(T val); };
template<typename T> struct WarpReducer<T, 32> { __device__ inline static T reduce(T val) {
    val += __shfl_down_sync(0xffffffff, val, 16, 32);
    val += __shfl_down_sync(0x0000ffff, val, 8, 16);
    val += __shfl_down_sync(0x000000ff, val, 4, 8);
    val += __shfl_down_sync(0x0000000f, val, 2, 4);
    val += __shfl_down_sync(0x00000003, val, 1, 2);
    return val;
}};
template<typename T> struct WarpReducer<T, 16> { __device__ inline static T reduce(T val) {
    val += __shfl_down_sync(0x0000ffff, val, 8, 16);
    val += __shfl_down_sync(0x000000ff, val, 4, 8);
    val += __shfl_down_sync(0x0000000f, val, 2, 4);
    val += __shfl_down_sync(0x00000003, val, 1, 2);
    return val;
}};
template<typename T> struct WarpReducer<T, 8> { __device__ inline static T reduce(T val) {
    val += __shfl_down_sync(0x000000ff, val, 4, 8);
    val += __shfl_down_sync(0x0000000f, val, 2, 4);
    val += __shfl_down_sync(0x00000003, val, 1, 2);
    return val;
}};
template<typename T> struct WarpReducer<T, 4> { __device__ inline static T reduce(T val) {
    val += __shfl_down_sync(0x0000000f, val, 2, 4);
    val += __shfl_down_sync(0x00000003, val, 1, 2);
    return val;
}};
template<typename T> struct WarpReducer<T, 2> { __device__ inline static T reduce(T val) {
    val += __shfl_down_sync(0x00000003, val, 1, 2);
    return val;
}};
template<typename T> struct WarpReducer<T, 1> { __device__ inline static T reduce(T val) { return val; }};

template<typename T> using UFunc = T(*)(T);
template<typename T> __device__ __host__ inline T idn(T val) { return val; }
template<typename T> __device__ __host__ inline T sqr(T val) { return val * val; }
template<typename T> __device__ __host__ inline Tuple2<T> idn(Tuple2<T> val) { return val; }
template<typename T> __device__ __host__ inline Tuple2<T> idn_sqr(Tuple2<T> val) { return {val.v1, val.v2 * val.v2}; }
}

template<int THREAD_PER_BLOCK, typename T, typename AccType, UFunc<AccType> ufunc>
__device__ static void reduce(const T* input, AccType* output, const int count, const int in_elem_step = 1) {

    static_assert(THREAD_PER_BLOCK % 32 == 0 && THREAD_PER_BLOCK >= 32, "");
    __shared__ char _sm_static[(THREAD_PER_BLOCK / 32) * sizeof(AccType)];
    AccType *ssum = reinterpret_cast<AccType*>(_sm_static);
    AccType sum = AccType(0.);

    const T* ptr = input + threadIdx.x * in_elem_step;
    const auto actual_step = THREAD_PER_BLOCK * in_elem_step;
    for (int i = threadIdx.x; i < count; i += THREAD_PER_BLOCK, ptr += actual_step) {
        auto value = static_cast<AccType>(*ptr);
        sum += ufunc(value);
    }
    sum = WarpReducer<AccType, 32>::reduce(sum);
    if (threadIdx.x % 32 == 0) { ssum[threadIdx.x / 32] = sum; }
    __syncthreads();

    sum = threadIdx.x < THREAD_PER_BLOCK / 32 ? ssum[threadIdx.x] : AccType(0.);
    sum = WarpReducer<AccType, THREAD_PER_BLOCK / 32>::reduce(sum);
    if (threadIdx.x == 0) { *output = sum; }
    __syncthreads();
}


template<typename T>
__device__ void fuse_param_and_affine(const T *input, T *output, const float *gamma, const float *beta,
                                      const int c_per_g, const int hw, const float eps,
                                      typename GNAccType<T>::type sum1, typename GNAccType<T>::type sum2) {
    using AccType = typename GNAccType<T>::type;
    extern __shared__ char _sm[];
    AccType* scale = reinterpret_cast<AccType*>(_sm);
    AccType* bias = scale + c_per_g;
    const int c_off = c_per_g * blockIdx.x;
    for (int i = threadIdx.x; i < c_per_g; i += blockDim.x) {
        AccType mean = sum1 / (c_per_g * hw) ;
        AccType var = sum2 / (c_per_g * hw) - mean * mean;
        AccType k = rsqrt(var + eps) * gamma[c_off + i];
        scale[i] = k;
        bias[i] = - mean * k + beta[c_off + i];
    }
    __syncthreads();

    const auto count = c_per_g * hw;
    const auto offset = count * blockIdx.x;
    const T* in_ptr = input + offset;
    T* out_ptr = output + offset;
    for (int i = threadIdx.x; i < count; i += blockDim.x) {
        auto c_idx = i / hw;
        out_ptr[i] = static_cast<AccType>(in_ptr[i]) * scale[c_idx] + bias[c_idx];
    }
}

template<int THREAD_PER_BLOCK, typename T>
__global__ void group_norm_1pass(const T *input, T *output, const float *gamma, const float *beta,
                                 const int c_per_g, const int hw, const float eps) {
    // 1 group per block, used when c_per_g * hw <= 4096
    // assert (c == g * c_per_g)
    using AccType = typename GNAccType<T>::type;

    __shared__ char _sums[sizeof(Tuple2<AccType>)];
    Tuple2<AccType> *sums = reinterpret_cast<Tuple2<AccType>*>(_sums);
    reduce<THREAD_PER_BLOCK, T, Tuple2<AccType>, idn_sqr<AccType> >(
        input + blockIdx.x * hw * c_per_g, sums, c_per_g * hw);

    fuse_param_and_affine<T>(input, output, gamma, beta, c_per_g, hw, eps, sums[0].v1, sums[0].v2);
}

template<typename T>
static Status group_norm_v2(const T *input, T* output, const float *gamma, const float *beta,
                            const int n, const int c, const int g, const int c_per_g, const int h, const int w,
                            const float eps, hipStream_t s) {
    using AccType = typename GNAccType<T>::type;
    static std::map<int, void(*)(
        const T*, T*, const float *, const float *,
        const int, const int, const float)> group_norm_1pass_funcs = {
        {32,  group_norm_1pass<32, T>},
        {64,  group_norm_1pass<64, T>},
        {128, group_norm_1pass<128, T>},
        {256, group_norm_1pass<256, T>},
        {512, group_norm_1pass<512, T>},
    };
    const int hw = h * w;
    auto block = getThreads(c_per_g * hw);
    auto grid = n * g;
    {
        group_norm_1pass_funcs[block]<<<grid, block, 2 * c_per_g * sizeof(AccType), s>>>(
            input, output, gamma, beta, c_per_g, hw, eps);
        auto err = hipGetLastError();
        if (err != hipSuccess)
            return Status(TNNERR_CUDA_TENSORRT_ERROR, "GN Plugin 1pass failed: " + std::to_string(err));
    }
    return TNN_OK;
}

Status CudaGroupNormLayerAcc::Init(Context *context, LayerParam *param, LayerResource *resource,
        const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return CudaLayerAcc::Init(context, param, resource, inputs, outputs);
}

Status CudaGroupNormLayerAcc::Reshape(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

Status CudaGroupNormLayerAcc::Forward(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    auto params = dynamic_cast<GroupNormLayerParam*>(param_);
    auto dtype = inputs[0]->GetBlobDesc().data_type;

    Blob *input_blob = inputs[0];
    Blob *scale_blob = inputs[1];
    Blob *bias_blob  = inputs[2];
    Blob *output_blob = outputs[0];
    auto input_dims = inputs[0]->GetBlobDesc().dims;
    if (dtype == DATA_TYPE_FLOAT) {
        float* input_data = static_cast<float*>(input_blob->GetHandle().base);
        float* scale_data = static_cast<float*>(scale_blob->GetHandle().base);
        float* bias_data  = static_cast<float*>(bias_blob->GetHandle().base);
        float* output_data = static_cast<float*>(output_blob->GetHandle().base);
        int channels_per_group = input_dims[1] / params->group;

        return group_norm_v2<float>(input_data, output_data, scale_data, bias_data,
                                    input_dims[0], input_dims[1], params->group, channels_per_group,
                                    input_dims[2], input_dims[3], params->eps, context_->GetStream());
    } else if (dtype == DATA_TYPE_HALF) {
        __half* input_data = static_cast<__half*>(input_blob->GetHandle().base);
        float* scale_data = static_cast<float*>(scale_blob->GetHandle().base);
        float* bias_data  = static_cast<float*>(bias_blob->GetHandle().base);
        __half* output_data = static_cast<__half*>(output_blob->GetHandle().base);
        int channels_per_group = input_dims[1] / params->group;

        return group_norm_v2<__half>(input_data, output_data, scale_data, bias_data,
                                    input_dims[0], input_dims[1], params->group, channels_per_group,
                                    input_dims[2], input_dims[3], params->eps, context_->GetStream());
    } else {
        return Status(TNNERR_CUDA_TENSORRT_ERROR, "Unexpected data type " + std::to_string(dtype));
    }

    return TNN_OK;
}

REGISTER_CUDA_ACC(GroupNorm, LAYER_GROUP_NORM);

}  // namespace TNN_NS

