#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/acc/cuda_layer_acc.h"
#include "tnn/utils/dims_utils.h"

namespace TNN_NS {

DECLARE_CUDA_ACC(Gelu, LAYER_GELU);

__global__ void gelu_kernel(const int n, const float *in, float *out) {
    CUDA_KERNEL_LOOP(index, n) {
        const auto x   = in[index];
        out[index] = 0.5f * x * (erff(x*0.707106793288165f) + 1.0f);
    }
}

Status CudaGeluLayerAcc::Init(Context *context, LayerParam *param, LayerResource *resource,
        const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return CudaLayerAcc::Init(context, param, resource, inputs, outputs);
}

Status CudaGeluLayerAcc::Reshape(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

Status CudaGeluLayerAcc::Forward(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    Blob *input_blob  = inputs[0];
    Blob *output_blob = outputs[0];
    int count = DimsVectorUtils::Count(output_blob->GetBlobDesc().dims);
    auto data_type    = input_blob->GetBlobDesc().data_type;
    if (data_type == DATA_TYPE_FLOAT) {
        float *input_data  = static_cast<float*>(input_blob->GetHandle().base);
        float *output_data = static_cast<float*>(output_blob->GetHandle().base);
        gelu_kernel<<<TNN_CUDA_GET_BLOCKS(count), TNN_CUDA_NUM_THREADS, 0, context_->GetStream()>>>(
            count, input_data, output_data);
    } else {
        LOGE("Error: layer acc dont support datatype: %d\n", input_blob->GetBlobDesc().data_type);
        return Status(TNNERR_MODEL_ERR, "Error: layer acc don't support datatype");
    }
    return TNN_OK;
}

REGISTER_CUDA_ACC(Gelu, LAYER_GELU);

}  // namespace TNN_NS
