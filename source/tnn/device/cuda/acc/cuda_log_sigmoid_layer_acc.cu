#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License./

#include "tnn/device/cuda/acc/cuda_layer_acc.h"
#include "tnn/utils/dims_vector_utils.h"

namespace TNN_NS {

DECLARE_CUDA_ACC(LogSigmoid, LAYER_LOGSIGMOID);

__global__ void log_sigmoid_kernel(int count, const float* input, float* output) {
    CUDA_KERNEL_LOOP(index, count) {
        output[index] = __logf(1.0f / (1.0f + __expf(-input[index])));
    }
}

Status CudaLogSigmoidLayerAcc::Init(Context *context, LayerParam *param, LayerResource *resource,
        const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return CudaLayerAcc::Init(context, param, resource, inputs, outputs);
}

Status CudaLogSigmoidLayerAcc::Reshape(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

Status CudaLogSigmoidLayerAcc::Forward(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    Blob *input_blob  = inputs[0];
    Blob *output_blob = outputs[0];
    int count = DimsVectorUtils::Count(output_blob->GetBlobDesc().dims);
    if (output_blob->GetBlobDesc().data_type == DATA_TYPE_FLOAT) {
        float *input_data  = static_cast<float *>(input_blob->GetHandle().base);
        float *output_data = static_cast<float *>(output_blob->GetHandle().base);
        log_sigmoid_kernel<<<TNN_CUDA_GET_BLOCKS(count), TNN_CUDA_NUM_THREADS, 0, context_->GetStream()>>>(count,
            input_data, output_data);
    } else {
        return Status(TNNERR_LAYER_ERR, "datatype not support");
    }
    return TNN_OK;
}

REGISTER_CUDA_ACC(LogSigmoid, LAYER_LOGSIGMOID);

}  // namespace TNN_NS
