#include "hip/hip_runtime.h"
// Tencent is pleased to support the open source community by making TNN available.
//
// Copyright (C) 2020 THL A29 Limited, a Tencent company. All rights reserved.
//
// Licensed under the BSD 3-Clause License (the "License"); you may not use this file except
// in compliance with the License. You may obtain a copy of the License at
//
// https://opensource.org/licenses/BSD-3-Clause
//
// Unless required by applicable law or agreed to in writing, software distributed
// under the License is distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR
// CONDITIONS OF ANY KIND, either express or implied. See the License for the
// specific language governing permissions and limitations under the License.

#include "tnn/device/cuda/acc/cuda_layer_acc.h"
#include "tnn/utils/dims_vector_utils.h"

namespace TNN_NS {

DECLARE_CUDA_ACC(Elu, LAYER_ELU);

__global__ void elu_kernel(const int n, const float* in, float* out, float alpha) {
    CUDA_KERNEL_LOOP(index, n) {
        out[index] = in[index] < 0 ? alpha * (exp(in[index]) - 1) : in[index];
    }
}

Status CudaEluLayerAcc::Init(Context *context, LayerParam *param, LayerResource *resource,
        const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return CudaLayerAcc::Init(context, param, resource, inputs, outputs);
}

Status CudaEluLayerAcc::Reshape(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    return TNN_OK;
}

Status CudaEluLayerAcc::Forward(const std::vector<Blob *> &inputs, const std::vector<Blob *> &outputs) {
    auto params = dynamic_cast<EluLayerParam *>(param_);
    Blob *input_blob  = inputs[0];
    Blob *output_blob = outputs[0];
    int count = DimsVectorUtils::Count(output_blob->GetBlobDesc().dims);
    float* input_data = static_cast<float*>(input_blob->GetHandle().base);
    float* output_data = static_cast<float*>(output_blob->GetHandle().base);
    elu_kernel<<<TNN_CUDA_GET_BLOCKS(count), TNN_CUDA_NUM_THREADS, 0, context_->GetStream()>>>(
        count, input_data, output_data, params->alpha);
    return TNN_OK;
}

REGISTER_CUDA_ACC(Elu, LAYER_ELU);

}  // namespace TNN_NS
