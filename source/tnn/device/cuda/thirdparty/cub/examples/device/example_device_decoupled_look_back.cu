#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_scan.cuh>

#include <thrust/device_vector.h>

#include <iostream>

template <class ScanTileStateT>
__global__ void init_kernel(ScanTileStateT tile_state, int blocks_in_grid)
{
  tile_state.InitializeStatus(blocks_in_grid);
}

template <class MessageT>
__global__ void decoupled_look_back_kernel(hipcub::ScanTileState<MessageT> tile_state)
{
  using scan_op_t         = hipcub::Sum;
  using scan_tile_state_t = hipcub::ScanTileState<MessageT>;
  using tile_prefix_op    = hipcub::TilePrefixCallbackOp<MessageT, scan_op_t, scan_tile_state_t>;
  using temp_storage_t    = typename tile_prefix_op::TempStorage;

  // Allocate temp storage in shared memory
  __shared__ temp_storage_t temp_storage;

  scan_op_t scan_op{};
  constexpr unsigned int threads_in_warp = 32;
  const unsigned int tid                 = threadIdx.x;

  // Construct prefix op
  tile_prefix_op prefix(tile_state, temp_storage, scan_op);
  const unsigned int tile_idx = prefix.GetTileIdx();

  // Compute block aggregate
  MessageT block_aggregate = blockIdx.x;

  if (tile_idx == 0)
  {
    // There are no blocks to look back to, immediately set the inclusive state
    if (tid == 0)
    {
      tile_state.SetInclusive(tile_idx, block_aggregate);
      printf("tile %d: inclusive = %d\n", tile_idx, block_aggregate);
    }
  }
  else
  {
    // Only the first warp in the block can perform the look back
    const unsigned int warp_id = tid / threads_in_warp;

    if (warp_id == 0)
    {
      // Perform the decoupled look-back
      // Invocation of the prefix will block until the look-back is complete.
      MessageT exclusive_prefix = prefix(block_aggregate);

      if (tid == 0)
      {
        MessageT inclusive_prefix = scan_op(exclusive_prefix, block_aggregate);
        printf("tile %d: exclusive = %d inclusive = %d\n", tile_idx, exclusive_prefix, inclusive_prefix);
      }
    }
  }
}

template <class MessageT>
void decoupled_look_back_example(int blocks_in_grid)
{
  using scan_tile_state_t = hipcub::ScanTileState<MessageT>;

  // Query temporary storage requirements
  std::size_t temp_storage_bytes{};
  scan_tile_state_t::AllocationSize(blocks_in_grid, temp_storage_bytes);

  // Allocate temporary storage
  thrust::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);
  std::uint8_t* d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

  // Initialize temporary storage
  scan_tile_state_t tile_status;
  tile_status.Init(blocks_in_grid, d_temp_storage, temp_storage_bytes);
  constexpr unsigned int threads_in_init_block = 256;
  const unsigned int blocks_in_init_grid       = ::cuda::ceil_div(blocks_in_grid, threads_in_init_block);
  init_kernel<<<blocks_in_init_grid, threads_in_init_block>>>(tile_status, blocks_in_grid);

  // Launch decoupled look-back
  constexpr unsigned int threads_in_block = 256;
  decoupled_look_back_kernel<<<blocks_in_grid, threads_in_block>>>(tile_status);

  // Wait for kernel to finish
  hipDeviceSynchronize();
}

int main()
{
  decoupled_look_back_example<int>(14);
}
