#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockMergeSort utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <cub/block/block_merge_sort.cuh>

#include <thrust/sort.h>

#include <algorithm>

#include <c2h/catch2_test_helper.cuh>

struct CustomLess
{
  template <typename DataType>
  __device__ __host__ bool operator()(const DataType& lhs, const DataType& rhs)
  {
    return lhs < rhs;
  }
};

template <int ThreadsInBlock, int ItemsPerThread, class KeyT, class ActionT>
__global__ void block_merge_sort_kernel(KeyT* data, int valid_items, KeyT oob_default, ActionT action)
{
  using BlockMergeSort = hipcub::BlockMergeSort<KeyT, ThreadsInBlock, ItemsPerThread>;

  __shared__ typename BlockMergeSort::TempStorage temp_storage_shuffle;

  KeyT thread_data[ItemsPerThread];

  const int thread_offset = static_cast<int>(threadIdx.x) * ItemsPerThread;

  for (int item = 0; item < ItemsPerThread; item++)
  {
    const int idx     = thread_offset + item;
    thread_data[item] = idx < valid_items ? data[idx] : KeyT();
  }
  __syncthreads();

  BlockMergeSort sort(temp_storage_shuffle);

  action(sort, thread_data, valid_items, oob_default);

  for (int item = 0; item < ItemsPerThread; item++)
  {
    const int idx = thread_offset + item;

    if (idx >= valid_items)
    {
      break;
    }

    data[idx] = thread_data[item];
  }
}

template <int ThreadsInBlock, int ItemsPerThread, class KeyT, class ValueT, class ActionT>
__global__ void block_merge_sort_kernel(KeyT* keys, ValueT* vals, int valid_items, KeyT oob_default, ActionT action)
{
  using BlockMergeSort = hipcub::BlockMergeSort<KeyT, ThreadsInBlock, ItemsPerThread, ValueT>;

  __shared__ typename BlockMergeSort::TempStorage temp_storage_shuffle;

  KeyT thread_keys[ItemsPerThread];
  ValueT thread_vals[ItemsPerThread];

  const int thread_offset = static_cast<int>(threadIdx.x) * ItemsPerThread;

  for (int item = 0; item < ItemsPerThread; item++)
  {
    const int idx     = thread_offset + item;
    thread_keys[item] = idx < valid_items ? keys[idx] : KeyT{};
    thread_vals[item] = idx < valid_items ? vals[idx] : ValueT{};
  }
  __syncthreads();

  BlockMergeSort sort(temp_storage_shuffle);

  action(sort, thread_keys, thread_vals, valid_items, oob_default);

  for (int item = 0; item < ItemsPerThread; item++)
  {
    const int idx = thread_offset + item;

    if (idx >= valid_items)
    {
      break;
    }

    keys[idx] = thread_keys[item];
    vals[idx] = thread_vals[item];
  }
}

struct stable_sort_keys_partial_tile_t
{
  template <class BlockMergeSortT, class KeyT, class DefaultT>
  __device__ void operator()(BlockMergeSortT& sort, KeyT& thread_data, int valid_items, DefaultT oob_default) const
  {
    sort.StableSort(thread_data, CustomLess{}, valid_items, oob_default);
  }
};

struct stable_sort_pairs_partial_tile_t
{
  template <class BlockMergeSortT, class KeyT, class ValueT, class DefaultT>
  __device__ void
  operator()(BlockMergeSortT& sort, KeyT& thread_keys, ValueT& thread_vals, int valid_items, DefaultT oob_default) const
  {
    sort.StableSort(thread_keys, thread_vals, CustomLess{}, valid_items, oob_default);
  }
};

struct stable_sort_pairs_full_tile_t
{
  template <class BlockMergeSortT, class KeyT, class ValueT, class DefaultT>
  __device__ void operator()(
    BlockMergeSortT& sort, KeyT& thread_keys, ValueT& thread_vals, int /* valid_items */, DefaultT /* oob_default */)
    const
  {
    sort.StableSort(thread_keys, thread_vals, CustomLess());
  }
};

struct stable_sort_keys_full_tile_t
{
  template <class BlockMergeSortT, class KeyT, class DefaultT>
  __device__ void
  operator()(BlockMergeSortT& sort, KeyT& thread_keys, int /* valid_items */, DefaultT /* oob_default */) const
  {
    sort.StableSort(thread_keys, CustomLess());
  }
};

template <int ItemsPerThread, int ThreadsInBlock, class KeyT, class ActionT>
void block_merge_sort(c2h::device_vector<KeyT>& keys, ActionT action)
{
  block_merge_sort_kernel<ThreadsInBlock, ItemsPerThread><<<1, ThreadsInBlock>>>(
    thrust::raw_pointer_cast(keys.data()), static_cast<int>(keys.size()), std::numeric_limits<KeyT>::max(), action);

  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());
}

template <int ItemsPerThread, int ThreadsInBlock, class KeyT, class ValueT, class ActionT>
void block_merge_sort(c2h::device_vector<KeyT>& keys, c2h::device_vector<ValueT>& vals, ActionT action)
{
  block_merge_sort_kernel<ThreadsInBlock, ItemsPerThread><<<1, ThreadsInBlock>>>(
    thrust::raw_pointer_cast(keys.data()),
    thrust::raw_pointer_cast(vals.data()),
    static_cast<int>(keys.size()),
    std::numeric_limits<KeyT>::max(),
    action);

  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());
}

// %PARAM% THREADS_IN_BLOCK bs 64:256

using key_types        = c2h::type_list<std::int32_t, std::int64_t>;
using threads_in_block = c2h::enum_type_list<int, THREADS_IN_BLOCK>;
using items_per_thread = c2h::enum_type_list<int, 1, 2, 10, 15>;

template <class TestType>
struct params_t
{
  using key_t = typename c2h::get<0, TestType>;

  static constexpr int items_per_thread = c2h::get<1, TestType>::value;
  static constexpr int threads_in_block = c2h::get<2, TestType>::value;
  static constexpr int tile_size        = items_per_thread * threads_in_block;
};

C2H_TEST("Block merge sort can sort keys in partial tiles",
         "[merge sort][block]",
         key_types,
         items_per_thread,
         threads_in_block)
{
  using params = params_t<TestType>;
  using key_t  = typename params::key_t;

  c2h::device_vector<key_t> d_keys(GENERATE_COPY(take(10, random(0, params::tile_size))));

  c2h::gen(C2H_SEED(10), d_keys);

  c2h::host_vector<key_t> h_reference = d_keys;
  std::stable_sort(thrust::raw_pointer_cast(h_reference.data()),
                   thrust::raw_pointer_cast(h_reference.data()) + h_reference.size(),
                   CustomLess{});

  block_merge_sort<params::items_per_thread, params::threads_in_block>(d_keys, stable_sort_keys_partial_tile_t{});

  REQUIRE(h_reference == d_keys);
}

C2H_TEST(
  "Block merge sort can sort keys in full tiles", "[merge sort][block]", key_types, items_per_thread, threads_in_block)
{
  using params = params_t<TestType>;
  using key_t  = typename params::key_t;

  c2h::device_vector<key_t> d_keys(params::tile_size);

  c2h::gen(C2H_SEED(10), d_keys);

  c2h::host_vector<key_t> h_reference = d_keys;
  std::stable_sort(thrust::raw_pointer_cast(h_reference.data()),
                   thrust::raw_pointer_cast(h_reference.data()) + h_reference.size(),
                   CustomLess{});

  block_merge_sort<params::items_per_thread, params::threads_in_block>(d_keys, stable_sort_keys_full_tile_t{});

  REQUIRE(h_reference == d_keys);
}

C2H_TEST("Block merge sort can sort pairs in partial tiles",
         "[merge sort][block]",
         key_types,
         items_per_thread,
         threads_in_block)
{
  using params  = params_t<TestType>;
  using key_t   = typename params::key_t;
  using value_t = key_t;
  using pair_t  = std::pair<key_t, value_t>;

  c2h::device_vector<key_t> d_keys(GENERATE_COPY(take(10, random(0, params::tile_size))));
  c2h::device_vector<value_t> d_vals(d_keys.size());

  c2h::gen(C2H_SEED(5), d_keys);
  c2h::gen(C2H_SEED(5), d_vals);

  c2h::host_vector<key_t> h_keys   = d_keys;
  c2h::host_vector<value_t> h_vals = d_vals;

  c2h::host_vector<pair_t> h_ref(d_keys.size());

  for (std::size_t idx = 0; idx < h_ref.size(); idx++)
  {
    h_ref[idx] = std::make_pair(h_keys[idx], h_vals[idx]);
  }

  std::stable_sort(thrust::raw_pointer_cast(h_ref.data()),
                   thrust::raw_pointer_cast(h_ref.data()) + h_ref.size(),
                   [](pair_t l, pair_t r) -> bool {
                     return l.first < r.first;
                   });

  for (std::size_t idx = 0; idx < h_ref.size(); idx++)
  {
    h_keys[idx] = h_ref[idx].first;
    h_vals[idx] = h_ref[idx].second;
  }

  block_merge_sort<params::items_per_thread, params::threads_in_block>(
    d_keys, d_vals, stable_sort_pairs_partial_tile_t{});

  REQUIRE(h_keys == d_keys);
  REQUIRE(h_vals == d_vals);
}

C2H_TEST(
  "Block merge sort can sort pairs in full tiles", "[merge sort][block]", key_types, items_per_thread, threads_in_block)
{
  using params  = params_t<TestType>;
  using key_t   = typename params::key_t;
  using value_t = key_t;
  using pair_t  = std::pair<key_t, value_t>;

  c2h::device_vector<key_t> d_keys(params::tile_size);
  c2h::device_vector<value_t> d_vals(d_keys.size());

  c2h::gen(C2H_SEED(5), d_keys);
  c2h::gen(C2H_SEED(5), d_vals);

  c2h::host_vector<key_t> h_keys   = d_keys;
  c2h::host_vector<value_t> h_vals = d_vals;

  c2h::host_vector<pair_t> h_ref(d_keys.size());

  for (std::size_t idx = 0; idx < h_ref.size(); idx++)
  {
    h_ref[idx] = std::make_pair(h_keys[idx], h_vals[idx]);
  }

  std::stable_sort(thrust::raw_pointer_cast(h_ref.data()),
                   thrust::raw_pointer_cast(h_ref.data()) + h_ref.size(),
                   [](pair_t l, pair_t r) -> bool {
                     return l.first < r.first;
                   });

  for (std::size_t idx = 0; idx < h_ref.size(); idx++)
  {
    h_keys[idx] = h_ref[idx].first;
    h_vals[idx] = h_ref[idx].second;
  }

  block_merge_sort<params::items_per_thread, params::threads_in_block>(d_keys, d_vals, stable_sort_pairs_full_tile_t{});

  REQUIRE(h_keys == d_keys);
  REQUIRE(h_vals == d_vals);
}

C2H_TEST("Block merge sort can sort pairs with mixed types", "[merge sort][block]", threads_in_block)
{
  using key_t   = std::int32_t;
  using value_t = std::int64_t;
  using pair_t  = std::pair<key_t, value_t>;

  constexpr int items_per_thread = 2;
  constexpr int threads_in_block = c2h::get<0, TestType>::value;
  constexpr int tile_size        = items_per_thread * threads_in_block;

  c2h::device_vector<key_t> d_keys(tile_size);
  c2h::device_vector<value_t> d_vals(d_keys.size());

  c2h::gen(C2H_SEED(5), d_keys);
  c2h::gen(C2H_SEED(5), d_vals);

  c2h::host_vector<key_t> h_keys   = d_keys;
  c2h::host_vector<value_t> h_vals = d_vals;

  c2h::host_vector<pair_t> h_ref(d_keys.size());

  for (std::size_t idx = 0; idx < h_ref.size(); idx++)
  {
    h_ref[idx] = std::make_pair(h_keys[idx], h_vals[idx]);
  }

  std::stable_sort(thrust::raw_pointer_cast(h_ref.data()),
                   thrust::raw_pointer_cast(h_ref.data()) + h_ref.size(),
                   [](pair_t l, pair_t r) -> bool {
                     return l.first < r.first;
                   });

  for (std::size_t idx = 0; idx < h_ref.size(); idx++)
  {
    h_keys[idx] = h_ref[idx].first;
    h_vals[idx] = h_ref[idx].second;
  }

  block_merge_sort<items_per_thread, threads_in_block>(d_keys, d_vals, stable_sort_pairs_full_tile_t{});

  REQUIRE(h_keys == d_keys);
  REQUIRE(h_vals == d_vals);
}

C2H_TEST("Block merge sort can sort large tiles", "[merge sort][block]", threads_in_block)
{
  using key_t = std::uint16_t;

  constexpr int items_per_thread = 2;

  // Repurpose block sizes
  constexpr int cmake_threads_in_block = c2h::get<0, TestType>::value;
  constexpr int threads_in_block       = cmake_threads_in_block < 256 ? 512 : 1024;

  constexpr int tile_size = threads_in_block * items_per_thread;

  c2h::device_vector<key_t> d_keys(tile_size);
  c2h::gen(C2H_SEED(10), d_keys);

  c2h::host_vector<key_t> h_reference = d_keys;
  std::stable_sort(thrust::raw_pointer_cast(h_reference.data()),
                   thrust::raw_pointer_cast(h_reference.data()) + h_reference.size(),
                   CustomLess{});

  block_merge_sort<items_per_thread, threads_in_block>(d_keys, stable_sort_keys_full_tile_t{});

  REQUIRE(h_reference == d_keys);
}

C2H_TEST("Block merge sort is stable", "[merge sort][block]", threads_in_block)
{
  using key_t = c2h::custom_type_t<c2h::less_comparable_t, c2h::equal_comparable_t>;

  constexpr int items_per_thread = 2;
  constexpr int threads_in_block = c2h::get<0, TestType>::value;
  constexpr int tile_size        = threads_in_block * items_per_thread;

  c2h::device_vector<key_t> d_keys(tile_size);
  c2h::gen(C2H_SEED(10), d_keys);

  c2h::host_vector<key_t> h_reference = d_keys;
  std::stable_sort(thrust::raw_pointer_cast(h_reference.data()),
                   thrust::raw_pointer_cast(h_reference.data()) + h_reference.size(),
                   CustomLess{});

  block_merge_sort<items_per_thread, threads_in_block>(d_keys, stable_sort_keys_full_tile_t{});

  REQUIRE(h_reference == d_keys);
}
