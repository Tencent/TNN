#include <cub/util_debug.cuh>
#include <cub/util_device.cuh>

#include <c2h/catch2_test_helper.cuh>

TEST_CASE("HipcubDebug returns input error", "[debug][utils]")
{
  REQUIRE(HipcubDebug(hipSuccess) == hipSuccess);
  REQUIRE(HipcubDebug(hipErrorInvalidConfiguration) == hipErrorInvalidConfiguration);
}

TEST_CASE("HipcubDebug returns new errors", "[debug][utils]")
{
  hipcub::EmptyKernel<int><<<0, 0>>>();
  hipError_t error = hipPeekAtLastError();

  REQUIRE(error != hipSuccess);
  REQUIRE(HipcubDebug(hipSuccess) != hipSuccess);
}

TEST_CASE("HipcubDebug prefers input errors", "[debug][utils]")
{
  hipcub::EmptyKernel<int><<<0, 0>>>();
  hipError_t error = hipPeekAtLastError();

  REQUIRE(error != hipSuccess);
  REQUIRE(HipcubDebug(hipErrorOutOfMemory) != hipSuccess);
}

TEST_CASE("HipcubDebug resets last error", "[debug][utils]")
{
  hipcub::EmptyKernel<int><<<0, 0>>>();
  hipError_t error = hipPeekAtLastError();

  REQUIRE(error != hipSuccess);
  REQUIRE(HipcubDebug(hipSuccess) != hipSuccess);
  REQUIRE(HipcubDebug(hipSuccess) == hipSuccess);
}
