/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <cstdint>

#include "catch2_test_device_reduce.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>
#include <c2h/extended_types.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::TransformReduce, device_transform_reduce);

// %PARAM% TEST_LAUNCH lid 0:1:2

using types = c2h::type_list<std::uint32_t, std::uint64_t>;

template <class T>
struct square_t
{
  __host__ __device__ T operator()(const T& x) const
  {
    return x * x;
  }
};

C2H_TEST("Device transform reduce works with pointers", "[reduce][device]", types)
{
  using item_t         = c2h::get<0, TestType>;
  using init_t         = item_t;
  using offset_t       = std::int32_t;
  using reduction_op_t = hipcub::Sum;
  using transform_op_t = square_t<item_t>;

  constexpr int max_items = 5000000;
  constexpr int min_items = 1;

  const int num_items = GENERATE_COPY(take(3, random(min_items, max_items)));

  item_t init{42};
  c2h::device_vector<item_t> out(1);
  c2h::device_vector<item_t> in(num_items + 1);
  c2h::gen(C2H_SEED(2), in);

  item_t* d_in  = thrust::raw_pointer_cast(in.data());
  item_t* d_out = thrust::raw_pointer_cast(out.data());

  const c2h::host_vector<item_t> h_in = in;
  c2h::host_vector<item_t> h_transformed_in(h_in.size() - 1);

  SECTION("when aligned")
  {
    device_transform_reduce(d_in, d_out, num_items, reduction_op_t{}, transform_op_t{}, init);

    std::transform(h_in.begin(), h_in.end() - 1, h_transformed_in.begin(), transform_op_t{});
    const item_t expected = std::accumulate(h_transformed_in.begin(), h_transformed_in.end(), init);

    INFO("num_items: " << num_items);
    REQUIRE(expected == out[0]);
  }

  SECTION("when unaligned")
  {
    device_transform_reduce(d_in + 1, d_out, num_items, reduction_op_t{}, transform_op_t{}, init);

    std::transform(h_in.begin() + 1, h_in.end(), h_transformed_in.begin(), transform_op_t{});
    const item_t expected = std::accumulate(h_transformed_in.begin(), h_transformed_in.end(), init);

    INFO("num_items: " << num_items);
    REQUIRE(expected == out[0]);
  }
}

C2H_TEST("Device transform reduce works with iterators", "[reduce][device]", types)
{
  using item_t         = c2h::get<0, TestType>;
  using init_t         = item_t;
  using offset_t       = std::int32_t;
  using reduction_op_t = hipcub::Sum;
  using transform_op_t = square_t<item_t>;

  constexpr int max_items = 5000000;
  constexpr int min_items = 1;

  const int num_items = GENERATE_COPY(take(3, random(min_items, max_items)));

  const item_t magic_val{2};
  c2h::device_vector<item_t> in(num_items, magic_val);
  c2h::device_vector<item_t> out(1);

  device_transform_reduce(in.begin(), out.begin(), num_items, reduction_op_t{}, transform_op_t{}, init_t{});

  const item_t expected = num_items * magic_val * magic_val;
  const item_t actual   = out[0];

  INFO("num_items: " << num_items);
  REQUIRE(expected == actual);
}

struct input_t
{
  std::uint32_t a;
  std::uint32_t b;
};

struct transformed_input_t
{
  std::uint64_t a;
  std::uint64_t b;
};

struct init_t
{
  char a;
  char b;
};

struct accum_t
{
  std::uint64_t a;
  std::uint64_t b;

  __host__ __device__ accum_t()
      : a{42}
      , b{42}
  {}

  __host__ __device__ accum_t(const transformed_input_t& other)
      : a{other.a}
      , b{other.b}
  {}

  __host__ __device__ accum_t(const init_t& other)
      : a{static_cast<std::uint64_t>(other.a)}
      , b{static_cast<std::uint64_t>(other.b)}
  {}

  __host__ __device__ accum_t& operator=(const transformed_input_t& other)
  {
    a = other.a;
    b = other.b;
    return *this;
  }
};

struct output_t
{
  std::uint64_t a;
  std::uint64_t b;

  __host__ __device__ output_t()
      : a{42}
      , b{42}
  {}

  __host__ __device__ output_t(const accum_t& other)
      : a{other.a}
      , b{other.b}
  {}

  __host__ __device__ output_t(const init_t& other)
      : a{static_cast<std::uint64_t>(other.a)}
      , b{static_cast<std::uint64_t>(other.b)}
  {}
};

struct transform_op_t
{
  __host__ __device__ transformed_input_t operator()(const input_t& x) const
  {
    return {static_cast<std::uint64_t>(x.a * x.a), static_cast<std::uint64_t>(x.b * x.b)};
  }
};

struct reduction_op_t
{
  __host__ __device__ accum_t operator()(accum_t x, accum_t y) const
  {
    accum_t result{};
    result.a = x.a + y.a;
    result.b = x.b + y.b;
    return result;
  }
};

C2H_TEST("Device transform reduce doesn't let input type into reduction op", "[reduce][device]")
{
  constexpr int max_items = 5000000;
  constexpr int min_items = 1;

  const int num_items = GENERATE_COPY(take(3, random(min_items, max_items)));

  const init_t init{3, 3};
  const input_t magic_val{2, 2};

  c2h::device_vector<input_t> in(num_items, magic_val);
  c2h::device_vector<output_t> out(1);

  input_t* d_in   = thrust::raw_pointer_cast(in.data());
  output_t* d_out = thrust::raw_pointer_cast(out.data());

  device_transform_reduce(d_in, d_out, num_items, reduction_op_t{}, transform_op_t{}, init);

  const std::uint64_t expected = num_items * magic_val.a * magic_val.a + init.a;
  const output_t actual        = out[0];

  INFO("num_items: " << num_items);
  REQUIRE(expected == actual.a);
  REQUIRE(expected == actual.b);
}
