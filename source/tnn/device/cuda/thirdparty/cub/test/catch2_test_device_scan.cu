#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_scan.cuh>

#include <cstdint>

#include "catch2_test_device_reduce.cuh"
#include "catch2_test_device_scan.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::InclusiveScanInit, device_inclusive_scan_with_init);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::ExclusiveSum, device_exclusive_sum);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::ExclusiveScan, device_exclusive_scan);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::InclusiveSum, device_inclusive_sum);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::InclusiveScan, device_inclusive_scan);

// %PARAM% TEST_LAUNCH lid 0:1:2
// %PARAM% TEST_TYPES types 0:1:2:3

// List of types to test
using custom_t =
  c2h::custom_type_t<c2h::accumulateable_t,
                     c2h::equal_comparable_t,
                     c2h::lexicographical_less_comparable_t,
                     c2h::lexicographical_greater_comparable_t>;

#if TEST_TYPES == 0
using full_type_list = c2h::type_list<type_pair<std::uint8_t, std::int32_t>, type_pair<std::int8_t>>;
#elif TEST_TYPES == 1
using full_type_list = c2h::type_list<type_pair<std::int32_t>, type_pair<std::uint64_t>>;
#elif TEST_TYPES == 2
using full_type_list = c2h::type_list<type_pair<uchar3>, type_pair<ulonglong4>>;
#elif TEST_TYPES == 3
// clang-format off
using full_type_list = c2h::type_list<
type_pair<custom_t>
#if TEST_HALF_T
, type_pair<half_t> // testing half
#endif
#if TEST_BF_T
, type_pair<bfloat16_t> // testing bf16
#endif
>;
// clang-format on
#endif

/**
 * @brief Input data generation mode
 */
enum class gen_data_t : int
{
  /// Uniform random data generation
  GEN_TYPE_RANDOM,
  /// Constant value as input data
  GEN_TYPE_CONST
};

C2H_TEST("Device scan works with all device interfaces", "[scan][device]", full_type_list)
{
  using params   = params_t<TestType>;
  using input_t  = typename params::item_t;
  using output_t = typename params::output_t;
  using offset_t = int32_t;

  constexpr offset_t min_items = 1;
  constexpr offset_t max_items = 1000000;

  // Generate the input sizes to test for
  const offset_t num_items = GENERATE_COPY(
    take(3, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));

  // Input data generation to test
  const gen_data_t data_gen_mode = GENERATE_COPY(gen_data_t::GEN_TYPE_RANDOM, gen_data_t::GEN_TYPE_CONST);

  // Generate input data
  c2h::device_vector<input_t> in_items(num_items);
  if (data_gen_mode == gen_data_t::GEN_TYPE_RANDOM)
  {
    c2h::gen(C2H_SEED(2), in_items);
  }
  else
  {
    input_t default_constant{};
    init_default_constant(default_constant);
    thrust::fill(c2h::device_policy, in_items.begin(), in_items.end(), default_constant);
  }
  auto d_in_it = thrust::raw_pointer_cast(in_items.data());

// Skip DeviceScan::InclusiveSum and DeviceScan::ExclusiveSum tests for extended floating-point
// types because of unbounded epsilon due to pseudo associativity of the addition operation over
// floating point numbers
#if TEST_TYPES != 3
  SECTION("inclusive sum")
  {
    using op_t    = hipcub::Sum;
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, input_t>;

    // Prepare verification data
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<output_t> expected_result(num_items);
    compute_inclusive_scan_reference(host_items.cbegin(), host_items.cend(), expected_result.begin(), op_t{}, accum_t{});

    // Run test
    c2h::device_vector<output_t> out_result(num_items);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    device_inclusive_sum(d_in_it, d_out_it, num_items);

    // Verify result
    REQUIRE(expected_result == out_result);

    // Run test in-place
    _CCCL_IF_CONSTEXPR (std::is_same<input_t, output_t>::value)
    {
      device_inclusive_sum(d_in_it, d_in_it, num_items);

      // Verify result
      REQUIRE(expected_result == in_items);
    }
  }

  SECTION("exclusive sum")
  {
    using op_t    = hipcub::Sum;
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, input_t>;

    // Prepare verification data
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<output_t> expected_result(num_items);
    compute_exclusive_scan_reference(host_items.cbegin(), host_items.cend(), expected_result.begin(), accum_t{}, op_t{});

    // Run test
    c2h::device_vector<output_t> out_result(num_items);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    device_exclusive_sum(d_in_it, d_out_it, num_items);

    // Verify result
    REQUIRE(expected_result == out_result);

    // Run test in-place
    _CCCL_IF_CONSTEXPR (std::is_same<input_t, output_t>::value)
    {
      device_exclusive_sum(d_in_it, d_in_it, num_items);

      // Verify result
      REQUIRE(expected_result == in_items);
    }
  }
#endif

  SECTION("inclusive scan")
  {
    using op_t    = hipcub::Min;
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, input_t>;

    // Prepare verification data
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<output_t> expected_result(num_items);
    compute_inclusive_scan_reference(
      host_items.cbegin(), host_items.cend(), expected_result.begin(), op_t{}, hipcub::NumericTraits<accum_t>::Max());

    // Run test
    c2h::device_vector<output_t> out_result(num_items);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    device_inclusive_scan(unwrap_it(d_in_it), unwrap_it(d_out_it), op_t{}, num_items);

    // Verify result
    REQUIRE(expected_result == out_result);

    // Run test in-place
    _CCCL_IF_CONSTEXPR (std::is_same<input_t, output_t>::value)
    {
      device_inclusive_scan(unwrap_it(d_in_it), unwrap_it(d_in_it), op_t{}, num_items);

      // Verify result
      REQUIRE(expected_result == in_items);
    }
  }

  SECTION("inclusive scan with init value")
  {
    using op_t    = hipcub::Sum;
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, input_t>;

    // Scan operator
    auto scan_op = unwrap_op(reference_extended_fp(d_in_it), op_t{});

    // Prepare verification data
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<output_t> expected_result(num_items);

    // Run test
    c2h::device_vector<output_t> out_result(num_items);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    accum_t init_value{};
    init_default_constant(init_value);
    compute_inclusive_scan_reference(
      host_items.cbegin(), host_items.cend(), expected_result.begin(), scan_op, init_value);

    device_inclusive_scan_with_init(unwrap_it(d_in_it), unwrap_it(d_out_it), scan_op, init_value, num_items);

    // Verify result
    REQUIRE(expected_result == out_result);

    // Run test in-place
    _CCCL_IF_CONSTEXPR (std::is_same<input_t, output_t>::value)
    {
      device_inclusive_scan_with_init(unwrap_it(d_in_it), unwrap_it(d_in_it), scan_op, init_value, num_items);

      // Verify result
      REQUIRE(expected_result == in_items);
    }
  }

  SECTION("exclusive scan")
  {
    using op_t    = hipcub::Sum;
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, input_t>;

    // Scan operator
    auto scan_op = unwrap_op(reference_extended_fp(d_in_it), op_t{});

    // Prepare verification data
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<output_t> expected_result(num_items);
    compute_exclusive_scan_reference(
      host_items.cbegin(), host_items.cend(), expected_result.begin(), accum_t{}, scan_op);

    // Run test
    c2h::device_vector<output_t> out_result(num_items);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    using init_t  = hipcub::detail::value_t<decltype(unwrap_it(d_out_it))>;
    device_exclusive_scan(unwrap_it(d_in_it), unwrap_it(d_out_it), scan_op, init_t{}, num_items);

    // Verify result
    REQUIRE(expected_result == out_result);

    // Run test in-place
    _CCCL_IF_CONSTEXPR (std::is_same<input_t, output_t>::value)
    {
      device_exclusive_scan(unwrap_it(d_in_it), unwrap_it(d_in_it), scan_op, init_t{}, num_items);

      // Verify result
      REQUIRE(expected_result == in_items);
    }
  }

  SECTION("exclusive scan with future-init value")
  {
    using op_t    = hipcub::Sum;
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, input_t>;

    // Scan operator
    auto scan_op = unwrap_op(reference_extended_fp(d_in_it), op_t{});

    // Prepare verification data
    accum_t init_value{};
    init_default_constant(init_value);
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<output_t> expected_result(num_items);
    compute_exclusive_scan_reference(
      host_items.cbegin(), host_items.cend(), expected_result.begin(), init_value, scan_op);

    // Run test
    c2h::device_vector<output_t> out_result(num_items);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    using init_t  = hipcub::detail::value_t<decltype(unwrap_it(d_out_it))>;
    c2h::device_vector<init_t> d_initial_value(1);
    d_initial_value[0]     = static_cast<init_t>(*unwrap_it(&init_value));
    auto future_init_value = hipcub::FutureValue<init_t>(thrust::raw_pointer_cast(d_initial_value.data()));
    device_exclusive_scan(unwrap_it(d_in_it), unwrap_it(d_out_it), scan_op, future_init_value, num_items);

    // Verify result
    REQUIRE(expected_result == out_result);

    // Run test in-place
    _CCCL_IF_CONSTEXPR (std::is_same<input_t, output_t>::value)
    {
      device_exclusive_scan(unwrap_it(d_in_it), unwrap_it(d_in_it), scan_op, future_init_value, num_items);

      // Verify result
      REQUIRE(expected_result == in_items);
    }
  }
}
