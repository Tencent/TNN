#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_partition.cuh>

#include <thrust/distance.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/tabulate_output_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/partition.h>
#include <thrust/reverse.h>

#include <cuda/cmath>

#include <algorithm>

#include "catch2_test_device_select_common.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DevicePartition::If, partition_if);

// %PARAM% TEST_LAUNCH lid 0:1:2

struct always_false_t
{
  template <typename T>
  __device__ bool operator()(const T&) const
  {
    return false;
  }
};

struct always_true_t
{
  template <typename T>
  __device__ bool operator()(const T&) const
  {
    return true;
  }
};

using all_types =
  c2h::type_list<std::uint8_t,
                 std::uint16_t,
                 std::uint32_t,
                 std::uint64_t,
                 ulonglong2,
                 ulonglong4,
                 int,
                 long2,
                 c2h::custom_type_t<c2h::less_comparable_t, c2h::equal_comparable_t>>;

using types = c2h::
  type_list<std::uint8_t, std::uint32_t, ulonglong4, c2h::custom_type_t<c2h::less_comparable_t, c2h::equal_comparable_t>>;

// List of offset types to be used for testing large number of items
using offset_types = c2h::type_list<std::int32_t, std::uint32_t, std::uint64_t>;

C2H_TEST("DevicePartition::If can run with empty input", "[device][partition_if]", types)
{
  using type = typename c2h::get<0, TestType>;

  constexpr int num_items = 0;
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 42);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_if(in.begin(), out.begin(), d_num_selected_out, num_items, always_true_t{});

  REQUIRE(num_selected_out[0] == 0);
}

C2H_TEST("DevicePartition::If handles all matched", "[device][partition_if]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_if(in.begin(), out.begin(), d_first_num_selected_out, num_items, always_true_t{});

  REQUIRE(num_selected_out[0] == num_items);
  REQUIRE(out == in);
}

C2H_TEST("DevicePartition::If handles no matched", "[device][partition_if]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_if(in.begin(), out.begin(), d_first_num_selected_out, num_items, always_false_t{});

  // The false partition is in reverse order
  thrust::reverse(c2h::device_policy, out.begin(), out.end());

  REQUIRE(num_selected_out[0] == 0);
  REQUIRE(out == in);
}

C2H_TEST("DevicePartition::If does not change input", "[device][partition_if]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // just pick one of the input elements as boundary
  less_than_t<type> le{in[num_items / 2]};

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // copy input first
  c2h::device_vector<type> reference = in;

  partition_if(in.begin(), out.begin(), d_first_num_selected_out, num_items, le);

  REQUIRE(reference == in);
}

C2H_TEST("DevicePartition::If is stable", "[device][partition_if]")
{
  using type = c2h::custom_type_t<c2h::less_comparable_t, c2h::equal_comparable_t>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // just pick one of the input elements as boundary
  less_than_t<type> le{in[num_items / 2]};

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  // The main difference between stable_partition and DevicePartition::If is that the false partition is in reverse
  // order
  const auto boundary = std::stable_partition(reference.begin(), reference.end(), le);
  std::reverse(boundary, reference.end());

  partition_if(in.begin(), out.begin(), d_first_num_selected_out, num_items, le);

  REQUIRE(num_selected_out[0] == thrust::distance(reference.begin(), boundary));
  REQUIRE(reference == out);
}

C2H_TEST("DevicePartition::If works with iterators", "[device][partition_if]", all_types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // just pick one of the input elements as boundary
  less_than_t<type> le{in[num_items / 2]};

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  // The main difference between stable_partition and DevicePartition::If is that the false partition is in reverse
  // order
  const auto boundary = std::stable_partition(reference.begin(), reference.end(), le);
  std::reverse(boundary, reference.end());

  partition_if(in.begin(), out.begin(), d_first_num_selected_out, num_items, le);

  REQUIRE(num_selected_out[0] == thrust::distance(reference.begin(), boundary));
  REQUIRE(reference == out);
}

C2H_TEST("DevicePartition::If works with pointers", "[device][partition_if]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // just pick one of the input elements as boundary
  less_than_t<type> le{in[num_items / 2]};

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  // The main difference between stable_partition and DevicePartition::If is that the false partition is in reverse
  // order
  const auto boundary = std::stable_partition(reference.begin(), reference.end(), le);
  std::reverse(boundary, reference.end());

  partition_if(
    thrust::raw_pointer_cast(in.data()), thrust::raw_pointer_cast(out.data()), d_first_num_selected_out, num_items, le);

  REQUIRE(num_selected_out[0] == thrust::distance(reference.begin(), boundary));
  REQUIRE(reference == out);
}

template <class T>
struct convertible_from_T
{
  T val_;

  convertible_from_T() = default;
  __host__ __device__ convertible_from_T(const T& val) noexcept
      : val_(val)
  {}
  __host__ __device__ convertible_from_T& operator=(const T& val) noexcept
  {
    val_ = val;
  }
  // Converting back to T helps satisfy all the machinery that T supports
  __host__ __device__ operator T() const noexcept
  {
    return val_;
  }
};

C2H_TEST("DevicePartition::If works with a different output type", "[device][partition_if]")
{
  using type = c2h::custom_type_t<c2h::less_comparable_t, c2h::equal_comparable_t>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<convertible_from_T<type>> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // just pick one of the input elements as boundary
  less_than_t<type> le{in[num_items / 2]};

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  // The main difference between stable_partition and DevicePartition::If is that the false partition is in reverse
  // order
  const auto boundary = std::stable_partition(reference.begin(), reference.end(), le);
  std::reverse(boundary, reference.end());

  partition_if(in.begin(), out.begin(), d_first_num_selected_out, num_items, le);

  REQUIRE(num_selected_out[0] == thrust::distance(reference.begin(), boundary));
  REQUIRE(reference == out);
}

C2H_TEST("DevicePartition::If works for very large number of items", "[device][partition_if]", offset_types)
try
{
  using type     = std::int64_t;
  using offset_t = typename c2h::get<0, TestType>;

  auto num_items_max_ull =
    std::min(static_cast<std::size_t>(::cuda::std::numeric_limits<offset_t>::max()),
             ::cuda::std::numeric_limits<std::uint32_t>::max() + static_cast<std::size_t>(2000000ULL));
  offset_t num_items_max = static_cast<offset_t>(num_items_max_ull);
  offset_t num_items_min =
    num_items_max_ull > 10000 ? static_cast<offset_t>(num_items_max_ull - 10000ULL) : offset_t{0};
  offset_t num_items = GENERATE_COPY(
    values(
      {num_items_max, static_cast<offset_t>(num_items_max - 1), static_cast<offset_t>(1), static_cast<offset_t>(3)}),
    take(2, random(num_items_min, num_items_max)));

  auto in = thrust::make_counting_iterator(offset_t{0});

  // We select the first <cut_off_index> items and reject the rest
  const offset_t cut_off_index = num_items / 4;

  // Prepare tabulate output iterator to verify results in a memory-efficient way:
  // We use a tabulate iterator that checks whenever the partition algorithm writes an output whether that item
  // corresponds to the expected value at that index and, if correct, sets a boolean flag at that index.
  static constexpr auto bits_per_element = 8 * sizeof(std::uint32_t);
  c2h::device_vector<std::uint32_t> correctness_flags(::cuda::ceil_div(num_items, bits_per_element));
  auto expected_selected_it = thrust::make_counting_iterator(offset_t{0});
  auto expected_rejected_it = thrust::make_reverse_iterator(
    thrust::make_counting_iterator(offset_t{cut_off_index}) + (num_items - cut_off_index));
  auto expected_result_op =
    make_index_to_expected_partition_op(expected_selected_it, expected_rejected_it, cut_off_index);
  auto expected_result_it =
    thrust::make_transform_iterator(thrust::make_counting_iterator(offset_t{0}), expected_result_op);
  auto check_result_op = make_checking_write_op(expected_result_it, thrust::raw_pointer_cast(correctness_flags.data()));
  auto check_result_it = thrust::make_tabulate_output_iterator(check_result_op);

  // Needs to be device accessible
  c2h::device_vector<offset_t> num_selected_out(1, 0);
  offset_t* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // Run test
  partition_if(
    in, check_result_it, d_first_num_selected_out, num_items, less_than_t<type>{static_cast<type>(cut_off_index)});

  // Ensure that we created the correct output
  REQUIRE(num_selected_out[0] == cut_off_index);
  bool all_results_correct = are_all_flags_set(correctness_flags, num_items);
  REQUIRE(all_results_correct == true);
}
catch (std::bad_alloc&)
{
  // Exceeding memory is not a failure.
}
