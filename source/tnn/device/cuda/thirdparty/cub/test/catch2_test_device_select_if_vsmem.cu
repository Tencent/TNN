/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_select.cuh>

#include <algorithm>

#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/vector.cuh>

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSelect::If, select_if);

using types = c2h::type_list<
  // Type large enough to dispatch to the fallback policy
  c2h::custom_type_t<c2h::equal_comparable_t, c2h::less_comparable_t, c2h::huge_data<256>::type>,
  // Type large enough to require virtual shared memory
  c2h::custom_type_t<c2h::equal_comparable_t, c2h::less_comparable_t, c2h::huge_data<512>::type>>;

template <typename T>
struct less_than_t
{
  T compare;

  explicit __host__ less_than_t(T compare)
      : compare(compare)
  {}

  __host__ __device__ bool operator()(const T& a) const
  {
    return a < compare;
  }
};

C2H_TEST("DeviceSelect::If works for large types", "[select_if][vsmem][device]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 10000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  // Just pick one of the input elements as boundary
  less_than_t<type> le{in[num_items / 2]};

  // Run test
  c2h::device_vector<int> num_selected_out(1, 0);
  select_if(in.begin(), out.begin(), num_selected_out.begin(), num_items, le);

  // Ensure that we create the same output as std
  c2h::host_vector<type> reference = in;
  std::stable_partition(reference.begin(), reference.end(), le);

  out.resize(num_selected_out[0]);
  reference.resize(num_selected_out[0]);
  REQUIRE(reference == out);
}
