#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_partition.cuh>

#include <thrust/count.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/tabulate_output_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/partition.h>
#include <thrust/reverse.h>

#include <cuda/cmath>

#include <algorithm>

#include "catch2_test_device_select_common.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>

template <class T, class FlagT>
static c2h::host_vector<T> get_reference(const c2h::device_vector<T>& in, const c2h::device_vector<FlagT>& flags)
{
  struct selector
  {
    const T* ref_begin      = nullptr;
    const FlagT* flag_begin = nullptr;

    constexpr selector(const T* ref, const FlagT* flag) noexcept
        : ref_begin(ref)
        , flag_begin(flag)
    {}

    bool operator()(const T& val) const
    {
      const auto pos = &val - ref_begin;
      return static_cast<bool>(flag_begin[pos]);
    }
  };

  c2h::host_vector<T> reference   = in;
  c2h::host_vector<FlagT> h_flags = flags;

  const selector pred{thrust::raw_pointer_cast(reference.data()), thrust::raw_pointer_cast(h_flags.data())};
  const auto boundary = std::stable_partition(reference.begin(), reference.end(), pred);
  std::reverse(boundary, reference.end()); // the false partition is in reverse order
  return reference;
}

DECLARE_LAUNCH_WRAPPER(hipcub::DevicePartition::Flagged, partition_flagged);

// %PARAM% TEST_LAUNCH lid 0:1:2

using all_types =
  c2h::type_list<std::uint8_t,
                 std::uint16_t,
                 std::uint32_t,
                 std::uint64_t,
                 ulonglong2,
                 ulonglong4,
                 int,
                 long2,
                 c2h::custom_type_t<c2h::equal_comparable_t>>;

using types = c2h::type_list<std::uint8_t, std::uint32_t, ulonglong4, c2h::custom_type_t<c2h::equal_comparable_t>>;

// List of offset types to be used for testing large number of items
using offset_types = c2h::type_list<std::int32_t, std::uint32_t, std::uint64_t>;

C2H_TEST("DevicePartition::Flagged can run with empty input", "[device][partition_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  constexpr int num_items = 0;
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::device_vector<char> flags(num_items);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 42);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected_out[0] == 0);
}

C2H_TEST("DevicePartition::Flagged handles all matched", "[device][partition_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<char> flags(num_items, static_cast<char>(1));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected_out[0] == num_items);
  REQUIRE(out == in);
}

C2H_TEST("DevicePartition::Flagged handles no matched", "[device][partition_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<char> flags(num_items, static_cast<char>(0));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  // The false partition is in reverse order
  thrust::reverse(c2h::device_policy, out.begin(), out.end());

  REQUIRE(num_selected_out[0] == 0);
  REQUIRE(out == in);
}

C2H_TEST("DevicePartition::Flagged does not change input", "[device][partition_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // copy input first
  c2h::device_vector<type> reference = in;

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == in);
}

C2H_TEST("DevicePartition::Flagged is stable", "[device][partition_flagged]")
{
  using type = c2h::custom_type_t<c2h::equal_comparable_t>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DevicePartition::Flagged works with iterators", "[device][partition_flagged]", all_types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DevicePartition::Flagged works with pointers", "[device][partition_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(
    thrust::raw_pointer_cast(in.data()),
    thrust::raw_pointer_cast(flags.data()),
    thrust::raw_pointer_cast(out.data()),
    d_num_selected_out,
    num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

struct convertible_to_bool
{
  int val_;

  convertible_to_bool() = default;
  __host__ __device__ convertible_to_bool(const int val) noexcept
      : val_(val)
  {}

  __host__ __device__ operator bool() const noexcept
  {
    return static_cast<bool>(val_);
  }
  __host__ __device__ friend bool operator==(const convertible_to_bool& lhs, const int& rhs) noexcept
  {
    return lhs.val_ == rhs;
  }
  __host__ __device__ friend bool operator==(const int& lhs, const convertible_to_bool& rhs) noexcept
  {
    return lhs == rhs.val_;
  }
};

C2H_TEST("DevicePartition::Flagged works with flags that are convertible to bool", "[device][partition_flagged]")
{
  using type = c2h::custom_type_t<c2h::equal_comparable_t>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> iflags(num_items);
  c2h::gen(C2H_SEED(1), iflags, 0, 1);

  c2h::device_vector<convertible_to_bool> flags = iflags;
  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DevicePartition::Flagged works with flags that alias input", "[device][partition_flagged]")
{
  using type = int;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> out(num_items);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(flags, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(flags.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

template <class T>
struct convertible_from_T
{
  T val_;

  convertible_from_T() = default;
  __host__ __device__ convertible_from_T(const T& val) noexcept
      : val_(val)
  {}
  __host__ __device__ convertible_from_T& operator=(const T& val) noexcept
  {
    val_ = val;
  }
  // Converting back to T helps satisfy all the machinery that T supports
  __host__ __device__ operator T() const noexcept
  {
    return val_;
  }
};

C2H_TEST("DevicePartition::Flagged works with different output type", "[device][partition_flagged]")
{
  using type = c2h::custom_type_t<c2h::equal_comparable_t>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<convertible_from_T<type>> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  partition_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DevicePartition::Flagged works for very large number of items", "[device][partition_flagged]", offset_types)
try
{
  using type     = std::int64_t;
  using offset_t = typename c2h::get<0, TestType>;

  auto num_items_max_ull =
    std::min(static_cast<std::size_t>(::cuda::std::numeric_limits<offset_t>::max()),
             ::cuda::std::numeric_limits<std::uint32_t>::max() + static_cast<std::size_t>(2000000ULL));
  offset_t num_items_max = static_cast<offset_t>(num_items_max_ull);
  offset_t num_items_min =
    num_items_max_ull > 10000 ? static_cast<offset_t>(num_items_max_ull - 10000ULL) : offset_t{0};
  offset_t num_items = GENERATE_COPY(
    values(
      {num_items_max, static_cast<offset_t>(num_items_max - 1), static_cast<offset_t>(1), static_cast<offset_t>(3)}),
    take(2, random(num_items_min, num_items_max)));

  // We select the first <cut_off_index> items and reject the rest
  const offset_t cut_off_index = num_items / 4;

  auto in       = thrust::make_counting_iterator(offset_t{0});
  auto in_flags = thrust::make_transform_iterator(
    thrust::make_counting_iterator(offset_t{0}), less_than_t<type>{static_cast<type>(cut_off_index)});

  // Prepare tabulate output iterator to verify results in a memory-efficient way:
  // We use a tabulate iterator that checks whenever the partition algorithm writes an output whether that item
  // corresponds to the expected value at that index and, if correct, sets a boolean flag at that index.
  static constexpr auto bits_per_element = 8 * sizeof(std::uint32_t);
  c2h::device_vector<std::uint32_t> correctness_flags(::cuda::ceil_div(num_items, bits_per_element));
  auto expected_selected_it = thrust::make_counting_iterator(offset_t{0});
  auto expected_rejected_it = thrust::make_reverse_iterator(
    thrust::make_counting_iterator(offset_t{cut_off_index}) + (num_items - cut_off_index));
  auto expected_result_op =
    make_index_to_expected_partition_op(expected_selected_it, expected_rejected_it, cut_off_index);
  auto expected_result_it =
    thrust::make_transform_iterator(thrust::make_counting_iterator(offset_t{0}), expected_result_op);
  auto check_result_op = make_checking_write_op(expected_result_it, thrust::raw_pointer_cast(correctness_flags.data()));
  auto check_result_it = thrust::make_tabulate_output_iterator(check_result_op);

  // Needs to be device accessible
  c2h::device_vector<offset_t> num_selected_out(1, 0);
  offset_t* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // Run test
  partition_flagged(in, in_flags, check_result_it, d_first_num_selected_out, num_items);

  // Ensure that we created the correct output
  REQUIRE(num_selected_out[0] == cut_off_index);
  bool all_results_correct = are_all_flags_set(correctness_flags, num_items);
  REQUIRE(all_results_correct == true);
}
catch (std::bad_alloc&)
{
  // Exceeding memory is not a failure.
}
