#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/util_macro.cuh>
#include <cub/util_ptx.cuh>
#include <cub/warp/warp_merge_sort.cuh>

#include <thrust/iterator/constant_iterator.h>

#include <cuda/std/type_traits>

#include <algorithm>

#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>

struct CustomLess
{
  template <typename T>
  __device__ __host__ bool operator()(const T& lhs, const T& rhs)
  {
    return lhs < rhs;
  }
};

/**
 * @brief Kernel to dispatch to the appropriate WarpMergeSort member function, sorting keys-only.
 */
template <int ITEMS_PER_THREAD,
          int LOGICAL_WARP_THREADS,
          int TOTAL_WARPS,
          typename T,
          typename SegmentSizeItT,
          typename ActionT>
__global__ void warp_merge_sort_kernel(T* in, T* out, SegmentSizeItT segment_sizes, T oob_default, ActionT action)
{
  using warp_merge_sort_t = hipcub::WarpMergeSort<T, ITEMS_PER_THREAD, LOGICAL_WARP_THREADS>;
  using storage_t         = typename warp_merge_sort_t::TempStorage;

  // Get linear thread and warp index
  const int tid     = threadIdx.x;
  const int warp_id = tid / LOGICAL_WARP_THREADS;

  // Test case of partially finished CTA
  if (warp_id >= TOTAL_WARPS)
  {
    return;
  }

  // Thread-local storage & warp-scope temporary storage allocation
  T thread_data[ITEMS_PER_THREAD];
  __shared__ storage_t storage[TOTAL_WARPS];

  // Instantiate warp-scope algorithm
  warp_merge_sort_t warp_sort(storage[warp_id]);

  const int warp_offset   = LOGICAL_WARP_THREADS * ITEMS_PER_THREAD * warp_id;
  const int thread_offset = warp_offset + warp_sort.get_linear_tid() * ITEMS_PER_THREAD;
  const int valid_items   = segment_sizes[warp_id];

  // Load data
  for (int item = 0; item < ITEMS_PER_THREAD; item++)
  {
    const int idx     = thread_offset + item;
    thread_data[item] = in[idx];
  }
  hipcub::WARP_SYNC(warp_sort.get_member_mask());

  // Run merge sort test
  action(warp_sort, thread_data, valid_items, oob_default);

  // Store data
  for (int item = 0; item < ITEMS_PER_THREAD; item++)
  {
    const int idx = thread_offset + item;
    out[idx]      = (idx - warp_offset) >= valid_items ? oob_default : thread_data[item];
  }
}

/**
 * @brief Kernel to dispatch to the appropriate WarpMergeSort member function, sorting key-value
 * pairs.
 */
template <int ITEMS_PER_THREAD,
          int LOGICAL_WARP_THREADS,
          int TOTAL_WARPS,
          typename KeyT,
          typename ValueT,
          typename SegmentSizeItT,
          typename ActionT>
__global__ void warp_merge_sort_kernel(
  KeyT* keys_in,
  KeyT* keys_out,
  ValueT* values_in,
  ValueT* values_out,
  SegmentSizeItT segment_sizes,
  KeyT oob_default,
  ActionT action)
{
  using warp_merge_sort_t = hipcub::WarpMergeSort<KeyT, ITEMS_PER_THREAD, LOGICAL_WARP_THREADS, ValueT>;
  using storage_t         = typename warp_merge_sort_t::TempStorage;

  // Get linear thread and warp index
  const int tid     = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);
  const int warp_id = tid / LOGICAL_WARP_THREADS;

  // Test case of partially finished CTA
  if (warp_id >= TOTAL_WARPS)
  {
    return;
  }

  // Thread-local storage & warp-scope temporary storage allocation
  KeyT keys[ITEMS_PER_THREAD];
  ValueT values[ITEMS_PER_THREAD];
  __shared__ storage_t storage[TOTAL_WARPS];

  // Instantiate warp-scope algorithm
  warp_merge_sort_t warp_sort(storage[warp_id]);

  const int warp_offset   = LOGICAL_WARP_THREADS * ITEMS_PER_THREAD * warp_id;
  const int thread_offset = warp_offset + warp_sort.get_linear_tid() * ITEMS_PER_THREAD;
  const int valid_items   = segment_sizes[warp_id];

  // Load data
  for (int item = 0; item < ITEMS_PER_THREAD; item++)
  {
    const int idx = thread_offset + item;
    keys[item]    = keys_in[idx];
    values[item]  = values_in[idx];
  }
  hipcub::WARP_SYNC(warp_sort.get_member_mask());

  // Run merge sort test
  action(warp_sort, keys, values, valid_items, oob_default);

  // Store data
  for (int item = 0; item < ITEMS_PER_THREAD; item++)
  {
    const int idx   = thread_offset + item;
    keys_out[idx]   = (idx - warp_offset) >= valid_items ? oob_default : keys[item];
    values_out[idx] = (idx - warp_offset) >= valid_items ? ValueT{} : values[item];
  }
}

// -----------------------------------------------------------
// Dimensions being instantiated:
// {full,partial} x {stable, 'unstable'} x {keys, kv-pairs}
// -----------------------------------------------------------

/**
 * @brief Delegate wrapper for WarpMergeSort::StableSort on keys-only
 */
struct warp_stable_sort_keys_t
{
  template <typename T, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void
  operator()(WarpSortT& warp_sort, T (&thread_data)[ITEMS_PER_THREAD], int /*valid_items*/, T /*oob_default*/) const
  {
    warp_sort.StableSort(thread_data, CustomLess{});
  }
};

/**
 * @brief Delegate wrapper for partial WarpMergeSort::StableSort keys-only
 */
struct warp_partial_stable_sort_keys_t
{
  template <typename T, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void
  operator()(WarpSortT& warp_sort, T (&thread_data)[ITEMS_PER_THREAD], int valid_items, T oob_default) const
  {
    warp_sort.StableSort(thread_data, CustomLess{}, valid_items, oob_default);
  }
};

/**
 * @brief Delegate wrapper for WarpMergeSort::Sort on keys-only
 */
struct warp_sort_keys_t
{
  template <typename T, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void
  operator()(WarpSortT& warp_sort, T (&thread_data)[ITEMS_PER_THREAD], int /*valid_items*/, T /*oob_default*/) const
  {
    warp_sort.Sort(thread_data, CustomLess{});
  }
};

/**
 * @brief Delegate wrapper for partial WarpMergeSort::StableSort keys-only
 */
struct warp_partial_sort_keys_t
{
  template <typename T, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void
  operator()(WarpSortT& warp_sort, T (&thread_data)[ITEMS_PER_THREAD], int valid_items, T oob_default) const
  {
    warp_sort.Sort(thread_data, CustomLess{}, valid_items, oob_default);
  }
};

/**
 * @brief Delegate wrapper for WarpMergeSort::StableSort on key-value pairs
 */
struct warp_stable_sort_pairs_t
{
  template <typename KeyT, typename ValueT, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void operator()(
    WarpSortT& warp_sort,
    KeyT (&keys)[ITEMS_PER_THREAD],
    ValueT (&values)[ITEMS_PER_THREAD],
    int /*valid_items*/,
    KeyT /*oob_default*/) const
  {
    warp_sort.StableSort(keys, values, CustomLess{});
  }
};

/**
 * @brief Delegate wrapper for partial WarpMergeSort::StableSort key-value pairs
 */
struct warp_partial_stable_sort_pairs_t
{
  template <typename KeyT, typename ValueT, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void operator()(
    WarpSortT& warp_sort,
    KeyT (&keys)[ITEMS_PER_THREAD],
    ValueT (&values)[ITEMS_PER_THREAD],
    int valid_items,
    KeyT oob_default) const
  {
    warp_sort.StableSort(keys, values, CustomLess{}, valid_items, oob_default);
  }
};

/**
 * @brief Delegate wrapper for WarpMergeSort::Sort on key-value pairs
 */
struct warp_sort_pairs_t
{
  template <typename KeyT, typename ValueT, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void operator()(
    WarpSortT& warp_sort,
    KeyT (&keys)[ITEMS_PER_THREAD],
    ValueT (&values)[ITEMS_PER_THREAD],
    int /*valid_items*/,
    KeyT /*oob_default*/) const
  {
    warp_sort.Sort(keys, values, CustomLess{});
  }
};

/**
 * @brief Delegate wrapper for partial WarpMergeSort::StableSort key-value pairs
 */
struct warp_partial_sort_pairs_t
{
  template <typename KeyT, typename ValueT, int ITEMS_PER_THREAD, typename WarpSortT>
  __device__ void operator()(
    WarpSortT& warp_sort,
    KeyT (&keys)[ITEMS_PER_THREAD],
    ValueT (&values)[ITEMS_PER_THREAD],
    int valid_items,
    KeyT oob_default) const
  {
    warp_sort.Sort(keys, values, CustomLess{}, valid_items, oob_default);
  }
};

/**
 * @brief Dispatch helper function for sorting keys
 */
template <int ITEMS_PER_THREAD,
          int LOGICAL_WARP_THREADS,
          int TOTAL_WARPS,
          typename T,
          typename SegmentSizesItT,
          typename ActionT>
void warp_merge_sort(
  c2h::device_vector<T>& in, c2h::device_vector<T>& out, SegmentSizesItT segment_sizes, T oob_default, ActionT action)
{
  warp_merge_sort_kernel<ITEMS_PER_THREAD, LOGICAL_WARP_THREADS, TOTAL_WARPS>
    <<<1, LOGICAL_WARP_THREADS * TOTAL_WARPS>>>(
      thrust::raw_pointer_cast(in.data()), thrust::raw_pointer_cast(out.data()), segment_sizes, oob_default, action);

  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());
}

/**
 * @brief Dispatch helper function for sorting key-value pairs
 */
template <int ITEMS_PER_THREAD,
          int LOGICAL_WARP_THREADS,
          int TOTAL_WARPS,
          typename KeyT,
          typename ValueT,
          typename SegmentSizesItT,
          typename ActionT>
void warp_merge_sort(
  c2h::device_vector<KeyT>& keys_in,
  c2h::device_vector<KeyT>& keys_out,
  c2h::device_vector<ValueT>& values_in,
  c2h::device_vector<ValueT>& values_out,
  SegmentSizesItT segment_sizes,
  KeyT oob_default,
  ActionT action)
{
  warp_merge_sort_kernel<ITEMS_PER_THREAD, LOGICAL_WARP_THREADS, TOTAL_WARPS>
    <<<1, LOGICAL_WARP_THREADS * TOTAL_WARPS>>>(
      thrust::raw_pointer_cast(keys_in.data()),
      thrust::raw_pointer_cast(keys_out.data()),
      thrust::raw_pointer_cast(values_in.data()),
      thrust::raw_pointer_cast(values_out.data()),
      segment_sizes,
      oob_default,
      action);

  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());
}

/**
 * @brief Performs a stable sort on per-warp segments of data and assigns oob_default to items that
 * are out-of-bounds.
 */
template <typename RandomItT, typename SegmentSizeItT, typename T>
void compute_host_reference(
  RandomItT h_data, SegmentSizeItT segment_sizes, unsigned int num_segments, T oob_default, int logical_warp_items)
{
  for (unsigned int segment_id = 0; segment_id < num_segments; segment_id++)
  {
    unsigned int segment_size = segment_sizes[segment_id];
    std::stable_sort(h_data, h_data + segment_size);
    std::fill(h_data + segment_size, h_data + logical_warp_items, oob_default);
    h_data += logical_warp_items;
  }
}

/**
 * @brief Stability requirement of the sorting algorithm
 */
enum class stability
{
  stable,
  unstable
};

// List of key types to test
using custom_t  = c2h::custom_type_t<c2h::equal_comparable_t, c2h::lexicographical_less_comparable_t>;
using key_types = c2h::type_list<std::uint8_t, std::int32_t, std::int64_t, custom_t>;

// List of value types
using value_types = c2h::type_list<std::int32_t, custom_t>;

// Logical warp sizes to test
using logical_warp_threads = c2h::enum_type_list<int, 32, 4>;

// Number of items per thread to test
using items_per_thread_list = c2h::enum_type_list<int, 1, 4, 7>;

// Whether the sort is required to be stable or not
using stability_list = c2h::enum_type_list<stability, stability::stable, stability::unstable>;

template <typename TestType>
struct params_t
{
  using type = typename c2h::get<0, TestType>;

  static constexpr int logical_warp_threads = c2h::get<1, TestType>::value;
  static constexpr int items_per_thread     = c2h::get<2, TestType>::value;
  static constexpr int logical_warp_items   = logical_warp_threads * items_per_thread;
  static constexpr int total_warps          = 2;
  static constexpr int tile_size            = items_per_thread * total_warps * logical_warp_threads;
  static constexpr bool is_stable           = c2h::get<3, TestType>::value == stability::stable;
};

C2H_TEST(
  "Warp sort on keys-only works", "[sort][warp]", key_types, logical_warp_threads, items_per_thread_list, stability_list)
{
  using params             = params_t<TestType>;
  using type               = typename params::type;
  using warp_sort_delegate = ::cuda::std::_If<params::is_stable, warp_stable_sort_keys_t, warp_sort_keys_t>;

  // Prepare test data
  c2h::device_vector<type> d_in(params::tile_size);
  c2h::device_vector<type> d_out(params::tile_size);
  auto segment_sizes     = thrust::make_constant_iterator(params::logical_warp_items);
  const auto oob_default = std::numeric_limits<type>::max();
  c2h::gen(C2H_SEED(10), d_in);

  // Run test
  warp_merge_sort<params::items_per_thread, params::logical_warp_threads, params::total_warps>(
    d_in, d_out, segment_sizes, oob_default, warp_sort_delegate{});

  // Prepare verification data
  c2h::host_vector<type> h_in_out = d_in;
  compute_host_reference(h_in_out.begin(), segment_sizes, params::total_warps, oob_default, params::logical_warp_items);

  // Verify results
  REQUIRE(h_in_out == d_out);
}

C2H_TEST("Warp sort keys-only on partial warp-tile works",
         "[sort][warp]",
         key_types,
         logical_warp_threads,
         items_per_thread_list,
         stability_list)
{
  using params = params_t<TestType>;
  using type   = typename params::type;
  using warp_sort_delegate =
    ::cuda::std::_If<params::is_stable, warp_partial_stable_sort_keys_t, warp_partial_sort_keys_t>;

  // Prepare test data
  c2h::device_vector<type> d_in(params::tile_size);
  c2h::device_vector<type> d_out(params::tile_size);
  c2h::device_vector<int> d_segment_sizes(params::total_warps);
  const auto oob_default = std::numeric_limits<type>::max();
  c2h::gen(C2H_SEED(5), d_in);
  c2h::gen(C2H_SEED(5), d_segment_sizes, 0, params::logical_warp_items);

  // Run test
  warp_merge_sort<params::items_per_thread, params::logical_warp_threads, params::total_warps>(
    d_in, d_out, d_segment_sizes.cbegin(), oob_default, warp_sort_delegate{});

  // Prepare verification data
  c2h::host_vector<type> h_in_out     = d_in;
  c2h::host_vector<int> segment_sizes = d_segment_sizes;
  compute_host_reference(h_in_out.begin(), segment_sizes, params::total_warps, oob_default, params::logical_warp_items);

  // Verify results
  REQUIRE(h_in_out == d_out);
}

C2H_TEST("Warp sort on keys-value pairs works",
         "[sort][warp]",
         key_types,
         logical_warp_threads,
         items_per_thread_list,
         stability_list,
         value_types)
{
  using params             = params_t<TestType>;
  using key_type           = typename params::type;
  using value_type         = typename c2h::get<4, TestType>;
  using warp_sort_delegate = ::cuda::std::_If<params::is_stable, warp_stable_sort_pairs_t, warp_sort_pairs_t>;

  // Prepare test data
  c2h::device_vector<key_type> d_keys_in(params::tile_size);
  c2h::device_vector<key_type> d_keys_out(params::tile_size);
  c2h::device_vector<value_type> d_values_in(params::tile_size);
  c2h::device_vector<value_type> d_values_out(params::tile_size);
  auto segment_sizes     = thrust::make_constant_iterator(params::logical_warp_items);
  const auto oob_default = std::numeric_limits<key_type>::max();
  c2h::gen(C2H_SEED(10), d_keys_in);

  // Run test
  warp_merge_sort<params::items_per_thread, params::logical_warp_threads, params::total_warps>(
    d_keys_in, d_keys_out, d_values_in, d_values_out, segment_sizes, oob_default, warp_stable_sort_pairs_t{});

  // Prepare verification data
  c2h::host_vector<key_type> h_keys_in_out     = d_keys_in;
  c2h::host_vector<value_type> h_values_in_out = d_values_in;
  auto cpu_kv_pairs = thrust::make_zip_iterator(h_keys_in_out.begin(), h_values_in_out.begin());
  compute_host_reference(
    cpu_kv_pairs,
    segment_sizes,
    params::total_warps,
    thrust::make_tuple(oob_default, value_type{}),
    params::logical_warp_items);

  // Verify results
  REQUIRE(h_keys_in_out == d_keys_out);
  REQUIRE(h_values_in_out == d_values_out);
}

C2H_TEST("Warp sort on key-value pairs of a partial warp-tile works",
         "[sort][warp]",
         key_types,
         logical_warp_threads,
         items_per_thread_list,
         stability_list,
         value_types)
{
  using params     = params_t<TestType>;
  using key_type   = typename params::type;
  using value_type = typename c2h::get<4, TestType>;
  using warp_sort_delegate =
    ::cuda::std::_If<params::is_stable, warp_partial_stable_sort_pairs_t, warp_partial_sort_pairs_t>;

  // Prepare test data
  c2h::device_vector<key_type> d_keys_in(params::tile_size);
  c2h::device_vector<key_type> d_keys_out(params::tile_size);
  c2h::device_vector<value_type> d_values_in(params::tile_size);
  c2h::device_vector<value_type> d_values_out(params::tile_size);
  c2h::device_vector<int> d_segment_sizes(params::total_warps);
  const auto oob_default = std::numeric_limits<key_type>::max();
  c2h::gen(C2H_SEED(5), d_keys_in);
  c2h::gen(C2H_SEED(5), d_segment_sizes, 0, params::logical_warp_items);

  // Run test
  warp_merge_sort<params::items_per_thread, params::logical_warp_threads, params::total_warps>(
    d_keys_in, d_keys_out, d_values_in, d_values_out, d_segment_sizes.cbegin(), oob_default, warp_sort_delegate{});

  // Prepare verification data
  c2h::host_vector<key_type> h_keys_in_out     = d_keys_in;
  c2h::host_vector<value_type> h_values_in_out = d_values_in;
  c2h::host_vector<int> segment_sizes          = d_segment_sizes;
  auto cpu_kv_pairs = thrust::make_zip_iterator(h_keys_in_out.begin(), h_values_in_out.begin());
  compute_host_reference(
    cpu_kv_pairs,
    segment_sizes,
    params::total_warps,
    thrust::make_tuple(oob_default, value_type{}),
    params::logical_warp_items);

  // Verify results
  REQUIRE(h_keys_in_out == d_keys_out);
  REQUIRE(h_values_in_out == d_values_out);
}
