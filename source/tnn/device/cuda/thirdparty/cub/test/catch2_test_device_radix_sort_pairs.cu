#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_radix_sort.cuh>
#include <cub/util_type.cuh>

#include <thrust/memory.h>

#include <cuda/std/type_traits>

#include <algorithm>
#include <cstdint>
#include <limits>
#include <new> // bad_alloc

#include "catch2_large_array_sort_helper.cuh"
#include "catch2_radix_sort_helper.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceRadixSort::SortPairs, sort_pairs);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceRadixSort::SortPairsDescending, sort_pairs_descending);

using custom_value_t = c2h::custom_type_t<c2h::equal_comparable_t>;
using value_types    = c2h::type_list<cuda::std::uint8_t, cuda::std::uint64_t, custom_value_t>;

// hipcub::detail::ChooseOffsetsT only selected 32/64 bit unsigned types:
using num_items_types = c2h::type_list<cuda::std::uint32_t, cuda::std::uint64_t>;

C2H_TEST("DeviceRadixSort::SortPairs: Basic testing", "[pairs][radix][sort][device]", value_types, num_items_types)
{
  using key_t       = cuda::std::uint32_t;
  using value_t     = c2h::get<0, TestType>;
  using num_items_t = c2h::get<1, TestType>;

  constexpr num_items_t min_num_items = 1 << 5;
  constexpr num_items_t max_num_items = 1 << 20;
  const num_items_t num_items =
    GENERATE_COPY(num_items_t{0}, num_items_t{1}, take(5, random(min_num_items, max_num_items)));

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);

  c2h::device_vector<value_t> in_values(num_items);
  c2h::device_vector<value_t> out_values(num_items);

  const int num_key_seeds   = 1;
  const int num_value_seeds = 1;
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);
  c2h::gen(C2H_SEED(num_value_seeds), in_values);

  const bool is_descending = GENERATE(false, true);

  if (is_descending)
  {
    sort_pairs_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      thrust::raw_pointer_cast(out_keys.data()),
      thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(out_values.data()),
      num_items,
      begin_bit<key_t>(),
      end_bit<key_t>());
  }
  else
  {
    sort_pairs(thrust::raw_pointer_cast(in_keys.data()),
               thrust::raw_pointer_cast(out_keys.data()),
               thrust::raw_pointer_cast(in_values.data()),
               thrust::raw_pointer_cast(out_values.data()),
               num_items,
               begin_bit<key_t>(),
               end_bit<key_t>());
  }

  auto refs        = radix_sort_reference(in_keys, in_values, is_descending);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  REQUIRE(ref_keys == out_keys);
  REQUIRE(ref_values == out_values);
}

C2H_TEST("DeviceRadixSort::SortPairs: DoubleBuffer API", "[pairs][radix][sort][device]", value_types)
{
  using key_t   = cuda::std::uint32_t;
  using value_t = c2h::get<0, TestType>;

  constexpr std::size_t max_num_items = 1 << 18;
  const std::size_t num_items         = GENERATE_COPY(take(1, random(max_num_items / 2, max_num_items)));

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);

  c2h::device_vector<value_t> in_values(num_items);
  c2h::device_vector<value_t> out_values(num_items);

  const int num_key_seeds   = 1;
  const int num_value_seeds = 1;
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);
  c2h::gen(C2H_SEED(num_value_seeds), in_values);

  const bool is_descending = GENERATE(false, true);

  hipcub::DoubleBuffer<key_t> key_buffer(
    thrust::raw_pointer_cast(in_keys.data()), thrust::raw_pointer_cast(out_keys.data()));
  hipcub::DoubleBuffer<value_t> value_buffer(
    thrust::raw_pointer_cast(in_values.data()), thrust::raw_pointer_cast(out_values.data()));

  double_buffer_sort_t action(is_descending);
  action.initialize();
  launch(action, key_buffer, value_buffer, num_items, begin_bit<key_t>(), end_bit<key_t>());

  key_buffer.selector   = action.selector();
  value_buffer.selector = action.selector();
  action.finalize();

  auto refs        = radix_sort_reference(in_keys, in_values, is_descending);
  auto& ref_keys   = refs.first;
  auto& ref_values = refs.second;

  auto& keys   = key_buffer.selector == 0 ? in_keys : out_keys;
  auto& values = value_buffer.selector == 0 ? in_values : out_values;

  REQUIRE(ref_keys == keys);
  REQUIRE(ref_values == values);
}

template <typename key_t, typename value_t, typename num_items_t>
void do_large_offset_test(std::size_t num_items)
{
  const bool is_descending = GENERATE(false, true);

  CAPTURE(num_items, is_descending);

  try
  {
    large_array_sort_helper<key_t, value_t> arrays;
    arrays.initialize_for_stable_pair_sort(C2H_SEED(1), num_items, is_descending);

    TIME(c2h::cpu_timer timer);

    double_buffer_sort_t action(is_descending);
    action.initialize();
    const num_items_t typed_num_items = static_cast<num_items_t>(num_items);
    launch(action, arrays.keys_buffer, arrays.values_buffer, typed_num_items, begin_bit<key_t>(), end_bit<key_t>());

    TIME(timer.print_elapsed_seconds_and_reset("Device sort"));

    arrays.keys_buffer.selector   = action.selector();
    arrays.values_buffer.selector = action.selector();
    action.finalize();

    auto& keys   = arrays.keys_buffer.selector == 0 ? arrays.keys_in : arrays.keys_out;
    auto& values = arrays.values_buffer.selector == 0 ? arrays.values_in : arrays.values_out;

    arrays.verify_stable_pair_sort(num_items, is_descending, keys, values);
  }
  catch (std::bad_alloc& e)
  {
    (void) e;
#ifdef DEBUG_CHECKED_ALLOC_FAILURE
    const std::size_t num_bytes = num_items * (sizeof(key_t) + sizeof(value_t));
    std::cerr
      << "Skipping radix sort test with " << num_items << " elements (" << num_bytes << " bytes): " << e.what() << "\n";
#endif // DEBUG_CHECKED_ALLOC_FAILURE
  }
}

C2H_TEST("DeviceRadixSort::SortPairs: 32-bit overflow check", "[large][pairs][radix][sort][device]")
{
  using key_t       = std::uint8_t;
  using value_t     = std::uint8_t;
  using num_items_t = std::uint32_t;

  // Test problem size at the maximum offset value to ensure that internal calculations do not overflow.
  const std::size_t num_items = std::numeric_limits<num_items_t>::max();

  do_large_offset_test<key_t, value_t, num_items_t>(num_items);
}

C2H_TEST("DeviceRadixSort::SortPairs: Large Offsets", "[large][pairs][radix][sort][device]")
{
  using key_t       = std::uint8_t;
  using value_t     = std::uint8_t;
  using num_items_t = std::uint64_t;

  constexpr std::size_t min_num_items = std::size_t{1} << 32;
  constexpr std::size_t max_num_items = min_num_items + (std::size_t{1} << 30);
  const std::size_t num_items         = GENERATE_COPY(take(1, random(min_num_items, max_num_items)));

  do_large_offset_test<key_t, value_t, num_items_t>(num_items);
}
