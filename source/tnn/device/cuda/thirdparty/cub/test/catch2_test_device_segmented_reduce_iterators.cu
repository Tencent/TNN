/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_segmented_reduce.cuh>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <cstdint>

#include "catch2_test_device_reduce.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::Reduce, device_segmented_reduce);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::Sum, device_segmented_sum);

// %PARAM% TEST_LAUNCH lid 0:1

// List of types to test
using custom_t           = c2h::custom_type_t<c2h::accumulateable_t, c2h::equal_comparable_t>;
using iterator_type_list = c2h::type_list<type_pair<custom_t>, type_pair<std::int64_t>>;
using offsets            = c2h::type_list<std::int32_t, std::uint32_t>;

C2H_TEST("Device segmented reduce works with fancy input iterators", "[reduce][device]", iterator_type_list, offsets)
{
  using type_pair_t = typename c2h::get<0, TestType>;
  using item_t      = typename type_pair_t::input_t;
  using output_t    = typename type_pair_t::output_t;
  using offset_t    = typename c2h::get<1, TestType>;

  constexpr int min_items = 1;
  constexpr int max_items = 1000000;

  // Number of items
  const int num_items = GENERATE_COPY(
    take(2, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));
  INFO("Test num_items: " << num_items);

  // Range of segment sizes to generate
  const std::tuple<offset_t, offset_t> seg_size_range =
    GENERATE_COPY(table<offset_t, offset_t>({{1, 1}, {1, num_items}, {num_items, num_items}}));
  INFO("Test seg_size_range: [" << std::get<0>(seg_size_range) << ", " << std::get<1>(seg_size_range) << "]");

  // Generate input segments
  c2h::device_vector<offset_t> segment_offsets = c2h::gen_uniform_offsets<offset_t>(
    C2H_SEED(1), num_items, std::get<0>(seg_size_range), std::get<1>(seg_size_range));
  const offset_t num_segments = static_cast<offset_t>(segment_offsets.size() - 1);
  auto d_offsets_it           = thrust::raw_pointer_cast(segment_offsets.data());

  // Prepare input data
  item_t default_constant{};
  init_default_constant(default_constant);
  auto in_it = thrust::make_constant_iterator(default_constant);

  using op_t   = hipcub::Sum;
  using init_t = output_t;

  // Binary reduction operator
  auto reduction_op = op_t{};

  // Prepare verification data
  using accum_t = ::cuda::std::__accumulator_t<op_t, item_t, init_t>;
  c2h::host_vector<output_t> expected_result(num_segments);
  compute_segmented_problem_reference(in_it, segment_offsets, reduction_op, accum_t{}, expected_result.begin());

  // Run test
  c2h::device_vector<output_t> out_result(num_segments);
  auto d_out_it = thrust::raw_pointer_cast(out_result.data());
  device_segmented_reduce(in_it, d_out_it, num_segments, d_offsets_it, d_offsets_it + 1, reduction_op, init_t{});

  // Verify result
  REQUIRE(expected_result == out_result);
}
