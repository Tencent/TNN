#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_for.cuh>
#include <cub/device/device_transform.cuh>

#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/zip_function.h>

#include <sstream>

#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>
#include <c2h/test_util_vec.cuh>

// %PARAM% TEST_LAUNCH lid 0:1:2

using hipcub::detail::transform::Algorithm;

template <Algorithm Alg>
struct policy_hub_for_alg
{
  struct max_policy : hipcub::ChainedPolicy<300, max_policy, max_policy>
  {
    static constexpr int min_bif         = 64 * 1024;
    static constexpr Algorithm algorithm = Alg;
    using algo_policy =
      ::cuda::std::_If<Alg == Algorithm::prefetch,
                       hipcub::detail::transform::prefetch_policy_t<256>,
                       hipcub::detail::transform::async_copy_policy_t<256>>;
  };
};

template <Algorithm Alg,
          typename Offset,
          typename... RandomAccessIteratorsIn,
          typename RandomAccessIteratorOut,
          typename TransformOp>
HIPCUB_RUNTIME_FUNCTION static hipError_t transform_many_with_alg_entry_point(
  void* d_temp_storage,
  size_t& temp_storage_bytes,
  ::cuda::std::tuple<RandomAccessIteratorsIn...> inputs,
  RandomAccessIteratorOut output,
  Offset num_items,
  TransformOp transform_op,
  hipStream_t stream = nullptr)
{
  if (d_temp_storage == nullptr)
  {
    temp_storage_bytes = 1;
    return hipSuccess;
  }

  constexpr bool RequiresStableAddress = false;
  return hipcub::detail::transform::dispatch_t<RequiresStableAddress,
                                            Offset,
                                            ::cuda::std::tuple<RandomAccessIteratorsIn...>,
                                            RandomAccessIteratorOut,
                                            TransformOp,
                                            policy_hub_for_alg<Alg>>{}
    .dispatch(inputs, output, num_items, transform_op, stream);
}

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceTransform::Transform, transform_many);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceTransform::TransformStableArgumentAddresses, transform_many_stable);
DECLARE_TMPL_LAUNCH_WRAPPER(transform_many_with_alg_entry_point,
                            transform_many_with_alg,
                            ESCAPE_LIST(Algorithm Alg, typename Offset),
                            ESCAPE_LIST(Alg, Offset));

using algorithms =
  c2h::enum_type_list<Algorithm,
                      Algorithm::prefetch
#ifdef _CUB_HAS_TRANSFORM_UBLKCP
                      ,
                      Algorithm::ublkcp
#endif // _CUB_HAS_TRANSFORM_UBLKCP
                      >;

using offset_types = c2h::type_list<std::int32_t, std::int64_t>;

#ifdef _CUB_HAS_TRANSFORM_UBLKCP
#  define FILTER_UBLKCP                                \
    if (alg == Algorithm::ublkcp && ptx_version < 900) \
    {                                                  \
      return;                                          \
    }
#else // _CUB_HAS_TRANSFORM_UBLKCP
#  define FILTER_UBLKCP
#endif // _CUB_HAS_TRANSFORM_UBLKCP

#define FILTER_UNSUPPORTED_ALGS                                           \
  int ptx_version = 0;                                                    \
  REQUIRE(hipcub::PtxVersion(ptx_version) == hipSuccess);                   \
  _CCCL_DIAG_PUSH                                                         \
  _CCCL_DIAG_SUPPRESS_MSVC(4127) /* conditional expression is constant */ \
  FILTER_UBLKCP                                                           \
  _CCCL_DIAG_POP

C2H_TEST("DeviceTransform::Transform BabelStream add",
         "[device][device_transform]",
         c2h::type_list<std::uint8_t, std::uint16_t, std::uint32_t, std::uint64_t /*, int3, double3*/>,
         offset_types,
         algorithms)
{
  using type         = typename c2h::get<0, TestType>;
  using offset_t     = typename c2h::get<1, TestType>;
  constexpr auto alg = c2h::get<2, TestType>::value;
  FILTER_UNSUPPORTED_ALGS
  const int num_items = GENERATE(0, 1, 15, 16, 17, 127, 128, 129, 4095, 4096, 4097); // edge cases around 16 and 128
  CAPTURE(c2h::demangle(typeid(type).name()), c2h::demangle(typeid(offset_t).name()), alg, num_items);

  c2h::device_vector<type> a(num_items);
  c2h::device_vector<type> b(num_items);
  c2h::gen(C2H_SEED(1), a);
  c2h::gen(C2H_SEED(1), b);

  c2h::device_vector<type> result(num_items);
  transform_many_with_alg<alg, offset_t>(
    ::cuda::std::make_tuple(a.begin(), b.begin()), result.begin(), num_items, ::cuda::std::plus<type>{});

  // compute reference and verify
  c2h::host_vector<type> a_h = a;
  c2h::host_vector<type> b_h = b;
  c2h::host_vector<type> reference_h(num_items);
  std::transform(a_h.begin(), a_h.end(), b_h.begin(), reference_h.begin(), std::plus<type>{});
  REQUIRE(reference_h == result);
}

template <int Alignment>
struct alignas(Alignment) overaligned_addable_t
{
  int value;

  overaligned_addable_t() = default;

  _CCCL_HOST_DEVICE overaligned_addable_t(int val)
      : value{val}
  {}

  _CCCL_HOST_DEVICE static void check(const overaligned_addable_t& obj)
  {
    if (reinterpret_cast<uintptr_t>(&obj) % Alignment != 0)
    {
      printf("Error: object not aligned to %d: %p\n", Alignment, &obj);
      ::cuda::std::terminate();
    }
  }

  _CCCL_HOST_DEVICE friend auto operator==(const overaligned_addable_t& a, const overaligned_addable_t& b) -> bool
  {
    check(a);
    check(b);
    return a.value == b.value;
  }

  _CCCL_HOST_DEVICE friend auto
  operator+(const overaligned_addable_t& a, const overaligned_addable_t& b) -> overaligned_addable_t
  {
    check(a);
    check(b);
    return overaligned_addable_t{a.value + b.value};
  }

  _CCCL_HOST friend auto operator<<(std::ostream& os, const overaligned_addable_t& obj) -> std::ostream&
  {
    check(obj);
    return os << "over{" << obj.value << "}";
  }
};

using overaligned_types =
  c2h::type_list<overaligned_addable_t<32>
#ifndef _CCCL_COMPILER_MSVC // error C2719: [...] formal parameter with requested alignment of 256 won't be aligned
                 ,
                 overaligned_addable_t<256>
#endif // _CCCL_COMPILER_MSVC
                 >;

// test with types exceeding the memcpy_async and bulk copy alignments (16 and 128 bytes respectively)
C2H_TEST("DeviceTransform::Transform overaligned type", "[device][device_transform]", overaligned_types)
{
  using type = c2h::get<0, TestType>;
  CAPTURE(c2h::demangle(typeid(type).name()));

  const int num_items = GENERATE(0, 1, 100, 1000);
  c2h::device_vector<int> a(num_items, 3); // put some integers at the front, so SMEM has to handle different alignments
  c2h::device_vector<type> b(num_items, 4);

  c2h::device_vector<type> result(num_items);
  // we need raw pointers here to halfen the conversion sequence from device_reference<int> -> int -> type when calling
  // plus(...), which is too long to compile
  transform_many(::cuda::std::make_tuple(thrust::raw_pointer_cast(a.data()), thrust::raw_pointer_cast(b.data())),
                 result.begin(),
                 num_items,
                 ::cuda::std::plus<type>{});

  REQUIRE(result == c2h::device_vector<type>(num_items, 7));
}

C2H_TEST("DeviceTransform::Transform huge type", "[device][device_transform]")
{
  using huge_t = c2h::custom_type_t<c2h::equal_comparable_t, c2h::accumulateable_t, c2h::huge_data<666>::type>;
  static_assert(alignof(huge_t) == 8, "Need a large type with alignment < 16");
  CAPTURE(c2h::demangle(typeid(huge_t).name()));

  const int num_items = GENERATE(0, 1, 100, 1000);
  c2h::device_vector<huge_t> a(num_items);
  c2h::device_vector<huge_t> b(num_items);
  c2h::gen(C2H_SEED(1), a);
  c2h::gen(C2H_SEED(1), b);

  c2h::device_vector<huge_t> result(num_items);
  transform_many(::cuda::std::make_tuple(a.begin(), b.begin()), result.begin(), num_items, ::cuda::std::plus<huge_t>{});

  c2h::host_vector<huge_t> a_h = a;
  c2h::host_vector<huge_t> b_h = b;
  c2h::host_vector<huge_t> reference_h(num_items);
  std::transform(a_h.begin(), a_h.end(), b_h.begin(), reference_h.begin(), std::plus<huge_t>{});
  REQUIRE(result == reference_h);
}

struct times_seven
{
  _CCCL_HOST_DEVICE auto operator()(unsigned char v) const -> char
  {
    return static_cast<unsigned char>(v * 7);
  }
};

C2H_TEST("DeviceTransform::Transform with large input", "[device][device_transform]", algorithms)
try
{
  using type         = unsigned char;
  using offset_t     = cuda::std::int64_t;
  constexpr auto alg = c2h::get<0, TestType>::value;
  FILTER_UNSUPPORTED_ALGS
  CAPTURE(alg);

  constexpr offset_t num_items = (offset_t{1} << 32) + 123456; // a few thread blocks beyond 4GiB
  c2h::device_vector<type> input(num_items);
  c2h::gen(C2H_SEED(1), input);

  c2h::device_vector<type> result(num_items);
  transform_many_with_alg<alg, offset_t>(
    ::cuda::std::make_tuple(input.begin()), result.begin(), num_items, times_seven{});

  // compute reference and verify
  c2h::host_vector<type> input_h = input;
  c2h::host_vector<type> reference_h(num_items);
  std::transform(input_h.begin(), input_h.end(), reference_h.begin(), times_seven{});
  REQUIRE((reference_h == result));
}
catch (const std::bad_alloc&)
{
  // allocation failure is not a test failure, so we can run tests on smaller GPUs
}

template <typename T>
struct nstream_kernel
{
  static constexpr T scalar = 42;

  _CCCL_HOST_DEVICE T operator()(const T& ai, const T& bi, const T& ci) const
  {
    return ai + bi + scalar * ci;
  }
};

// overwrites one input stream
C2H_TEST("DeviceTransform::Transform BabelStream nstream",
         "[device][device_transform]",
         c2h::type_list<std::uint8_t, std::uint16_t, std::uint32_t, std::uint64_t>,
         offset_types,
         algorithms)
{
  using type         = typename c2h::get<0, TestType>;
  using offset_t     = typename c2h::get<1, TestType>;
  constexpr auto alg = c2h::get<2, TestType>::value;
  FILTER_UNSUPPORTED_ALGS
  CAPTURE(c2h::demangle(typeid(type).name()), c2h::demangle(typeid(offset_t).name()), alg);

  const int num_items = GENERATE(0, 1, 100, 1000, 10000);
  c2h::device_vector<type> a(num_items);
  c2h::device_vector<type> b(num_items);
  c2h::device_vector<type> c(num_items);
  c2h::gen(C2H_SEED(1), a, type{10}, type{100});
  c2h::gen(C2H_SEED(1), b, type{10}, type{100});
  c2h::gen(C2H_SEED(1), c, type{10}, type{100});

  // copy to host before changing
  c2h::host_vector<type> a_h = a;
  c2h::host_vector<type> b_h = b;
  c2h::host_vector<type> c_h = c;

  transform_many_with_alg<alg, offset_t>(
    ::cuda::std::make_tuple(a.begin(), b.begin(), c.begin()), a.begin(), num_items, nstream_kernel<type>{});

  // compute reference and verify
  auto z = thrust::make_zip_iterator(a_h.begin(), b_h.begin(), c_h.begin());
  std::transform(z, z + num_items, a_h.begin(), thrust::make_zip_function(nstream_kernel<type>{}));
  REQUIRE(a_h == a);
}

struct sum_five
{
  __device__ auto operator()(std::int8_t a, std::int16_t b, std::int32_t c, std::int64_t d, float e) const -> double
  {
    return a + b + c + d + e;
  }
};

C2H_TEST("DeviceTransform::Transform add five streams", "[device][device_transform]", algorithms)
{
  using offset_t     = int;
  constexpr auto alg = c2h::get<0, TestType>::value;
  FILTER_UNSUPPORTED_ALGS

  constexpr int num_items = 100;
  c2h::device_vector<std::int8_t> a(num_items, 1);
  c2h::device_vector<std::int16_t> b(num_items, 2);
  c2h::device_vector<std::int32_t> c(num_items, 3);
  c2h::device_vector<std::int64_t> d(num_items, 4);
  c2h::device_vector<float> e(num_items, 5);

  c2h::device_vector<double> result(num_items);
  transform_many_with_alg<alg, offset_t>(
    ::cuda::std::make_tuple(a.begin(), b.begin(), c.begin(), d.begin(), e.begin()),
    result.begin(),
    num_items,
    sum_five{});

  // compute reference and verify
  c2h::device_vector<double> reference(num_items, 1 + 2 + 3 + 4 + 5);
  REQUIRE(reference == result);
}

struct give_me_five
{
  __device__ auto operator()() const -> int
  {
    return 5;
  }
};

C2H_TEST("DeviceTransform::Transform no streams", "[device][device_transform]")
{
  constexpr int num_items = 100;
  c2h::device_vector<int> result(num_items);
  transform_many(::cuda::std::tuple<>{}, result.begin(), num_items, give_me_five{});

  // compute reference and verify
  c2h::device_vector<int> reference(num_items, 5);
  REQUIRE(reference == result);
}

C2H_TEST("DeviceTransform::Transform fancy input iterator types", "[device][device_transform]")
{
  using type = int;

  constexpr int num_items = 100;
  thrust::counting_iterator<type> a{0};
  thrust::counting_iterator<type> b{10};

  c2h::device_vector<type> result(num_items);
  transform_many(::cuda::std::make_tuple(a, b), result.begin(), num_items, ::cuda::std::plus<type>{});

  // compute reference and verify
  c2h::host_vector<type> reference_h(num_items);
  std::transform(a, a + num_items, b, reference_h.begin(), std::plus<type>{});
  REQUIRE(reference_h == result);
}

C2H_TEST("DeviceTransform::Transform fancy output iterator type", "[device][device_transform]", algorithms)
{
  using type         = int;
  using offset_t     = int;
  constexpr auto alg = c2h::get<0, TestType>::value;
  FILTER_UNSUPPORTED_ALGS

  constexpr int num_items = 100;
  c2h::device_vector<type> a(num_items, 13);
  c2h::device_vector<type> b(num_items, 35);
  c2h::device_vector<type> result(num_items);

  using thrust::placeholders::_1;
  auto out = thrust::make_transform_output_iterator(result.begin(), _1 + 4);
  transform_many_with_alg<alg, offset_t>(
    ::cuda::std::make_tuple(a.begin(), b.begin()), out, num_items, ::cuda::std::plus<type>{});
  REQUIRE(result == c2h::device_vector<type>(num_items, (13 + 35) + 4));
}

C2H_TEST("DeviceTransform::Transform mixed input iterator types", "[device][device_transform]")
{
  using type = int;

  constexpr int num_items = 100;
  thrust::counting_iterator<type> a{0};
  c2h::device_vector<type> b(num_items, 10);

  c2h::device_vector<type> result(num_items);
  transform_many(::cuda::std::make_tuple(a, b.begin()), result.begin(), num_items, ::cuda::std::plus<type>{});

  // compute reference and verify
  c2h::host_vector<type> b_h = b;
  c2h::host_vector<type> reference_h(num_items);
  std::transform(a, a + num_items, b_h.begin(), reference_h.begin(), std::plus<type>{});
  REQUIRE(reference_h == result);
}

struct plus_needs_stable_address
{
  int* a;
  int* b;

  _CCCL_HOST_DEVICE int operator()(const int& v) const
  {
    const auto i = &v - a;
    return v + b[i];
  }
};

C2H_TEST("DeviceTransform::Transform address stability", "[device][device_transform]")
{
  using type = int;

  constexpr int num_items = 100;
  c2h::device_vector<type> a(num_items);
  c2h::device_vector<type> b(num_items);
  thrust::sequence(a.begin(), a.end());
  thrust::sequence(b.begin(), b.end(), 42);

  c2h::device_vector<type> result(num_items);
  transform_many_stable(
    ::cuda::std::make_tuple(thrust::raw_pointer_cast(a.data())),
    result.begin(),
    num_items,
    plus_needs_stable_address{thrust::raw_pointer_cast(a.data()), thrust::raw_pointer_cast(b.data())});

  // compute reference and verify
  c2h::device_vector<type> a_h = a;
  c2h::device_vector<type> b_h = b;
  c2h::host_vector<type> reference_h(num_items);
  std::transform(a_h.begin(), a_h.end(), b_h.begin(), reference_h.begin(), std::plus<type>{});
  REQUIRE(reference_h == result);
}

// Non-trivially-copyable/relocatable type which cannot be copied using std::memcpy or hipMemcpy
struct non_trivial
{
  int data;

  non_trivial() = default;

  _CCCL_HOST_DEVICE explicit non_trivial(int data)
      : data(data)
  {}

  _CCCL_HOST_DEVICE non_trivial(const non_trivial& nt)
      : data(nt.data)
  {}

  _CCCL_HOST_DEVICE auto operator=(const non_trivial& nt) -> non_trivial&
  {
    data = nt.data;
    return *this;
  }

  _CCCL_HOST_DEVICE auto operator-() const -> non_trivial
  {
    return non_trivial{-data};
  }

  friend _CCCL_HOST_DEVICE auto operator==(non_trivial a, non_trivial b) -> bool
  {
    return a.data == b.data;
  }
};
static_assert(!::cuda::std::is_trivially_copyable<non_trivial>::value, ""); // as required by the standard
static_assert(!thrust::is_trivially_relocatable<non_trivial>::value, ""); // CUB uses this check internally

// Note(bgruber): I gave up on writing a test that checks whether the copy ctor/assignment operator is actually called
// (e.g. by tracking/counting invocations of those), since C++ allows (but not guarantees) elision of these operations.
// Also thrust algorithms perform a lot of copies in-between, so the test needs to use only raw allocations and
// iteration for setup and checking.
C2H_TEST("DeviceTransform::Transform not trivially relocatable", "[device][device_transform]")
{
  constexpr int num_items = 100;
  c2h::device_vector<non_trivial> input(num_items, non_trivial{42});
  c2h::device_vector<non_trivial> result(num_items);
  transform_many(
    ::cuda::std::make_tuple(thrust::raw_pointer_cast(input.data())), result.begin(), num_items, ::cuda::std::negate<>{});

  const auto reference = c2h::device_vector<non_trivial>(num_items, non_trivial{-42});
  REQUIRE((reference == result));
}

C2H_TEST("DeviceTransform::Transform buffer start alignment",
         "[device][device_transform]",
         c2h::type_list<std::uint8_t, std::uint16_t, float, double>)
{
  using type = typename c2h::get<0, TestType>;

  constexpr int num_items = 1000;
  const int offset        = GENERATE(1, 2, 4, 8, 16, 32, 64, 128); // global memory is always at least 256 byte aligned
  CAPTURE(c2h::demangle(typeid(type).name()), offset);
  c2h::device_vector<type> input(num_items);
  thrust::sequence(input.begin(), input.end());
  c2h::device_vector<type> result(num_items);
  using thrust::placeholders::_1;
  transform_many(::cuda::std::make_tuple(input.begin() + offset),
                 result.begin() + offset,
                 num_items - offset,
                 _1 * 10); // FIXME(bgruber): does not work on negative

  c2h::device_vector<type> reference(num_items);
  thrust::tabulate(reference.begin() + offset, reference.end(), (_1 + offset) * 10);
  REQUIRE(reference == result);
}

namespace Catch
{
template <typename T>
struct StringMaker<hipcub::detail::transform::aligned_base_ptr<T>>
{
  static auto convert(hipcub::detail::transform::aligned_base_ptr<T> abp) -> std::string
  {
    std::stringstream ss;
    ss << "{ptr: " << abp.ptr << ", head_padding: " << abp.head_padding << "}";
    return ss.str();
  }
};
} // namespace Catch

// TODO(bgruber): rewrite this example using int3
C2H_TEST("DeviceTransform::Transform aligned_base_ptr", "[device][device_transform]")
{
  alignas(128) int arr[256];
  using namespace hipcub::detail::transform;
  CHECK(make_aligned_base_ptr(&arr[0], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[0]), 0});
  CHECK(make_aligned_base_ptr(&arr[1], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[0]), 4});
  CHECK(make_aligned_base_ptr(&arr[5], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[0]), 20});
  CHECK(make_aligned_base_ptr(&arr[31], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[0]), 124});
  CHECK(make_aligned_base_ptr(&arr[32], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[32]), 0});
  CHECK(make_aligned_base_ptr(&arr[33], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[32]), 4});
  CHECK(make_aligned_base_ptr(&arr[127], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[96]), 124});
  CHECK(make_aligned_base_ptr(&arr[128], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[128]), 0});
  CHECK(make_aligned_base_ptr(&arr[129], 128) == aligned_base_ptr<int>{reinterpret_cast<char*>(&arr[128]), 4});
}
