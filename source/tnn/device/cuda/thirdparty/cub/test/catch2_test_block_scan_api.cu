#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/block/block_scan.cuh>

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cuda/std/numeric>

#include <c2h/catch2_test_helper.cuh>

constexpr int num_items_per_thread = 2;
constexpr int block_num_threads    = 64;

// example-begin inclusive-scan-array-init-value
__global__ void InclusiveBlockScanKernel(int* output)
{
  // Specialize BlockScan for a 1D block of 64 threads of type int
  using block_scan_t   = hipcub::BlockScan<int, 64>;
  using temp_storage_t = block_scan_t::TempStorage;

  // Allocate shared memory for BlockScan
  __shared__ temp_storage_t temp_storage;

  int initial_value = 1;
  int thread_data[] = {
    +1 * ((int) threadIdx.x * num_items_per_thread), // item 0
    -1 * ((int) threadIdx.x * num_items_per_thread + 1) // item 1
  };
  //  input: {[0, -1], [2, -3],[4, -5], ... [126, -127]}

  // Collectively compute the block-wide inclusive scan max
  block_scan_t(temp_storage).InclusiveScan(thread_data, thread_data, initial_value, hipcub::Max());

  // output: {[1, 1], [2, 2],[3, 3], ... [126, 126]}
  // ...
  // example-end inclusive-scan-array-init-value
  output[threadIdx.x * 2]     = thread_data[0];
  output[threadIdx.x * 2 + 1] = thread_data[1];
}

C2H_TEST("Block array-based inclusive scan works with initial value", "[scan][block]")
{
  thrust::device_vector<int> d_out(block_num_threads * num_items_per_thread);

  InclusiveBlockScanKernel<<<1, block_num_threads>>>(thrust::raw_pointer_cast(d_out.data()));
  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());

  c2h::host_vector<int> expected(d_out.size());
  for (size_t i = 0; i < expected.size() - 1; i += 2)
  {
    expected[i]     = static_cast<int>(i);
    expected[i + 1] = static_cast<int>(i);
  }

  // When initial value = 1 for the given input the first two
  // elements of the result are equal to 1.
  expected[0] = 1;
  expected[1] = 1;

  REQUIRE(expected == d_out);
}

// example-begin inclusive-scan-array-aggregate-init-value
__global__ void InclusiveBlockScanKernelAggregate(int* output, int* d_block_aggregate)
{
  // Specialize BlockScan for a 1D block of 64 threads of type int
  using block_scan_t   = hipcub::BlockScan<int, 64>;
  using temp_storage_t = block_scan_t::TempStorage;

  // Allocate shared memory for BlockScan
  __shared__ temp_storage_t temp_storage;

  int initial_value = 1;
  int thread_data[] = {
    +1 * ((int) threadIdx.x * num_items_per_thread), // item 0
    -1 * ((int) threadIdx.x * num_items_per_thread + 1) // item 1
  };
  //  input: {[0, -1], [2, -3],[4, -5], ... [126, -127]}

  // Collectively compute the block-wide inclusive scan max
  int block_aggregate;
  block_scan_t(temp_storage).InclusiveScan(thread_data, thread_data, initial_value, hipcub::Max(), block_aggregate);

  // output: {[1, 1], [2, 2],[3, 3], ... [126, 126]}
  // block_aggregate = 126;
  // ...
  // example-end inclusive-scan-array-aggregate-init-value

  *d_block_aggregate          = block_aggregate;
  output[threadIdx.x * 2]     = thread_data[0];
  output[threadIdx.x * 2 + 1] = thread_data[1];
}

C2H_TEST("Block array-based inclusive scan with block aggregate works with initial value", "[scan][block]")
{
  thrust::device_vector<int> d_out(block_num_threads * num_items_per_thread);

  c2h::device_vector<int> d_block_aggregate(1);
  InclusiveBlockScanKernelAggregate<<<1, block_num_threads>>>(
    thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_block_aggregate.data()));
  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());

  c2h::host_vector<int> expected(d_out.size());
  for (size_t i = 0; i < expected.size() - 1; i += 2)
  {
    expected[i]     = static_cast<int>(i);
    expected[i + 1] = static_cast<int>(i);
  }

  // When initial value = 1 for the given input the first two
  // elements of the result are equal to 1.
  expected[0] = 1;
  expected[1] = 1;

  REQUIRE(d_out == expected);
  REQUIRE(d_block_aggregate[0] == 126);
}
