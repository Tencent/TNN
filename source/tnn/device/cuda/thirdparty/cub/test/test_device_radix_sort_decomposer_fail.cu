/******************************************************************************
 * Copyright (c) 2011-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_radix_sort.cuh>

struct custom_t
{
  std::uint16_t i;
  float f;
};

struct decomposer_t
{
  // expected-error {{"DecomposerT must be a callable object returning a tuple of references"}}
  __host__ __device__ std::uint16_t& operator()(custom_t& key) const
  {
    return key.i;
  }
};

int main()
{
  custom_t* d_in{};
  custom_t* d_out{};
  std::size_t temp_storage_bytes{};
  std::uint8_t* d_temp_storage{};

  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_in, d_out, 0, decomposer_t{});
}
