/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_scan.cuh>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <c2h/catch2_test_helper.cuh>

C2H_TEST("Device inclusive scan works", "[scan][device]")
{
  // example-begin device-inclusive-scan
  thrust::device_vector<int> input{0, -1, 2, -3, 4, -5, 6};
  thrust::device_vector<int> out(input.size());

  int init = 1;
  size_t temp_storage_bytes{};

  hipcub::DeviceScan::InclusiveScanInit(
    nullptr, temp_storage_bytes, input.begin(), out.begin(), hipcub::Max{}, init, static_cast<int>(input.size()));

  // Allocate temporary storage for inclusive scan
  thrust::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);

  // Run inclusive prefix sum
  hipcub::DeviceScan::InclusiveScanInit(
    thrust::raw_pointer_cast(temp_storage.data()),
    temp_storage_bytes,
    input.begin(),
    out.begin(),
    hipcub::Max{},
    init,
    static_cast<int>(input.size()));

  thrust::host_vector<int> expected{1, 1, 2, 2, 4, 4, 6};
  // example-end device-inclusive-scan

  REQUIRE(expected == out);
}
