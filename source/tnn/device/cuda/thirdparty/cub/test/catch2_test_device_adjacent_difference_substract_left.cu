#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_adjacent_difference.cuh>

#include <thrust/iterator/discard_iterator.h>

#include <algorithm>
#include <numeric>

#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceAdjacentDifference::SubtractLeft, adjacent_difference_subtract_left);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceAdjacentDifference::SubtractLeftCopy, adjacent_difference_subtract_left_copy);

// %PARAM% TEST_LAUNCH lid 0:1:2

using all_types =
  c2h::type_list<std::uint8_t,
                 std::uint64_t,
                 std::int8_t,
                 std::int64_t,
                 ulonglong2,
                 c2h::custom_type_t<c2h::equal_comparable_t, c2h::subtractable_t>>;

using types = c2h::type_list<std::uint8_t, std::int32_t>;

C2H_TEST("DeviceAdjacentDifference::SubtractLeft can run with empty input", "[device][adjacent_difference]", types)
{
  using type = typename c2h::get<0, TestType>;

  constexpr int num_items = 0;
  c2h::device_vector<type> in(num_items);

  adjacent_difference_subtract_left(in.begin(), num_items, hipcub::Difference{});
}

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy can run with empty input", "[device][adjacent_difference]", types)
{
  using type = typename c2h::get<0, TestType>;

  constexpr int num_items = 0;
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);

  adjacent_difference_subtract_left_copy(in.begin(), out.begin(), num_items, hipcub::Difference{});
}

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy does not change the input", "[device][adjacent_difference]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<type> reference = in;
  adjacent_difference_subtract_left_copy(in.begin(), thrust::discard_iterator<>(), num_items, hipcub::Difference{});

  REQUIRE(reference == in);
}

C2H_TEST("DeviceAdjacentDifference::SubtractLeft works with iterators", "[device][adjacent_difference]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::host_vector<type> h_in = in;
  c2h::host_vector<type> reference(num_items);
  std::adjacent_difference(h_in.begin(), h_in.end(), reference.begin(), std::minus<type>{});

  adjacent_difference_subtract_left(in.begin(), num_items, hipcub::Difference{});

  REQUIRE(reference == in);
}

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy works with iterators", "[device][adjacent_difference]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::host_vector<type> h_in = in;
  c2h::host_vector<type> reference(num_items);
  std::adjacent_difference(h_in.begin(), h_in.end(), reference.begin(), std::minus<type>{});

  adjacent_difference_subtract_left_copy(in.begin(), out.begin(), num_items, hipcub::Difference{});

  REQUIRE(reference == out);
}

C2H_TEST("DeviceAdjacentDifference::SubtractLeft works with pointers", "[device][adjacent_difference]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::host_vector<type> h_in = in;
  c2h::host_vector<type> reference(num_items);
  std::adjacent_difference(h_in.begin(), h_in.end(), reference.begin(), std::minus<type>{});

  adjacent_difference_subtract_left(thrust::raw_pointer_cast(in.data()), num_items, hipcub::Difference{});

  REQUIRE(reference == in);
}

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy works with pointers", "[device][adjacent_difference]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::host_vector<type> h_in = in;
  c2h::host_vector<type> reference(num_items);
  std::adjacent_difference(h_in.begin(), h_in.end(), reference.begin(), std::minus<type>{});

  adjacent_difference_subtract_left_copy(
    thrust::raw_pointer_cast(in.data()), thrust::raw_pointer_cast(out.data()), num_items, hipcub::Difference{});

  REQUIRE(reference == out);
}

template <class T>
struct cust_diff
{
  template <class T2, cuda::std::__enable_if_t<cuda::std::is_same<T, T2>::value, int> = 0>
  __host__ __device__ constexpr T2 operator()(const T2& lhs, const T2& rhs) const noexcept
  {
    return lhs - rhs;
  }

  __host__ __device__ constexpr ulonglong2 operator()(const ulonglong2& lhs, const ulonglong2& rhs) const noexcept
  {
    return ulonglong2{lhs.x - rhs.x, lhs.y - rhs.y};
  }
};

C2H_TEST("DeviceAdjacentDifference::SubtractLeft works with custom difference",
         "[device][adjacent_difference]",
         all_types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::host_vector<type> h_in = in;
  c2h::host_vector<type> reference(num_items);
  std::adjacent_difference(h_in.begin(), h_in.end(), reference.begin(), cust_diff<type>{});

  adjacent_difference_subtract_left(in.begin(), num_items, cust_diff<type>{});

  REQUIRE(reference == in);
}

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy works with custom difference",
         "[device][adjacent_difference]",
         all_types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::host_vector<type> h_in = in;
  c2h::host_vector<type> reference(num_items);
  std::adjacent_difference(h_in.begin(), h_in.end(), reference.begin(), cust_diff<type>{});

  adjacent_difference_subtract_left_copy(in.begin(), out.begin(), num_items, cust_diff<type>{});

  REQUIRE(reference == out);
}

template <class T>
struct convertible_from_T
{
  T val_;

  convertible_from_T() = default;
  __host__ __device__ convertible_from_T(const T& val) noexcept
      : val_(val)
  {}
  __host__ __device__ convertible_from_T& operator=(const T& val) noexcept
  {
    val_ = val;
  }
  // Converting back to T helps satisfy all the machinery that T supports
  __host__ __device__ operator T() const noexcept
  {
    return val_;
  }
};

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy works with a different output type",
         "[device][adjacent_difference]",
         types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<convertible_from_T<type>> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::host_vector<type> h_in = in;
  c2h::host_vector<type> reference(num_items);
  std::adjacent_difference(h_in.begin(), h_in.end(), reference.begin(), cust_diff<type>{});

  adjacent_difference_subtract_left_copy(in.begin(), out.begin(), num_items, cust_diff<type>{});

  REQUIRE(reference == out);
}

struct check_difference
{
  int* d_error;

  template <class T>
  __device__ T operator()(const T& lhs, const T& rhs) const noexcept
  {
    const T result = lhs - rhs;
    if (result != 1)
    {
      atomicAdd(d_error, 1);
    }
    return result;
  }
};

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy works with large indexes", "[device][adjacent_difference]")
{
  constexpr cuda::std::size_t num_items = 1ll << 33;
  c2h::device_vector<int> error(1);
  int* d_error = thrust::raw_pointer_cast(error.data());
  adjacent_difference_subtract_left_copy(
    thrust::counting_iterator<cuda::std::size_t>{0}, thrust::discard_iterator<>{}, num_items, check_difference{d_error});
  const int h_error = error[0];
  REQUIRE(h_error == 0);
}

struct invocation_counter
{
  __host__ explicit invocation_counter(unsigned long long* addr)
      : counts_(addr)
  {}

  template <class T>
  __device__ T operator()(const T& lhs, const T& rhs) const noexcept
  {
    // Use legacy atomics to support testing on older archs:
    atomicAdd(counts_, 1ull);
    return lhs - rhs;
  }

private:
  unsigned long long* counts_;
};

C2H_TEST("DeviceAdjacentDifference::SubtractLeftCopy uses right number of invocations", "[device][adjacent_difference]")
{
  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<unsigned long long> counts(1, 0);
  adjacent_difference_subtract_left_copy(
    thrust::counting_iterator<cuda::std::size_t>{0},
    thrust::discard_iterator<>(),
    num_items,
    invocation_counter{thrust::raw_pointer_cast(counts.data())});

  REQUIRE(counts.front() == static_cast<unsigned long long>(num_items - 1));
}
