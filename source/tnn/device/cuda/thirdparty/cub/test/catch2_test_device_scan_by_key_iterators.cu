#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_scan.cuh>

#include <cstdint>

#include "catch2_test_device_reduce.cuh"
#include "catch2_test_device_scan.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>
#include <c2h/extended_types.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::ExclusiveSumByKey, device_exclusive_sum_by_key);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::ExclusiveScanByKey, device_exclusive_scan_by_key);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::InclusiveSumByKey, device_inclusive_sum_by_key);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceScan::InclusiveScanByKey, device_inclusive_scan_by_key);

// %PARAM% TEST_LAUNCH lid 0:1
// %PARAM% TEST_TYPES types 0:1:2:3

// List of types to test
using custom_t =
  c2h::custom_type_t<c2h::accumulateable_t,
                     c2h::equal_comparable_t,
                     c2h::lexicographical_less_comparable_t,
                     c2h::lexicographical_greater_comparable_t>;

// type_quad's parameters and defaults:
// type_quad<value_in_t, value_out_t=value_in_t, key_t=int32_t, equality_op_t=hipcub::Equality>
#if TEST_TYPES == 0
using full_type_list = c2h::type_list<type_quad<std::uint8_t, std::int32_t, float>,
                                      type_quad<std::int8_t, std::int8_t, std::int32_t, Mod2Equality>>;
#elif TEST_TYPES == 1
using full_type_list = c2h::type_list<type_quad<std::int32_t>, type_quad<std::uint64_t>>;
#elif TEST_TYPES == 2
using full_type_list =
  c2h::type_list<type_quad<uchar3, uchar3, custom_t>, type_quad<ulonglong4, ulonglong4, std::uint8_t, Mod2Equality>>;
#elif TEST_TYPES == 3
using full_type_list = c2h::type_list<type_quad<custom_t, custom_t, custom_t>>;
#endif

/**
 * @brief Input data generation mode
 */
enum class gen_data_t : int
{
  /// Uniform random data generation
  GEN_TYPE_RANDOM,
  /// Constant value as input data
  GEN_TYPE_CONST
};

C2H_TEST("Device scan works with fancy iterators", "[by_key][scan][device]", full_type_list)
{
  using params   = params_t<TestType>;
  using key_t    = typename params::type_pair_t::key_t;
  using value_t  = typename params::item_t;
  using output_t = typename params::output_t;
  using offset_t = std::uint32_t;
  using eq_op_t  = typename params::type_pair_t::eq_op_t;

  constexpr offset_t min_items = 1;
  constexpr offset_t max_items = 1000000;

  // Generate the input sizes to test for
  const offset_t num_items = GENERATE_COPY(
    take(2, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));
  INFO("Test num_items: " << num_items);

  // Range of segment sizes to generate (a segment is a series of consecutive equal keys)
  const std::tuple<offset_t, offset_t> seg_size_range =
    GENERATE_COPY(table<offset_t, offset_t>({{1, 1}, {1, num_items}, {num_items, num_items}}));
  INFO("Test seg_size_range: [" << std::get<0>(seg_size_range) << ", " << std::get<1>(seg_size_range) << "]");

  // Generate input segments
  c2h::device_vector<offset_t> segment_offsets = c2h::gen_uniform_offsets<offset_t>(
    C2H_SEED(1), num_items, std::get<0>(seg_size_range), std::get<1>(seg_size_range));

  // Get array of keys from segment offsets
  c2h::device_vector<key_t> segment_keys(num_items);
  c2h::init_key_segments(segment_offsets, segment_keys);
  auto d_keys_it = segment_keys.begin();
  c2h::host_vector<key_t> h_segment_keys(segment_keys);

  // Prepare input data
  value_t default_constant{};
  init_default_constant(default_constant);
  auto values_in_it = thrust::make_constant_iterator(default_constant);

  SECTION("inclusive sum")
  {
    using op_t = hipcub::Sum;

    // Prepare verification data
    c2h::host_vector<output_t> expected_result(num_items);
    compute_inclusive_scan_by_key_reference(
      values_in_it, h_segment_keys.cbegin(), expected_result.begin(), op_t{}, eq_op_t{}, num_items);

    // Run test
    c2h::device_vector<output_t> out_values(num_items);
    device_inclusive_sum_by_key(d_keys_it, values_in_it, out_values.begin(), num_items, eq_op_t{});

    // Verify result
    REQUIRE(expected_result == out_values);
  }

  SECTION("exclusive sum")
  {
    using op_t = hipcub::Sum;

    // Prepare verification data
    c2h::host_vector<output_t> expected_result(num_items);
    compute_exclusive_scan_by_key_reference(
      values_in_it, h_segment_keys.cbegin(), expected_result.begin(), op_t{}, eq_op_t{}, output_t{}, num_items);

    // Run test
    c2h::device_vector<output_t> out_values(num_items);
    device_exclusive_sum_by_key(d_keys_it, values_in_it, out_values.begin(), num_items, eq_op_t{});

    // Verify result
    REQUIRE(expected_result == out_values);
  }

  SECTION("inclusive scan")
  {
    using op_t = hipcub::Min;

    // Prepare verification data
    c2h::host_vector<output_t> expected_result(num_items);
    compute_inclusive_scan_by_key_reference(
      values_in_it, h_segment_keys.cbegin(), expected_result.begin(), op_t{}, eq_op_t{}, num_items);

    // Run test
    c2h::device_vector<output_t> out_values(num_items);
    device_inclusive_scan_by_key(d_keys_it, values_in_it, out_values.begin(), op_t{}, num_items, eq_op_t{});

    // Verify result
    REQUIRE(expected_result == out_values);
  }

  SECTION("exclusive scan")
  {
    using op_t = hipcub::Sum;

    // Scan operator
    auto scan_op = op_t{};

    // Prepare verification data
    c2h::host_vector<output_t> expected_result(num_items);
    compute_exclusive_scan_by_key_reference(
      values_in_it, h_segment_keys.cbegin(), expected_result.begin(), scan_op, eq_op_t{}, output_t{}, num_items);

    // Run test
    c2h::device_vector<output_t> out_values(num_items);
    using init_t = value_t;
    device_exclusive_scan_by_key(d_keys_it, values_in_it, out_values.begin(), scan_op, init_t{}, num_items, eq_op_t{});

    // Verify result
    REQUIRE(expected_result == out_values);
  }
}
