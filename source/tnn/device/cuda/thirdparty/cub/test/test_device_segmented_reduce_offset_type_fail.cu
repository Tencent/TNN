/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// %PARAM% TEST_ERR err 0:1:2:3:4:5

#include <cub/device/device_segmented_reduce.cuh>

int main()
{
  using offset_t = float; // error
  // using offset_t = int; // ok
  float *d_in{}, *d_out{};
  offset_t* d_offsets{};
  std::size_t temp_storage_bytes{};
  std::uint8_t* d_temp_storage{};

#if TEST_ERR == 0
  // expected-error {{"Offset iterator type should be integral."}}
  hipcub::DeviceSegmentedReduce::Reduce(
    d_temp_storage, temp_storage_bytes, d_in, d_out, 0, d_offsets, d_offsets + 1, hipcub::Min(), 0);

#elif TEST_ERR == 1
  // expected-error {{"Offset iterator type should be integral."}}
  hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, 0, d_offsets, d_offsets + 1);

#elif TEST_ERR == 2
  // expected-error {{"Offset iterator type should be integral."}}
  hipcub::DeviceSegmentedReduce::Min(d_temp_storage, temp_storage_bytes, d_in, d_out, 0, d_offsets, d_offsets + 1);

#elif TEST_ERR == 3
  // expected-error {{"Offset iterator type should be integral."}}
  hipcub::DeviceSegmentedReduce::ArgMin(d_temp_storage, temp_storage_bytes, d_in, d_out, 0, d_offsets, d_offsets + 1);

#elif TEST_ERR == 4
  // expected-error {{"Offset iterator type should be integral."}}
  hipcub::DeviceSegmentedReduce::Max(d_temp_storage, temp_storage_bytes, d_in, d_out, 0, d_offsets, d_offsets + 1);

#elif TEST_ERR == 5
  // expected-error {{"Offset iterator type should be integral."}}
  hipcub::DeviceSegmentedReduce::ArgMax(d_temp_storage, temp_storage_bytes, d_in, d_out, 0, d_offsets, d_offsets + 1);
#endif
}
