#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/
#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first
#include "catch2_radix_sort_helper.cuh"
#include <c2h/catch2_test_helper.cuh>
#include <catch2_segmented_sort_helper.cuh>

// FIXME: Graph launch disabled, algorithm syncs internally. WAR exists for device-launch, figure out how to enable for
// graph launch.

// TODO replace with DeviceSegmentedSort::SortKeys interface once https://github.com/NVIDIA/cccl/issues/50 is addressed
// Temporary wrapper that allows specializing the DeviceSegmentedSort algorithm for different offset types
template <bool IS_DESCENDING, typename KeyT, typename BeginOffsetIteratorT, typename EndOffsetIteratorT, typename NumItemsT>
HIPCUB_RUNTIME_FUNCTION _CCCL_FORCEINLINE static hipError_t dispatch_segmented_sort_wrapper(
  void* d_temp_storage,
  size_t& temp_storage_bytes,
  const KeyT* d_keys_in,
  KeyT* d_keys_out,
  NumItemsT num_items,
  NumItemsT num_segments,
  BeginOffsetIteratorT d_begin_offsets,
  EndOffsetIteratorT d_end_offsets,
  bool* selector,
  bool is_overwrite   = false,
  hipStream_t stream = 0)
{
  hipcub::DoubleBuffer<KeyT> d_keys(const_cast<KeyT*>(d_keys_in), d_keys_out);
  hipcub::DoubleBuffer<hipcub::NullType> d_values;
  auto status =
    hipcub::DispatchSegmentedSort<IS_DESCENDING, KeyT, hipcub::NullType, NumItemsT, BeginOffsetIteratorT, EndOffsetIteratorT>::
      Dispatch(
        d_temp_storage,
        temp_storage_bytes,
        d_keys,
        d_values,
        num_items,
        num_segments,
        d_begin_offsets,
        d_end_offsets,
        is_overwrite,
        stream);
  if (status != hipSuccess)
  {
    return status;
  }
  if (is_overwrite)
  {
    // Only write to selector in the DoubleBuffer invocation
    *selector = d_keys.Current() != d_keys_out;
  }
  return hipSuccess;
}

// %PARAM% TEST_LAUNCH lid 0:1

DECLARE_LAUNCH_WRAPPER(dispatch_segmented_sort_wrapper<true>, dispatch_segmented_sort_descending);
DECLARE_LAUNCH_WRAPPER(dispatch_segmented_sort_wrapper<false>, dispatch_segmented_sort);

using key_types =
  c2h::type_list<bool,
                 std::uint8_t,
                 std::uint64_t
#if TEST_HALF_T
                 ,
                 half_t
#endif
#if TEST_BF_T
                 ,
                 bfloat16_t
#endif
                 >;

C2H_TEST("DeviceSegmentedSortKeys: No segments", "[keys][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;

  const bool stable_sort     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<hipcub::NullType> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    stable_sort,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    int{},
    int{},
    nullptr,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

C2H_TEST("DeviceSegmentedSortKeys: Empty segments", "[keys][segmented][sort][device]")
{
  // Type doesn't affect the escape logic, so it should be fine
  // to test only one set of types here.

  using KeyT = std::uint8_t;

  const int num_segments     = GENERATE(take(2, random(1 << 2, 1 << 22)));
  const bool sort_stable     = GENERATE(unstable, stable);
  const bool sort_descending = GENERATE(ascending, descending);
  const bool sort_buffer     = GENERATE(pointers, double_buffer);

  c2h::device_vector<int> offsets(num_segments + 1, int{});
  const int* d_offsets = thrust::raw_pointer_cast(offsets.data());

  hipcub::DoubleBuffer<KeyT> keys_buffer(nullptr, nullptr);
  hipcub::DoubleBuffer<hipcub::NullType> values_buffer(nullptr, nullptr);
  values_buffer.selector = 1;

  call_cub_segmented_sort_api(
    sort_descending,
    sort_buffer,
    sort_stable,
    static_cast<KeyT*>(nullptr),
    static_cast<KeyT*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    static_cast<hipcub::NullType*>(nullptr),
    int{},
    num_segments,
    d_offsets,
    &keys_buffer.selector,
    &values_buffer.selector);

  REQUIRE(keys_buffer.selector == 0);
  REQUIRE(values_buffer.selector == 1);
}

C2H_TEST("DeviceSegmentedSortKeys: Same size segments, derived keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int segment_size = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)));

  const int segments = GENERATE_COPY(take(2, random(1 << 0, 1 << 5)), //
                                     take(2, random(1 << 5, 1 << 10)));

  test_same_size_segments_derived<KeyT>(segment_size, segments);
}

C2H_TEST("DeviceSegmentedSortKeys: Randomly sized segments, derived keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(
    take(2, random(1 << 0, 1 << 5)), //
    take(2, random(1 << 5, 1 << 10)),
    take(2, random(1 << 10, 1 << 15)),
    take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_derived<KeyT>(C2H_SEED(1), max_items, max_segment, segments);
}

C2H_TEST("DeviceSegmentedSortKeys: Randomly sized segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;

  const int max_items   = 1 << 22;
  const int max_segment = 6000;

  const int segments = GENERATE_COPY(take(2, random(1 << 15, 1 << 20)));

  test_random_size_segments_random<KeyT>(C2H_SEED(1), max_items, max_segment, segments);
}

C2H_TEST("DeviceSegmentedSortKeys: Edge case segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;
  test_edge_case_segments_random<KeyT>(C2H_SEED(4));
}

C2H_TEST("DeviceSegmentedSortKeys: Unspecified segments, random keys", "[keys][segmented][sort][device]", key_types)
{
  using KeyT = c2h::get<0, TestType>;
  test_unspecified_segments_random<KeyT>(C2H_SEED(4));
}

#if defined(CCCL_TEST_ENABLE_LARGE_SEGMENTED_SORT)

// we can reuse the same structure of DeviceSegmentedRadixSortKeys for simplicity
C2H_TEST("DeviceSegmentedSortKeys: very large num. items and num. segments",
         "[keys][segmented][sort][device]",
         all_offset_types)
try
{
  using key_t                      = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using offset_t                   = c2h::get<0, TestType>;
  constexpr std::size_t Step       = 500;
  using segment_iterator_t         = segment_iterator<offset_t, Step>;
  constexpr std::size_t uint32_max = ::cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr int num_key_seeds      = 1;
  const bool is_descending         = GENERATE(false, true);
  const bool is_overwrite          = GENERATE(false, true);
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : ::cuda::std::numeric_limits<offset_t>::max();
  const std::size_t num_segments = ::cuda::ceil_div(num_items, Step);
  CAPTURE(c2h::type_name<offset_t>(), num_items, num_segments, is_descending, is_overwrite);

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);
  auto offsets =
    thrust::make_transform_iterator(thrust::make_counting_iterator(std::size_t{0}), segment_iterator_t{num_items});
  auto offsets_plus_1 = offsets + 1;
  // Allocate host/device-accessible memory to communicate the selected output buffer
  bool* selector_ptr = nullptr;
  if (is_overwrite)
  {
    REQUIRE(hipHostMalloc(&selector_ptr, sizeof(*selector_ptr)) == hipSuccess);
  }

  auto ref_keys     = segmented_radix_sort_reference(in_keys, is_descending, num_segments, offsets, offsets_plus_1);
  auto out_keys_ptr = thrust::raw_pointer_cast(out_keys.data());
  if (is_descending)
  {
    dispatch_segmented_sort_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      out_keys_ptr,
      static_cast<offset_t>(num_items),
      static_cast<offset_t>(num_segments),
      offsets,
      offsets_plus_1,
      selector_ptr,
      is_overwrite);
  }
  else
  {
    dispatch_segmented_sort(
      thrust::raw_pointer_cast(in_keys.data()),
      out_keys_ptr,
      static_cast<offset_t>(num_items),
      static_cast<offset_t>(num_segments),
      offsets,
      offsets_plus_1,
      selector_ptr,
      is_overwrite);
  }
  if (is_overwrite)
  {
    if (*selector_ptr)
    {
      std::swap(out_keys, in_keys);
    }
    REQUIRE(hipHostFree(selector_ptr) == hipSuccess);
  }
  REQUIRE((ref_keys == out_keys) == true);
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, unsufficient GPU memory. " << e.what() << "\n";
}

C2H_TEST("DeviceSegmentedSort::SortKeys: very large segments", "[keys][segmented][sort][device]", all_offset_types)
try
{
  using key_t                      = cuda::std::uint8_t; // minimize memory footprint to support a wider range of GPUs
  using offset_t                   = c2h::get<0, TestType>;
  constexpr std::size_t uint32_max = ::cuda::std::numeric_limits<std::uint32_t>::max();
  constexpr int num_key_seeds      = 1;
  const bool is_descending         = GENERATE(false, true);
  const bool is_overwrite          = GENERATE(false, true);
  constexpr std::size_t num_items =
    (sizeof(offset_t) == 8) ? uint32_max + (1 << 20) : ::cuda::std::numeric_limits<offset_t>::max();
  const std::size_t num_segments = 2;
  CAPTURE(c2h::type_name<offset_t>(), num_items, is_descending, is_overwrite);

  c2h::device_vector<key_t> in_keys(num_items);
  c2h::device_vector<key_t> out_keys(num_items);
  c2h::gen(C2H_SEED(num_key_seeds), in_keys);
  c2h::device_vector<offset_t> offsets(num_segments + 1);
  offsets[0] = 0;
  offsets[1] = static_cast<offset_t>(num_items);
  offsets[2] = static_cast<offset_t>(num_items);

  // Allocate host/device-accessible memory to communicate the selected output buffer
  bool* selector_ptr = nullptr;
  if (is_overwrite)
  {
    REQUIRE(hipSuccess == hipHostMalloc(&selector_ptr, sizeof(*selector_ptr)));
  }
  auto ref_keys     = segmented_radix_sort_reference(in_keys, is_descending, offsets);
  auto out_keys_ptr = thrust::raw_pointer_cast(out_keys.data());
  if (is_descending)
  {
    dispatch_segmented_sort_descending(
      thrust::raw_pointer_cast(in_keys.data()),
      out_keys_ptr,
      static_cast<offset_t>(num_items),
      static_cast<offset_t>(num_segments),
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1,
      selector_ptr,
      is_overwrite);
  }
  else
  {
    dispatch_segmented_sort(
      thrust::raw_pointer_cast(in_keys.data()),
      out_keys_ptr,
      static_cast<offset_t>(num_items),
      static_cast<offset_t>(num_segments),
      thrust::raw_pointer_cast(offsets.data()),
      offsets.cbegin() + 1,
      selector_ptr,
      is_overwrite);
  }
  if (is_overwrite)
  {
    if (*selector_ptr)
    {
      std::swap(out_keys, in_keys);
    }
    REQUIRE(hipSuccess == hipHostFree(selector_ptr));
  }
  REQUIRE((ref_keys == out_keys) == true);
}
catch (std::bad_alloc& e)
{
  std::cerr << "Skipping segmented sort test, unsufficient GPU memory. " << e.what() << "\n";
}

#endif // defined(CCCL_TEST_ENABLE_LARGE_SEGMENTED_SORT)
