/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_select.cuh>

#include <thrust/device_vector.h>
#include <thrust/equal.h>
#include <thrust/memory.h>

#include <cstddef>

#include <c2h/catch2_test_helper.cuh>

// example-begin segmented-select-iseven
struct is_even_t
{
  __host__ __device__ bool operator()(int flag) const
  {
    return !(flag % 2);
  }
};
// example-end segmented-select-iseven

C2H_TEST("hipcub::DeviceSelect::FlaggedIf works with int data elements", "[select][device]")
{
  // example-begin segmented-select-flaggedif
  constexpr int num_items            = 8;
  thrust::device_vector<int> d_in    = {0, 1, 2, 3, 4, 5, 6, 7};
  thrust::device_vector<int> d_flags = {8, 6, 7, 5, 3, 0, 9, 3};
  thrust::device_vector<int> d_out(num_items);
  thrust::device_vector<int> d_num_selected_out(num_items);
  is_even_t is_even{};

  // Determine temporary device storage requirements
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSelect::FlaggedIf(
    nullptr,
    temp_storage_bytes,
    d_in.begin(),
    d_flags.begin(),
    d_out.begin(),
    d_num_selected_out.data(),
    num_items,
    is_even);

  // Allocate temporary storage
  c2h::device_vector<char> temp_storage(temp_storage_bytes);

  // Run selection
  hipcub::DeviceSelect::FlaggedIf(
    thrust::raw_pointer_cast(temp_storage.data()),
    temp_storage_bytes,
    d_in.begin(),
    d_flags.begin(),
    d_out.begin(),
    d_num_selected_out.data(),
    num_items,
    is_even);

  thrust::device_vector<int> expected{0, 1, 5};
  // example-end segmented-select-flaggedif

  REQUIRE(d_num_selected_out[0] == static_cast<int>(expected.size()));
  d_out.resize(d_num_selected_out[0]);
  REQUIRE(d_out == expected);
}

C2H_TEST("hipcub::DeviceSelect::FlaggedIf in-place works with int data elements", "[select][device]")
{
  // example-begin segmented-select-flaggedif-inplace
  constexpr int num_items            = 8;
  thrust::device_vector<int> d_data  = {0, 1, 2, 3, 4, 5, 6, 7};
  thrust::device_vector<int> d_flags = {8, 6, 7, 5, 3, 0, 9, 3};
  thrust::device_vector<int> d_num_selected_out(num_items);
  is_even_t is_even{};

  // Determine temporary device storage requirements
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSelect::FlaggedIf(
    nullptr, temp_storage_bytes, d_data.begin(), d_flags.begin(), d_num_selected_out.data(), num_items, is_even);

  // Allocate temporary storage
  c2h::device_vector<char> temp_storage(temp_storage_bytes);

  // Run selection
  hipcub::DeviceSelect::FlaggedIf(
    thrust::raw_pointer_cast(temp_storage.data()),
    temp_storage_bytes,
    d_data.begin(),
    d_flags.begin(),
    d_num_selected_out.data(),
    num_items,
    is_even);

  thrust::device_vector<int> expected{0, 1, 5};
  // example-end segmented-select-flaggedif-inplace

  REQUIRE(d_num_selected_out[0] == static_cast<int>(expected.size()));
  d_data.resize(d_num_selected_out[0]);
  REQUIRE(d_data == expected);
}
