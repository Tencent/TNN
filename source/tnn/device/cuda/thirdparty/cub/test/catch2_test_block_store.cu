#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/block/block_store.cuh>
#include <cub/iterator/cache_modified_output_iterator.cuh>
#include <cub/iterator/discard_output_iterator.cuh>
#include <cub/util_allocator.cuh>
#include <cub/util_arch.cuh>

#include <c2h/catch2_test_helper.cuh>

template <int ItemsPerThread, int ThreadsInBlock, hipcub::BlockStoreAlgorithm /* StoreAlgorithm */>
struct output_idx
{
  static __device__ int get(int item)
  {
    return static_cast<int>(threadIdx.x) * ItemsPerThread + item;
  }
};

template <int ItemsPerThread, int ThreadsInBlock>
struct output_idx<ItemsPerThread, ThreadsInBlock, hipcub::BlockStoreAlgorithm::BLOCK_STORE_STRIPED>
{
  static __device__ int get(int item)
  {
    return static_cast<int>(threadIdx.x) + ThreadsInBlock * item;
  }
};

template <typename InputIteratorT,
          typename OutputIteratorT,
          int ItemsPerThread,
          int ThreadsInBlock,
          hipcub::BlockStoreAlgorithm StoreAlgorithm>
__global__ void kernel(std::integral_constant<bool, true>, InputIteratorT input, OutputIteratorT output, int num_items)
{
  using input_t       = hipcub::detail::value_t<InputIteratorT>;
  using block_store_t = hipcub::BlockStore<input_t, ThreadsInBlock, ItemsPerThread, StoreAlgorithm>;
  using storage_t     = typename block_store_t::TempStorage;

  __shared__ storage_t storage;
  block_store_t block_store(storage);

  input_t data[ItemsPerThread];

  for (int i = 0; i < ItemsPerThread; i++)
  {
    const int idx = output_idx<ItemsPerThread, ThreadsInBlock, StoreAlgorithm>::get(i);

    if (idx < num_items)
    {
      data[i] = input[idx];
    }
  }

  if (ItemsPerThread * ThreadsInBlock == num_items)
  {
    block_store.Store(output, data);
  }
  else
  {
    block_store.Store(output, data, num_items);
  }
}

template <typename InputIteratorT,
          typename OutputIteratorT,
          int ItemsPerThread,
          int ThreadsInBlock,
          hipcub::BlockStoreAlgorithm /* StoreAlgorithm */>
__global__ void kernel(std::integral_constant<bool, false>, InputIteratorT input, OutputIteratorT output, int num_items)
{
  for (int i = 0; i < ItemsPerThread; i++)
  {
    const int idx = output_idx<ItemsPerThread, ThreadsInBlock, hipcub::BlockStoreAlgorithm::BLOCK_STORE_DIRECT>::get(i);

    if (idx < num_items)
    {
      output[idx] = input[idx];
    }
  }
}

template <int ItemsPerThread,
          int ThreadsInBlock,
          hipcub::BlockStoreAlgorithm StoreAlgorithm,
          typename InputIteratorT,
          typename OutputIteratorT>
void block_store(InputIteratorT input, OutputIteratorT output, int num_items)
{
  using input_t                       = hipcub::detail::value_t<InputIteratorT>;
  using block_store_t                 = hipcub::BlockStore<input_t, ThreadsInBlock, ItemsPerThread, StoreAlgorithm>;
  using storage_t                     = typename block_store_t::TempStorage;
  constexpr bool sufficient_resources = sizeof(storage_t) <= hipcub::detail::max_smem_per_block;

  kernel<InputIteratorT, OutputIteratorT, ItemsPerThread, ThreadsInBlock, StoreAlgorithm>
    <<<1, ThreadsInBlock>>>(std::integral_constant<bool, sufficient_resources>{}, input, output, num_items);

  REQUIRE(hipSuccess == hipPeekAtLastError());
  REQUIRE(hipSuccess == hipDeviceSynchronize());
}

// %PARAM% IPT it 1:11

using types     = c2h::type_list<std::uint8_t, std::int32_t, std::int64_t>;
using vec_types = c2h::type_list<long2, double2>;

using even_threads_in_block = c2h::enum_type_list<int, 32, 128>;
using odd_threads_in_block  = c2h::enum_type_list<int, 15, 65>;
using a_block_size          = c2h::enum_type_list<int, 256>;

using items_per_thread = c2h::enum_type_list<int, IPT>;
using store_algorithm =
  c2h::enum_type_list<hipcub::BlockStoreAlgorithm,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_DIRECT,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_STRIPED,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_VECTORIZE,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_TRANSPOSE,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_WARP_TRANSPOSE,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_WARP_TRANSPOSE_TIMESLICED>;

using odd_store_algorithm =
  c2h::enum_type_list<hipcub::BlockStoreAlgorithm,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_DIRECT,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_STRIPED,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_VECTORIZE,
                      hipcub::BlockStoreAlgorithm::BLOCK_STORE_TRANSPOSE>;

template <class TestType>
struct params_t
{
  using type = typename c2h::get<0, TestType>;

  static constexpr int items_per_thread                     = c2h::get<1, TestType>::value;
  static constexpr int threads_in_block                     = c2h::get<2, TestType>::value;
  static constexpr int tile_size                            = items_per_thread * threads_in_block;
  static constexpr hipcub::BlockStoreAlgorithm store_algorithm = c2h::get<3, TestType>::value;
};

C2H_TEST("Block store works with even block sizes",
         "[store][block]",
         types,
         items_per_thread,
         even_threads_in_block,
         store_algorithm)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  c2h::device_vector<type> d_input(GENERATE_COPY(take(10, random(0, params::tile_size))));
  c2h::gen(C2H_SEED(10), d_input);

  c2h::device_vector<type> d_output(d_input.size());

  block_store<params::items_per_thread, params::threads_in_block, params::store_algorithm>(
    thrust::raw_pointer_cast(d_input.data()),
    thrust::raw_pointer_cast(d_output.data()),
    static_cast<int>(d_input.size()));

  REQUIRE(d_input == d_output);
}

C2H_TEST("Block store works with even odd sizes",
         "[store][block]",
         types,
         items_per_thread,
         odd_threads_in_block,
         odd_store_algorithm)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  c2h::device_vector<type> d_input(GENERATE_COPY(take(10, random(0, params::tile_size))));
  c2h::gen(C2H_SEED(10), d_input);

  c2h::device_vector<type> d_output(d_input.size());

  block_store<params::items_per_thread, params::threads_in_block, params::store_algorithm>(
    thrust::raw_pointer_cast(d_input.data()),
    thrust::raw_pointer_cast(d_output.data()),
    static_cast<int>(d_input.size()));

  REQUIRE(d_input == d_output);
}

C2H_TEST("Block store works with even vector types",
         "[store][block]",
         vec_types,
         items_per_thread,
         a_block_size,
         store_algorithm)
{
  using params = params_t<TestType>;
  using type   = typename params::type;

  c2h::device_vector<type> d_input(GENERATE_COPY(take(10, random(0, params::tile_size))));
  c2h::gen(C2H_SEED(10), d_input);

  c2h::device_vector<type> d_output(d_input.size());

  block_store<params::items_per_thread, params::threads_in_block, params::store_algorithm>(
    thrust::raw_pointer_cast(d_input.data()),
    thrust::raw_pointer_cast(d_output.data()),
    static_cast<int>(d_input.size()));

  REQUIRE(d_input == d_output);
}

C2H_TEST("Block store works with custom types", "[store][block]", items_per_thread, store_algorithm)
{
  using type                                                = c2h::custom_type_t<c2h::equal_comparable_t>;
  constexpr int items_per_thread                            = c2h::get<0, TestType>::value;
  constexpr int threads_in_block                            = 64;
  constexpr int tile_size                                   = items_per_thread * threads_in_block;
  static constexpr hipcub::BlockStoreAlgorithm store_algorithm = c2h::get<1, TestType>::value;

  c2h::device_vector<type> d_input(GENERATE_COPY(take(10, random(0, tile_size))));
  c2h::gen(C2H_SEED(10), d_input);

  c2h::device_vector<type> d_output(d_input.size());

  block_store<items_per_thread, threads_in_block, store_algorithm>(
    thrust::raw_pointer_cast(d_input.data()),
    thrust::raw_pointer_cast(d_output.data()),
    static_cast<int>(d_input.size()));

  REQUIRE(d_input == d_output);
}

C2H_TEST("Block store works with caching iterators", "[store][block]", items_per_thread, store_algorithm)
{
  using type                                                = int;
  constexpr int items_per_thread                            = c2h::get<0, TestType>::value;
  constexpr int threads_in_block                            = 64;
  constexpr int tile_size                                   = items_per_thread * threads_in_block;
  static constexpr hipcub::BlockStoreAlgorithm store_algorithm = c2h::get<1, TestType>::value;

  c2h::device_vector<type> d_input(GENERATE_COPY(take(10, random(0, tile_size))));
  c2h::gen(C2H_SEED(10), d_input);

  c2h::device_vector<type> d_output(d_input.size());
  hipcub::CacheModifiedOutputIterator<hipcub::CacheStoreModifier::STORE_DEFAULT, type> out(
    thrust::raw_pointer_cast(d_output.data()));

  block_store<items_per_thread, threads_in_block, store_algorithm>(
    thrust::raw_pointer_cast(d_input.data()), out, static_cast<int>(d_input.size()));

  REQUIRE(d_input == d_output);
}
