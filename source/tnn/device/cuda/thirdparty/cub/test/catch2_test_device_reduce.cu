#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/
#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <thrust/iterator/constant_iterator.h>

#include <cstdint>

#include "catch2_test_device_reduce.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>
#include <c2h/extended_types.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::Reduce, device_reduce);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::Sum, device_sum);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::Min, device_min);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::ArgMin, device_arg_min);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::Max, device_max);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::ArgMax, device_arg_max);

// %PARAM% TEST_LAUNCH lid 0:1:2
// %PARAM% TEST_TYPES types 0:1:2:3:4

// List of types to test
using custom_t =
  c2h::custom_type_t<c2h::accumulateable_t,
                     c2h::equal_comparable_t,
                     c2h::lexicographical_less_comparable_t,
                     c2h::lexicographical_greater_comparable_t>;

#if TEST_TYPES == 0
using full_type_list = c2h::type_list<type_pair<std::uint8_t>, type_pair<std::int8_t, std::int32_t>>;
#elif TEST_TYPES == 1
using full_type_list = c2h::type_list<type_pair<std::int32_t>, type_pair<std::int64_t>>;
#elif TEST_TYPES == 2
using full_type_list = c2h::type_list<type_pair<uchar3>, type_pair<ulonglong4>>;
#elif TEST_TYPES == 3
// clang-format off
using full_type_list = c2h::type_list<
type_pair<custom_t>
#if TEST_HALF_T
, type_pair<half_t> // testing half
#endif
#if TEST_BF_T
, type_pair<bfloat16_t> // testing bf16

>;
#endif
// clang-format on
#elif TEST_TYPES == 4
// DPX SIMD instructions
using full_type_list = c2h::type_list<type_pair<std::uint16_t>, type_pair<std::int16_t>>;
#endif

/**
 * @brief Input data generation mode
 */
enum class gen_data_t : int
{
  /// Uniform random data generation
  GEN_TYPE_RANDOM,
  /// Constant value as input data
  GEN_TYPE_CONST
};

C2H_TEST("Device reduce works with all device interfaces", "[reduce][device]", full_type_list)
{
  using params   = params_t<TestType>;
  using item_t   = typename params::item_t;
  using output_t = typename params::output_t;
  using offset_t = int32_t;

  constexpr int max_items    = 5000000;
  constexpr int min_items    = 1;
  constexpr int num_segments = 1;

  // Generate the input sizes to test for
  const int num_items = GENERATE_COPY(
    take(3, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));

  // Input data generation to test
  const gen_data_t data_gen_mode = GENERATE_COPY(gen_data_t::GEN_TYPE_RANDOM, gen_data_t::GEN_TYPE_CONST);

  // Generate input data
  c2h::device_vector<item_t> in_items(num_items);
  if (data_gen_mode == gen_data_t::GEN_TYPE_RANDOM)
  {
    c2h::gen(C2H_SEED(2), in_items);
  }
  else
  {
    item_t default_constant{};
    init_default_constant(default_constant);
    thrust::fill(c2h::device_policy, in_items.begin(), in_items.end(), default_constant);
  }
  auto d_in_it = thrust::raw_pointer_cast(in_items.data());

#if TEST_TYPES != 4
  SECTION("reduce")
  {
    using op_t = hipcub::Sum;

    // Binary reduction operator
    auto reduction_op = unwrap_op(reference_extended_fp(d_in_it), op_t{});

    // Prepare verification data
    using accum_t = ::cuda::std::__accumulator_t<op_t, item_t, output_t>;
    output_t expected_result =
      static_cast<output_t>(compute_single_problem_reference(in_items, reduction_op, accum_t{}));

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    using init_t  = hipcub::detail::value_t<decltype(unwrap_it(d_out_it))>;
    device_reduce(unwrap_it(d_in_it), unwrap_it(d_out_it), num_items, reduction_op, init_t{});

    // Verify result
    REQUIRE(expected_result == out_result[0]);
  }
#endif // TEST_TYPES != 4

// Skip DeviceReduce::Sum tests for extended floating-point types because of unbounded epsilon due
// to pseudo associativity of the addition operation over floating point numbers
#if TEST_TYPES != 3
  SECTION("sum")
  {
    using op_t    = hipcub::Sum;
    using accum_t = ::cuda::std::__accumulator_t<op_t, item_t, output_t>;

    // Prepare verification data
    output_t expected_result = static_cast<output_t>(compute_single_problem_reference(in_items, op_t{}, accum_t{}));

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = unwrap_it(thrust::raw_pointer_cast(out_result.data()));
    device_sum(d_in_it, d_out_it, num_items);

    // Verify result
    REQUIRE(expected_result == out_result[0]);
  }
#endif

  SECTION("min")
  {
    // Prepare verification data
    c2h::host_vector<item_t> host_items(in_items);
    auto expected_result = *std::min_element(host_items.cbegin(), host_items.cend());

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    device_min(unwrap_it(d_in_it), unwrap_it(d_out_it), num_items);

    // Verify result
    REQUIRE(expected_result == out_result[0]);
  }

  SECTION("max")
  {
    // Prepare verification data
    c2h::host_vector<item_t> host_items(in_items);
    auto expected_result = *std::max_element(host_items.cbegin(), host_items.cend());

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    device_max(unwrap_it(d_in_it), unwrap_it(d_out_it), num_items);

    // Verify result
    REQUIRE(expected_result == out_result[0]);
  }

#if TEST_TYPES != 4
  SECTION("argmax")
  {
    // Prepare verification data
    c2h::host_vector<item_t> host_items(in_items);
    auto expected_result = std::max_element(host_items.cbegin(), host_items.cend());

    // Run test

    using result_t = hipcub::KeyValuePair<int, unwrap_value_t<output_t>>;
    c2h::device_vector<result_t> out_result(num_segments);
    device_arg_max(unwrap_it(d_in_it), thrust::raw_pointer_cast(out_result.data()), num_items);

    // Verify result
    result_t gpu_result = out_result[0];
    output_t gpu_value  = static_cast<output_t>(gpu_result.value); // Explicitly rewrap the gpu value
    REQUIRE(expected_result[0] == gpu_value);
    REQUIRE((expected_result - host_items.cbegin()) == gpu_result.key);
  }

  SECTION("argmin")
  {
    // Prepare verification data
    c2h::host_vector<item_t> host_items(in_items);
    auto expected_result = std::min_element(host_items.cbegin(), host_items.cend());

    // Run test
    using result_t = hipcub::KeyValuePair<int, unwrap_value_t<output_t>>;
    c2h::device_vector<result_t> out_result(num_segments);
    device_arg_min(unwrap_it(d_in_it), thrust::raw_pointer_cast(out_result.data()), num_items);

    // Verify result
    result_t gpu_result = out_result[0];
    output_t gpu_value  = static_cast<output_t>(gpu_result.value); // Explicitly rewrap the gpu value
    REQUIRE(expected_result[0] == gpu_value);
    REQUIRE((expected_result - host_items.cbegin()) == gpu_result.key);
  }
#endif
}
