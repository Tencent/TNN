#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_segmented_reduce.cuh>

#include <cuda/std/limits>

#include "catch2_test_device_reduce.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>
#include <c2h/extended_types.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::Reduce, device_segmented_reduce);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::Sum, device_segmented_sum);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::Min, device_segmented_min);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::ArgMin, device_segmented_arg_min);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::Max, device_segmented_max);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSegmentedReduce::ArgMax, device_segmented_arg_max);

// %PARAM% TEST_LAUNCH lid 0:1:2
// %PARAM% TEST_TYPES types 0:1:2:3

// List of types to test
using custom_t =
  c2h::custom_type_t<c2h::accumulateable_t,
                     c2h::equal_comparable_t,
                     c2h::lexicographical_less_comparable_t,
                     c2h::lexicographical_greater_comparable_t>;

#if TEST_TYPES == 0
using full_type_list = c2h::type_list<type_pair<std::uint8_t>, type_pair<std::int8_t, std::int32_t>>;
#elif TEST_TYPES == 1
using full_type_list = c2h::type_list<type_pair<std::int32_t>, type_pair<std::int64_t>>;
#elif TEST_TYPES == 2
using full_type_list = c2h::type_list<type_pair<uchar3>, type_pair<ulonglong4>>;
#elif TEST_TYPES == 3
// clang-format off
using full_type_list = c2h::type_list<
type_pair<custom_t>
#if TEST_HALF_T
, type_pair<half_t> // testing half
#endif
#if TEST_BF_T
, type_pair<bfloat16_t> // testing bf16
#endif
>;
// clang-format on
#endif

using offsets = c2h::type_list<std::int32_t, std::uint32_t>;

C2H_TEST("Device reduce works with all device interfaces", "[segmented][reduce][device]", full_type_list, offsets)
{
  using type_pair_t = typename c2h::get<0, TestType>;
  using input_t     = typename type_pair_t::input_t;
  using output_t    = typename type_pair_t::output_t;
  using offset_t    = typename c2h::get<1, TestType>;

  constexpr int min_items = 1;
  constexpr int max_items = 1000000;

  // Number of items
  const int num_items = GENERATE_COPY(
    take(2, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));
  INFO("Test num_items: " << num_items);

  // Range of segment sizes to generate
  // Note that the segment range [0, 1] may also include one last segment with more than 1 items
  const std::tuple<offset_t, offset_t> seg_size_range =
    GENERATE_COPY(table<offset_t, offset_t>({{0, 1}, {1, num_items}, {num_items, num_items}}));
  INFO("Test seg_size_range: [" << std::get<0>(seg_size_range) << ", " << std::get<1>(seg_size_range) << "]");

  // Generate input segments
  c2h::device_vector<offset_t> segment_offsets = c2h::gen_uniform_offsets<offset_t>(
    C2H_SEED(1), num_items, std::get<0>(seg_size_range), std::get<1>(seg_size_range));
  const offset_t num_segments = static_cast<offset_t>(segment_offsets.size() - 1);
  auto d_offsets_it           = thrust::raw_pointer_cast(segment_offsets.data());

  // Generate input data
  c2h::device_vector<input_t> in_items(num_items);
  c2h::gen(C2H_SEED(2), in_items);
  auto d_in_it = thrust::raw_pointer_cast(in_items.data());

  SECTION("reduce")
  {
    using op_t = hipcub::Sum;

    // Binary reduction operator
    auto reduction_op = unwrap_op(reference_extended_fp(d_in_it), op_t{});

    // Prepare verification data
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, output_t>;
    c2h::host_vector<output_t> expected_result(num_segments);
    compute_segmented_problem_reference(in_items, segment_offsets, reduction_op, accum_t{}, expected_result.begin());

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    using init_t  = hipcub::detail::value_t<decltype(unwrap_it(d_out_it))>;
    device_segmented_reduce(
      unwrap_it(d_in_it), unwrap_it(d_out_it), num_segments, d_offsets_it, d_offsets_it + 1, reduction_op, init_t{});

    // Verify result
    REQUIRE(expected_result == out_result);
  }

// Skip DeviceReduce::Sum tests for extended floating-point types because of unbounded epsilon due
// to pseudo associativity of the addition operation over floating point numbers
#if TEST_TYPES != 3
  SECTION("sum")
  {
    using op_t    = hipcub::Sum;
    using accum_t = ::cuda::std::__accumulator_t<op_t, input_t, output_t>;

    // Prepare verification data
    c2h::host_vector<output_t> expected_result(num_segments);
    compute_segmented_problem_reference(in_items, segment_offsets, op_t{}, accum_t{}, expected_result.begin());

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = unwrap_it(thrust::raw_pointer_cast(out_result.data()));
    device_segmented_sum(d_in_it, d_out_it, num_segments, d_offsets_it, d_offsets_it + 1);

    // Verify result
    REQUIRE(expected_result == out_result);
  }
#endif

  SECTION("min")
  {
    using op_t = hipcub::Min;

    // Prepare verification data
    c2h::host_vector<output_t> expected_result(num_segments);
    compute_segmented_problem_reference(
      in_items, segment_offsets, op_t{}, hipcub::NumericTraits<input_t>::Max(), expected_result.begin());

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    device_segmented_min(unwrap_it(d_in_it), unwrap_it(d_out_it), num_segments, d_offsets_it, d_offsets_it + 1);

    // Verify result
    REQUIRE(expected_result == out_result);
  }

  SECTION("max")
  {
    using op_t = hipcub::Max;

    // Prepare verification data
    c2h::host_vector<output_t> expected_result(num_segments);
    compute_segmented_problem_reference(
      in_items, segment_offsets, op_t{}, hipcub::NumericTraits<input_t>::Lowest(), expected_result.begin());

    // Run test
    c2h::device_vector<output_t> out_result(num_segments);
    auto d_out_it = thrust::raw_pointer_cast(out_result.data());
    device_segmented_max(unwrap_it(d_in_it), unwrap_it(d_out_it), num_segments, d_offsets_it, d_offsets_it + 1);

    // Verify result
    REQUIRE(expected_result == out_result);
  }

  SECTION("argmax")
  {
    using result_t = hipcub::KeyValuePair<int, output_t>;

    // Prepare verification data
    c2h::host_vector<result_t> expected_result(num_segments);
    compute_segmented_argmax_reference(in_items, segment_offsets, expected_result.begin());

    // Run test
    c2h::device_vector<result_t> out_result(num_segments);
    device_segmented_arg_max(
      d_in_it, thrust::raw_pointer_cast(out_result.data()), num_segments, d_offsets_it, d_offsets_it + 1);

    // Verify result
    REQUIRE(expected_result == out_result);
  }

  SECTION("argmin")
  {
    using result_t = hipcub::KeyValuePair<int, output_t>;

    // Prepare verification data
    c2h::host_vector<input_t> host_items(in_items);
    c2h::host_vector<result_t> expected_result(num_segments);
    compute_segmented_argmin_reference(in_items, segment_offsets, expected_result.begin());

    // Run test
    c2h::device_vector<result_t> out_result(num_segments);
    device_segmented_arg_min(
      d_in_it, thrust::raw_pointer_cast(out_result.data()), num_segments, d_offsets_it, d_offsets_it + 1);
    // Verify result
    REQUIRE(expected_result == out_result);
  }
}
