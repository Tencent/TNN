#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_select.cuh>

#include <thrust/count.h>
#include <thrust/partition.h>
#include <thrust/reverse.h>

#include <algorithm>

#include "catch2_test_device_select_common.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>

template <class T, class FlagT>
static c2h::host_vector<T> get_reference(const c2h::device_vector<T>& in, const c2h::device_vector<FlagT>& flags)
{
  struct selector
  {
    const T* ref_begin      = nullptr;
    const FlagT* flag_begin = nullptr;

    constexpr selector(const T* ref, const FlagT* flag) noexcept
        : ref_begin(ref)
        , flag_begin(flag)
    {}

    bool operator()(const T& val) const
    {
      const auto pos = &val - ref_begin;
      return static_cast<bool>(flag_begin[pos]);
    }
  };

  c2h::host_vector<T> reference   = in;
  c2h::host_vector<FlagT> h_flags = flags;

  const selector pred{thrust::raw_pointer_cast(reference.data()), thrust::raw_pointer_cast(h_flags.data())};
  const auto boundary = std::stable_partition(reference.begin(), reference.end(), pred);
  reference.erase(boundary, reference.end());
  return reference;
}

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceSelect::Flagged, select_flagged);

// %PARAM% TEST_LAUNCH lid 0:1:2

using all_types =
  c2h::type_list<std::uint8_t,
                 std::uint16_t,
                 std::uint32_t,
                 std::uint64_t,
                 ulonglong2,
                 ulonglong4,
                 int,
                 long2,
                 c2h::custom_type_t<c2h::equal_comparable_t>>;

using types = c2h::type_list<std::uint8_t, std::uint32_t, ulonglong4, c2h::custom_type_t<c2h::equal_comparable_t>>;

C2H_TEST("DeviceSelect::Flagged can run with empty input", "[device][select_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  constexpr int num_items = 0;
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::device_vector<int> flags(num_items);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 42);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected_out[0] == 0);
}

C2H_TEST("DeviceSelect::Flagged handles all matched", "[device][select_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items, 1);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected_out[0] == num_items);
  REQUIRE(out == in);
}

C2H_TEST("DeviceSelect::Flagged handles no matched", "[device][select_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(0);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items, 0);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected_out[0] == 0);
}

C2H_TEST("DeviceSelect::Flagged does not change input", "[device][select_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);
  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // copy input first
  c2h::device_vector<type> reference = in;

  select_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == in);
}

C2H_TEST("DeviceSelect::Flagged is stable",
         "[device][select_flagged]",
         c2h::type_list<c2h::custom_type_t<c2h::equal_comparable_t>>)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);
  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  out.resize(num_selected_out[0]);
  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DeviceSelect::Flagged works with iterators", "[device][select_flagged]", all_types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);
  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.data(), flags.begin(), out.data(), d_num_selected_out, num_items);

  out.resize(num_selected_out[0]);
  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DeviceSelect::Flagged works with pointers", "[device][select_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(thrust::raw_pointer_cast(in.data()),
                 thrust::raw_pointer_cast(flags.data()),
                 thrust::raw_pointer_cast(out.data()),
                 d_num_selected_out,
                 num_items);

  out.resize(num_selected_out[0]);
  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

struct convertible_to_bool
{
  int val_;

  convertible_to_bool() = default;
  __host__ __device__ convertible_to_bool(const int val) noexcept
      : val_(val)
  {}

  __host__ __device__ operator bool() const noexcept
  {
    return static_cast<bool>(val_);
  }
  __host__ __device__ friend bool operator==(const convertible_to_bool& lhs, const int& rhs) noexcept
  {
    return lhs.val_ == rhs;
  }
  __host__ __device__ friend bool operator==(const int& lhs, const convertible_to_bool& rhs) noexcept
  {
    return lhs == rhs.val_;
  }
};

C2H_TEST("DeviceSelect::Flagged works with flags that are convertible to bool", "[device][select_flagged]")
{
  using type = c2h::custom_type_t<c2h::equal_comparable_t>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<type> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> iflags(num_items);
  c2h::gen(C2H_SEED(1), iflags, 0, 1);

  c2h::device_vector<convertible_to_bool> flags = iflags;
  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  out.resize(num_selected_out[0]);
  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DeviceSelect::Flagged works with flags that alias input", "[device][select_flagged]")
{
  using type = int;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> out(num_items);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);
  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(flags, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(flags.begin(), flags.begin(), out.begin(), d_num_selected_out, num_items);

  out.resize(num_selected_out[0]);
  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DeviceSelect::Flagged works in place", "[device][select_flagged]", types)
{
  using type = typename c2h::get<0, TestType>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.begin(), flags.begin(), d_num_selected_out, num_items);

  in.resize(num_selected_out[0]);
  REQUIRE(reference == in);
}

C2H_TEST("DeviceSelect::Flagged works in place with flags that alias input", "[device][select_flagged]")
{
  using type = int;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<int> flags(num_items);

  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(flags, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(flags.begin(), flags.begin(), d_num_selected_out, num_items);

  flags.resize(num_selected_out[0]);
  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == flags);
}

template <class T>
struct convertible_from_T
{
  T val_;

  convertible_from_T() = default;
  __host__ __device__ convertible_from_T(const T& val) noexcept
      : val_(val)
  {}
  __host__ __device__ convertible_from_T& operator=(const T& val) noexcept
  {
    val_ = val;
  }
  // Converting back to T helps satisfy all the machinery that T supports
  __host__ __device__ operator T() const noexcept
  {
    return val_;
  }
};

C2H_TEST("DeviceSelect::Flagged works with a different output type", "[device][select_flagged]")
{
  using type = c2h::custom_type_t<c2h::equal_comparable_t>;

  const int num_items = GENERATE_COPY(take(2, random(1, 1000000)));
  c2h::device_vector<type> in(num_items);
  c2h::device_vector<convertible_from_T<type>> out(num_items);
  c2h::gen(C2H_SEED(2), in);

  c2h::device_vector<int> flags(num_items);
  c2h::gen(C2H_SEED(1), flags, 0, 1);

  const int num_selected = static_cast<int>(thrust::count(c2h::device_policy, flags.begin(), flags.end(), 1));
  const c2h::host_vector<type> reference = get_reference(in, flags);

  // Needs to be device accessible
  c2h::device_vector<int> num_selected_out(1, 0);
  int* d_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  select_flagged(in.data(), flags.begin(), out.data(), d_num_selected_out, num_items);

  out.resize(num_selected_out[0]);
  REQUIRE(num_selected == num_selected_out[0]);
  REQUIRE(reference == out);
}

C2H_TEST("DeviceSelect::Flagged works for very large number of items", "[device][select_flagged]")
try
{
  using type     = std::int64_t;
  using offset_t = std::int64_t;

  // The partition size (the maximum number of items processed by a single kernel invocation) is an important boundary
  constexpr auto max_partition_size = static_cast<offset_t>(::cuda::std::numeric_limits<std::int32_t>::max());

  offset_t num_items = GENERATE_COPY(
    values({
      offset_t{2} * max_partition_size + offset_t{20000000}, // 3 partitions
      offset_t{2} * max_partition_size, // 2 partitions
      max_partition_size + offset_t{1}, // 2 partitions
      max_partition_size, // 1 partitions
      max_partition_size - offset_t{1} // 1 partitions
    }),
    take(2, random(max_partition_size - offset_t{1000000}, max_partition_size + offset_t{1000000})));

  // Input
  constexpr offset_t match_every_nth = 1000000;
  auto in                            = thrust::make_counting_iterator(static_cast<type>(0));
  auto flags_in = thrust::make_transform_iterator(in, mod_n<offset_t>{static_cast<offset_t>(match_every_nth)});

  // Needs to be device accessible
  c2h::device_vector<offset_t> num_selected_out(1, 0);
  offset_t* d_first_num_selected_out = thrust::raw_pointer_cast(num_selected_out.data());

  // Run test
  offset_t expected_num_copied = (num_items + match_every_nth - offset_t{1}) / match_every_nth;
  c2h::device_vector<type> out(expected_num_copied);
  select_flagged(in, flags_in, out.begin(), d_first_num_selected_out, num_items);

  // Ensure that we created the correct output
  REQUIRE(num_selected_out[0] == expected_num_copied);
  auto expected_out_it =
    thrust::make_transform_iterator(in, multiply_n<offset_t>{static_cast<offset_t>(match_every_nth)});
  bool all_results_correct = thrust::equal(out.cbegin(), out.cend(), expected_out_it);
  REQUIRE(all_results_correct == true);
}
catch (std::bad_alloc&)
{
  // Exceeding memory is not a failure.
}
