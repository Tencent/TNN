#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_histogram.cuh>
#include <cub/iterator/counting_input_iterator.cuh>

#include <cuda/std/__algorithm_>
#include <cuda/std/array>
#include <cuda/std/bit>
#include <cuda/std/type_traits>

#include <algorithm>
#include <limits>
#include <tuple>

#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/extended_types.cuh>
#include <c2h/vector.cuh>

// %PARAM% TEST_LAUNCH lid 0:1:2

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceHistogram::HistogramEven, histogram_even);
DECLARE_LAUNCH_WRAPPER(hipcub::DeviceHistogram::HistogramRange, histogram_range);

DECLARE_TMPL_LAUNCH_WRAPPER(hipcub::DeviceHistogram::MultiHistogramEven,
                            multi_histogram_even,
                            ESCAPE_LIST(int Channels, int ActiveChannels),
                            ESCAPE_LIST(Channels, ActiveChannels));

DECLARE_TMPL_LAUNCH_WRAPPER(hipcub::DeviceHistogram::MultiHistogramRange,
                            multi_histogram_range,
                            ESCAPE_LIST(int Channels, int ActiveChannels),
                            ESCAPE_LIST(Channels, ActiveChannels));

namespace cs = cuda::std;
using cs::array;
using cs::size_t;

template <typename T>
auto cast_if_half_pointer(T* p) -> T*
{
  return p;
}

#if TEST_HALF_T
auto cast_if_half_pointer(half_t* p) -> __half*
{
  return reinterpret_cast<__half*>(p);
}

auto cast_if_half_pointer(const half_t* p) -> const __half*
{
  return reinterpret_cast<const __half*>(p);
}
#endif

template <typename T>
using caller_vector = c2h::
#if TEST_LAUNCH == 1
  device_vector<T>;
#else
  host_vector<T>;
#endif

template <typename T, size_t N>
auto to_caller_vector_of_ptrs(array<c2h::device_vector<T>, N>& in)
  -> caller_vector<decltype(cast_if_half_pointer(cs::declval<T*>()))>
{
  c2h::host_vector<decltype(cast_if_half_pointer(cs::declval<T*>()))> r(N);
  for (size_t i = 0; i < N; i++)
  {
    r[i] = cast_if_half_pointer(thrust::raw_pointer_cast(in[i].data()));
  }
  return r;
}

template <int Channels, typename CounterT, size_t ActiveChannels, typename SampleT, typename TransformOp, typename OffsetT>
auto compute_reference_result(
  const c2h::host_vector<SampleT>& h_samples,
  const TransformOp& sample_to_bin_index,
  const array<int, ActiveChannels>& num_levels,
  OffsetT width,
  OffsetT height,
  OffsetT row_pitch) -> array<c2h::host_vector<CounterT>, ActiveChannels>
{
  auto h_histogram = array<c2h::host_vector<CounterT>, ActiveChannels>{};
  for (size_t c = 0; c < ActiveChannels; ++c)
  {
    h_histogram[c].resize(num_levels[c] - 1);
  }
  for (OffsetT row = 0; row < height; ++row)
  {
    for (OffsetT pixel = 0; pixel < width; ++pixel)
    {
      for (size_t c = 0; c < ActiveChannels; ++c)
      {
        // TODO(bgruber): use an mdspan to access h_samples
        const auto offset = row * (row_pitch / sizeof(SampleT)) + pixel * Channels + c;
        const int bin     = sample_to_bin_index(static_cast<int>(c), h_samples[offset]);
        if (bin >= 0 && bin < static_cast<int>(h_histogram[c].size())) // if bin is valid
        {
          ++h_histogram[c][bin];
        }
      }
    }
  }
  return h_histogram;
}

template <size_t ActiveChannels, typename LevelT>
auto setup_bin_levels_for_even(const array<int, ActiveChannels>& num_levels, LevelT max_level, int max_level_count)
  -> array<array<LevelT, ActiveChannels>, 2>
{
  array<array<LevelT, ActiveChannels>, 2> levels;
  auto& lower_level = levels[0];
  auto& upper_level = levels[1];

  // Create upper and lower levels between between [0:max_level], getting narrower with each channel. Example:
  //    max_level = 256
  //   num_levels = { 257, 129,  65 }
  //  lower_level = {   0,  64,  96 }
  //  upper_level = { 256, 192, 160 }

  // TODO(bgruber): eventually, we could just pick a random lower/upper bound for each channel

  const auto min_bin_width = max_level / (max_level_count - 1);
  REQUIRE(min_bin_width > 0);

  for (size_t c = 0; c < ActiveChannels; ++c)
  {
    const int num_bins        = num_levels[c] - 1;
    const auto min_hist_width = num_bins * min_bin_width;
    lower_level[c]            = static_cast<LevelT>(max_level / 2 - min_hist_width / 2);
    upper_level[c]            = static_cast<LevelT>(max_level / 2 + min_hist_width / 2);
    CAPTURE(c, num_levels[c]);
    REQUIRE(lower_level[c] < upper_level[c]);
  }
  return levels;
}

template <size_t ActiveChannels, typename LevelT>
auto setup_bin_levels_for_range(const array<int, ActiveChannels>& num_levels, LevelT max_level, int max_level_count)
  -> array<c2h::host_vector<LevelT>, ActiveChannels>
{
  // TODO(bgruber): eventually, we could just pick random levels for each channel

  const auto min_bin_width = max_level / (max_level_count - 1);
  REQUIRE(min_bin_width > 0);

  array<c2h::host_vector<LevelT>, ActiveChannels> levels;
  for (size_t c = 0; c < ActiveChannels; ++c)
  {
    levels[c].resize(num_levels[c]);
    const int num_bins        = num_levels[c] - 1;
    const auto min_hist_width = num_bins * min_bin_width;
    const auto lower_level    = (max_level / 2 - min_hist_width / 2);
    for (int l = 0; l < num_levels[c]; ++l)
    {
      levels[c][l] = static_cast<LevelT>(lower_level + l * min_bin_width);
      if (l > 0)
      {
        REQUIRE(levels[c][l - 1] < levels[c][l]);
      }
    }
  }
  return levels;
}

template <size_t ActiveChannels>
auto generate_level_counts_to_test(int max_level_count) -> array<int, ActiveChannels>
{
  // TODO(bgruber): eventually, just pick a random number of levels per channel

  // first channel tests maximum number of levels, later channels less and less
  array<int, ActiveChannels> r{max_level_count};
  for (size_t c = 1; c < ActiveChannels; ++c)
  {
    r[c] = r[c - 1] / 2 + 1;
  }
  return r;
}

struct bit_and_anything
{
  template <typename T>
  _CCCL_HOST_DEVICE auto operator()(const T& a, const T& b) const -> T
  {
    using U = typename hipcub::Traits<T>::UnsignedBits;
    return ::cuda::std::bit_cast<T>(static_cast<U>(::cuda::std::bit_cast<U>(a) & ::cuda::std::bit_cast<U>(b)));
  }
};

template <typename SampleT, int Channels, size_t ActiveChannels, typename CounterT, typename LevelT, typename OffsetT>
void test_even_and_range(LevelT max_level, int max_level_count, OffsetT width, OffsetT height, int entropy_reduction = 0)
{
  const auto padding_bytes = static_cast<OffsetT>(GENERATE(size_t{0}, 13 * sizeof(SampleT)));
  CAPTURE(
    c2h::type_name<SampleT>(),
    c2h::type_name<CounterT>(),
    c2h::type_name<LevelT>(),
    c2h::type_name<OffsetT>(),
    Channels,
    ActiveChannels,
    CoutCast(max_level),
    max_level_count,
    width,
    height,
    padding_bytes,
    entropy_reduction);

  // Prepare input image (samples)
  const OffsetT row_pitch = width * Channels * sizeof(SampleT) + padding_bytes;
  const auto num_levels   = generate_level_counts_to_test<ActiveChannels>(max_level_count);

  const OffsetT total_samples = height * (row_pitch / sizeof(SampleT));
  c2h::device_vector<SampleT> d_samples;
  d_samples.resize(total_samples);

  if (entropy_reduction >= 0)
  {
    c2h::gen(C2H_SEED(1), d_samples, SampleT{0}, static_cast<SampleT>(max_level));
    if (entropy_reduction > 0)
    {
      c2h::device_vector<SampleT> tmp(d_samples.size());
      for (int i = 0; i < entropy_reduction; ++i)
      {
        c2h::gen(C2H_SEED(1), tmp);
        thrust::transform(
          c2h::device_policy, d_samples.cbegin(), d_samples.cend(), tmp.cbegin(), d_samples.begin(), bit_and_anything{});
      }
    }
  }

  auto h_samples = c2h::host_vector<SampleT>(d_samples);

  // Allocate output histogram
  auto d_histogram = array<c2h::device_vector<CounterT>, ActiveChannels>();
  for (size_t c = 0; c < ActiveChannels; ++c)
  {
    d_histogram[c].resize(num_levels[c] - 1);
  }

  SECTION("HistogramEven")
  {
    // Setup levels
    const auto levels       = setup_bin_levels_for_even(num_levels, max_level, max_level_count);
    const auto& lower_level = levels[0]; // TODO(bgruber): use structured bindings in C++17
    const auto& upper_level = levels[1];
    CAPTURE(lower_level, upper_level);

    // Compute reference result
    auto fp_scales = array<LevelT, ActiveChannels>{}; // only used when LevelT is floating point
    std::ignore    = fp_scales; // casting to void was insufficient. TODO(bgruber): use [[maybe_unsued]] in C++17
    for (size_t c = 0; c < ActiveChannels; ++c)
    {
      _CCCL_IF_CONSTEXPR (!cs::is_integral<LevelT>::value)
      {
        fp_scales[c] = static_cast<LevelT>(num_levels[c] - 1) / static_cast<LevelT>(upper_level[c] - lower_level[c]);
      }
    }

    auto sample_to_bin_index = [&](int channel, SampleT sample) {
      using common_t             = typename cs::common_type<LevelT, SampleT>::type;
      const auto n               = num_levels[channel];
      const auto max             = static_cast<common_t>(upper_level[channel]);
      const auto min             = static_cast<common_t>(lower_level[channel]);
      const auto promoted_sample = static_cast<common_t>(sample);
      if (promoted_sample < min || promoted_sample >= max)
      {
        return n; // out of range
      }
      _CCCL_IF_CONSTEXPR (cs::is_integral<LevelT>::value)
      {
        // Accurate bin computation following the arithmetic we guarantee in the HistoEven docs
        return static_cast<int>(static_cast<uint64_t>(promoted_sample - min) * static_cast<uint64_t>(n - 1)
                                / static_cast<uint64_t>(max - min));
      }
      else
      {
        return static_cast<int>((sample - min) * fp_scales[channel]);
      }
      _CCCL_UNREACHABLE();
    };
    auto h_histogram = compute_reference_result<Channels, CounterT>(
      h_samples, sample_to_bin_index, num_levels, width, height, row_pitch);

    // Compute result and verify
    {
      const auto* sample_ptr = cast_if_half_pointer(thrust::raw_pointer_cast(d_samples.data()));
      _CCCL_IF_CONSTEXPR (ActiveChannels == 1 && Channels == 1)
      {
        histogram_even(
          sample_ptr,
          cast_if_half_pointer(thrust::raw_pointer_cast(d_histogram[0].data())),
          num_levels[0],
          cast_if_half_pointer(lower_level.data())[0],
          cast_if_half_pointer(upper_level.data())[0],
          width,
          height,
          row_pitch);
      }
      else
      {
        auto d_histogram_ptrs    = to_caller_vector_of_ptrs(d_histogram);
        const auto d_num_levels  = caller_vector<int>(num_levels.begin(), num_levels.end());
        const auto d_lower_level = caller_vector<LevelT>(lower_level.begin(), lower_level.end());
        const auto d_upper_level = caller_vector<LevelT>(upper_level.begin(), upper_level.end());
        multi_histogram_even<Channels, ActiveChannels>(
          sample_ptr,
          cast_if_half_pointer(thrust::raw_pointer_cast(d_histogram_ptrs.data())),
          thrust::raw_pointer_cast(d_num_levels.data()),
          cast_if_half_pointer(thrust::raw_pointer_cast(d_lower_level.data())),
          cast_if_half_pointer(thrust::raw_pointer_cast(d_upper_level.data())),
          width,
          height,
          row_pitch);
      }
    }
    for (size_t c = 0; c < ActiveChannels; ++c)
    {
      CHECK(h_histogram[c] == d_histogram[c]);
    }
  }

  SECTION("HistogramRange")
  {
    // Setup levels
    const auto h_levels = setup_bin_levels_for_range(num_levels, max_level, max_level_count);
    CAPTURE(h_levels);

    // Compute reference result
    const auto sample_to_bin_index = [&](int channel, SampleT sample) {
      const auto* l  = h_levels[channel].data();
      const auto n   = static_cast<int>(h_levels[channel].size());
      const auto* ub = std::upper_bound(l, l + n, static_cast<LevelT>(sample));
      return ub == l /* sample smaller than first bin */ ? n : static_cast<int>(std::distance(l, ub) - 1);
    };
    auto h_histogram = compute_reference_result<Channels, CounterT>(
      h_samples, sample_to_bin_index, num_levels, width, height, row_pitch);

    // Compute result and verify
    {
      const auto* sample_ptr = cast_if_half_pointer(thrust::raw_pointer_cast(d_samples.data()));
      auto d_levels          = array<c2h::device_vector<LevelT>, ActiveChannels>{};
      std::copy(h_levels.begin(), h_levels.end(), d_levels.begin());
      _CCCL_IF_CONSTEXPR (ActiveChannels == 1 && Channels == 1)
      {
        histogram_range(
          sample_ptr,
          cast_if_half_pointer(thrust::raw_pointer_cast(d_histogram[0].data())),
          num_levels[0],
          cast_if_half_pointer(thrust::raw_pointer_cast(d_levels[0].data())),
          width,
          height,
          row_pitch);
      }
      else
      {
        auto d_histogram_ptrs   = to_caller_vector_of_ptrs(d_histogram);
        const auto d_num_levels = caller_vector<int>(num_levels.begin(), num_levels.end());
        const auto level_ptrs   = to_caller_vector_of_ptrs(d_levels);
        multi_histogram_range<Channels, ActiveChannels>(
          sample_ptr,
          cast_if_half_pointer(thrust::raw_pointer_cast(d_histogram_ptrs.data())),
          thrust::raw_pointer_cast(d_num_levels.data()),
          cast_if_half_pointer(thrust::raw_pointer_cast(level_ptrs.data())),
          width,
          height,
          row_pitch);
      }
    }
    for (size_t c = 0; c < ActiveChannels; ++c)
    {
      CHECK(h_histogram[c] == d_histogram[c]);
    }
  }
}

using types =
  c2h::type_list<std::int8_t,
                 std::uint8_t,
                 std::int16_t,
                 std::uint16_t,
                 std::int32_t,
                 std::uint32_t,
                 std::int64_t,
                 std::uint64_t,
#if TEST_HALF_T
                 half_t,
#endif
                 float,
                 double>;

C2H_TEST("DeviceHistogram::Histogram* basic use", "[histogram][device]", types)
{
  using sample_t = c2h::get<0, TestType>;
  using level_t =
    typename cs::conditional<hipcub::NumericTraits<sample_t>::CATEGORY == hipcub::FLOATING_POINT, sample_t, int>::type;
  // Max for int8/uint8 is 2^8, for half_t is 2^10. Beyond, we would need a different level generation
  const auto max_level       = level_t{sizeof(sample_t) == 1 ? 126 : 1024};
  const auto max_level_count = (sizeof(sample_t) == 1 ? 126 : 1024) + 1;
  test_even_and_range<sample_t, 4, 3, int>(max_level, max_level_count, 1920, 1080);
}

// TODO(bgruber): float produces INFs in the HistogramRange test setup AND the HistogramEven implementation
// This test covers int32 and int64 arithmetic for bin computation
C2H_TEST("DeviceHistogram::Histogram* large levels", "[histogram][device]", c2h::remove<types, float>)
{
  using sample_t             = c2h::get<0, TestType>;
  using level_t              = sample_t;
  const auto max_level_count = 128;
  auto max_level             = hipcub::NumericTraits<level_t>::Max();
  _CCCL_IF_CONSTEXPR (sizeof(sample_t) > sizeof(int))
  {
    max_level /= static_cast<level_t>(max_level_count - 1); // cf. overflow detection in ScaleTransform::MayOverflow
  }
  test_even_and_range<sample_t, 4, 3, int>(max_level, max_level_count, 1920, 1080);
}

C2H_TEST("DeviceHistogram::Histogram* odd image sizes", "[histogram][device]")
{
  using sample_t                = int;
  using level_t                 = int;
  constexpr sample_t max_level  = 256;
  constexpr int max_level_count = 256 + 1;

  using P      = cs::pair<int, int>;
  const auto p = GENERATE(P{1920, 0}, P{0, 0}, P{0, 1080}, P{1, 1}, P{15, 1}, P{1, 15}, P{10000, 1}, P{1, 10000});
  test_even_and_range<sample_t, 4, 3, int, level_t, int>(max_level, max_level_count, p.first, p.second);
}

C2H_TEST("DeviceHistogram::Histogram* entropy", "[histogram][device]")
{
  const int entropy_reduction = GENERATE(-1, 3, 5); // entropy_reduction = -1 -> all samples == 0
  test_even_and_range<int, 4, 3, int>(256, 256 + 1, 1920, 1080, entropy_reduction);
}

template <int Channels, int ActiveChannels>
struct ChannelConfig
{
  static constexpr auto channels        = Channels;
  static constexpr auto active_channels = ActiveChannels;
};

C2H_TEST_LIST("DeviceHistogram::Histogram* channel configs",
              "[histogram][device]",
              ChannelConfig<1, 1>,
              ChannelConfig<3, 3>,
              ChannelConfig<4, 3>,
              ChannelConfig<4, 4>)
{
  test_even_and_range<int, TestType::channels, TestType::active_channels, int, int, int>(256, 256 + 1, 128, 32);
}

// Testing only HistogramEven is fine, because HistogramRange shares the loading logic and the different binning
// implementations are not affected by the iterator.
C2H_TEST("DeviceHistogram::HistogramEven sample iterator", "[histogram_even][device]")
{
  using sample_t                 = int;
  const auto width               = 100;
  const auto padding             = 13; // in elements
  const auto height              = 30;
  constexpr auto channels        = 4;
  constexpr auto active_channels = 3;
  const auto row_pitch           = (width + padding) * channels * static_cast<int>(sizeof(sample_t));
  const auto total_values        = (width + padding) * channels * height;

  const auto num_levels  = array<int, active_channels>{11, 3, 2};
  const auto lower_level = caller_vector<int>{0, -10, cs::numeric_limits<int>::lowest()};
  const auto upper_level = caller_vector<int>{total_values, 10, cs::numeric_limits<int>::max()};

  auto sample_iterator = hipcub::CountingInputIterator<sample_t>(0);

  // Channel #0: 0, 4,  8, 12
  // Channel #1: 1, 5,  9, 13
  // Channel #2: 2, 6, 10, 14
  // unused:     3, 7, 11, 15

  auto d_histogram = array<c2h::device_vector<int>, active_channels>();
  for (int c = 0; c < active_channels; ++c)
  {
    d_histogram[c].resize(num_levels[c] - 1);
  }

  multi_histogram_even<channels, active_channels>(
    sample_iterator,
    thrust::raw_pointer_cast(to_caller_vector_of_ptrs(d_histogram).data()),
    thrust::raw_pointer_cast(caller_vector<int>(num_levels.begin(), num_levels.end()).data()),
    thrust::raw_pointer_cast(lower_level.data()),
    thrust::raw_pointer_cast(upper_level.data()),
    width,
    height,
    row_pitch);

  CHECK(d_histogram[0] == c2h::host_vector<int>(10, (width * height) / 10));
  CHECK(d_histogram[1] == c2h::host_vector<int>{0, 3});
  CHECK(d_histogram[2] == c2h::host_vector<int>{width * height});
}

// Regression: https://github.com/NVIDIA/cub/issues/479
C2H_TEST("DeviceHistogram::Histogram* regression NVIDIA/cub#479", "[histogram][device]")
{
  test_even_and_range<float, 4, 3, int>(12, 7, 1920, 1080);
}

C2H_TEST("DeviceHistogram::Histogram* down-conversion size_t to int", "[histogram][device]")
{
  _CCCL_IF_CONSTEXPR (sizeof(size_t) != sizeof(int))
  {
    using offset_t = cs::make_signed<size_t>::type;
    test_even_and_range<unsigned char, 4, 3, int>(256, 256 + 1, offset_t{1920}, offset_t{1080});
  }
}

C2H_TEST("DeviceHistogram::HistogramRange levels/samples aliasing", "[histogram_range][device]")
{
  constexpr int num_levels = 7;
  constexpr int h_samples[]{
    0,  2,  4,  6,  8,  10, 12, // levels
    1, // bin 0
    3,  3, // bin 1
    5,  5,  5, // bin 2
    7,  7,  7,  7, // bin 3
    9,  9,  9,  9,  9, // bin 4
    11, 11, 11, 11, 11, 11 // bin 5
  };

  auto d_histogram = c2h::device_vector<int>(num_levels - 1);
  auto d_samples   = c2h::device_vector<int>(cs::begin(h_samples), cs::end(h_samples));
  histogram_range(
    thrust::raw_pointer_cast(d_samples.data()),
    thrust::raw_pointer_cast(d_histogram.data()),
    num_levels,
    thrust::raw_pointer_cast(d_samples.data()), // Alias levels with samples (fancy way to `d_histogram[bin]++`).
    static_cast<int>(d_samples.size()));

  auto h_histogram = c2h::host_vector<int>(d_histogram);
  for (int bin = 0; bin < num_levels - 1; bin++)
  {
    // Each bin should contain `bin + 1` samples, plus one extra, since samples also contain levels.
    CHECK(h_histogram[bin] == bin + 2);
  }
}

// We cannot use launch wrappers for this test, since it checks error codes explicitly.
#if TEST_LAUNCH == 0
// Our bin computation for HistogramEven is guaranteed only for when (max_level - min_level) * num_bins does not
// overflow using uint64_t arithmetic. In case of overflow, we expect hipErrorInvalidValue to be returned.
C2H_TEST_LIST("DeviceHistogram::HistogramEven bin computation does not overflow",
              "[histogram_even][device]",
              uint8_t,
              uint16_t,
              uint32_t,
              uint64_t)
{
  using sample_t                 = TestType;
  using counter_t                = uint32_t;
  constexpr sample_t lower_level = 0;
  constexpr sample_t upper_level = cs::numeric_limits<sample_t>::max();
  constexpr auto num_samples     = 1000;
  auto d_samples                 = hipcub::CountingInputIterator<sample_t>{0UL};
  auto d_histo_out               = c2h::device_vector<counter_t>(1024);
  const auto num_bins            = GENERATE(1, 2);

  // Verify we always initializes temp_storage_bytes
  constexpr size_t canary_bytes = 3;
  size_t temp_storage_bytes     = canary_bytes;
  const auto error1             = hipcub::DeviceHistogram::HistogramEven(
    nullptr,
    temp_storage_bytes,
    d_samples,
    raw_pointer_cast(d_histo_out.data()),
    num_bins + 1,
    lower_level,
    upper_level,
    num_samples);
  // CHECK(error1 == ???); // TODO(bgruber): add a new check? what is expected? It's neither 0 or 1.
  std::ignore = error1;
  CHECK(temp_storage_bytes != canary_bytes);

  auto temp_storage = c2h::device_vector<char>(temp_storage_bytes);
  const auto error2 = hipcub::DeviceHistogram::HistogramEven(
    raw_pointer_cast(temp_storage.data()),
    temp_storage_bytes,
    d_samples,
    raw_pointer_cast(d_histo_out.data()),
    num_bins + 1,
    lower_level,
    upper_level,
    num_samples);

  // Since test #1 is just a single bin, we expect it to succeed
  // Since we promote up to 64-bit integer arithmetic we expect tests to not overflow for types of
  // up to 4 bytes. For 64-bit and wider types, we do not perform further promotion to even wider
  // types, hence we expect hipErrorInvalidValue to be returned to indicate of a potential overflow
  // Ensure we do not return an error on querying temporary storage requirements
  CHECK(error2 == (num_bins == 1 || sizeof(sample_t) <= 4UL ? hipSuccess : hipErrorInvalidValue));
}
#endif // TEST_LAUNCH == 0

// Regression test for https://github.com/NVIDIA/cub/issues/489: integer rounding errors lead to incorrect bin detection
C2H_TEST("DeviceHistogram::HistogramEven bin calculation regression", "[histogram_even][device]")
{
  constexpr int num_levels   = 8;
  const auto h_histogram_ref = c2h::host_vector<int>{1, 5, 0, 2, 1, 0, 0};
  const auto d_samples       = c2h::device_vector<int>{2, 6, 7, 2, 3, 0, 2, 2, 6, 999};
  constexpr int lower_level  = 0;
  constexpr int upper_level  = 12;

  auto d_histogram = c2h::device_vector<int>(h_histogram_ref.size());
  histogram_even(
    thrust::raw_pointer_cast(d_samples.data()),
    thrust::raw_pointer_cast(d_histogram.data()),
    num_levels,
    lower_level,
    upper_level,
    static_cast<int>(d_samples.size()));
  CHECK(h_histogram_ref == d_histogram);
}
