#define CUB_DETAIL_BEFORE_NVTX_RANGE_SCOPE(name) static_assert(false, "");
#define CCCL_DISABLE_NVTX

#include <cub/device/device_for.cuh>

#include <thrust/iterator/counting_iterator.h>

#include <cuda/std/functional>

#if defined(CCCL_DISABLE_NVTX) && defined(NVTX_VERSION)
#  error "NVTX was included somewhere even though it is turned off via CCCL_DISABLE_NVTX"
#endif // defined(CCCL_DISABLE_NVTX) && defined(NVTX_VERSION)

int main()
{
  thrust::counting_iterator<int> it{0};
  hipcub::DeviceFor::ForEach(it, it + 16, ::cuda::std::negate<int>{});
  hipDeviceSynchronize();
}
