/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "insert_nested_NVTX_range_guard.h"
// above header needs to be included first

#include <cub/device/device_reduce.cuh>

#include <thrust/iterator/constant_iterator.h>

#include <cstdint>

#include "catch2_test_device_reduce.cuh"
#include "catch2_test_launch_helper.h"
#include <c2h/catch2_test_helper.cuh>
#include <c2h/custom_type.cuh>

DECLARE_LAUNCH_WRAPPER(hipcub::DeviceReduce::ReduceByKey, device_reduce_by_key);

// %PARAM% TEST_LAUNCH lid 0:1:2

// List of types to test
using custom_t           = c2h::custom_type_t<c2h::accumulateable_t, c2h::equal_comparable_t>;
using iterator_type_list = c2h::type_list<type_triple<custom_t>, type_triple<std::int64_t, std::int64_t, custom_t>>;

C2H_TEST("Device reduce-by-key works with iterators", "[by_key][reduce][device]", iterator_type_list)
{
  using params   = params_t<TestType>;
  using value_t  = typename params::item_t;
  using output_t = typename params::output_t;
  using key_t    = typename params::type_pair_t::key_t;
  using offset_t = uint32_t;

  constexpr offset_t min_items = 1;
  constexpr offset_t max_items = 1000000;

  // Number of items
  const offset_t num_items = GENERATE_COPY(
    take(2, random(min_items, max_items)),
    values({
      min_items,
      max_items,
    }));
  INFO("Test num_items: " << num_items);

  // Range of segment sizes to generate (a segment is a series of consecutive equal keys)
  const std::tuple<offset_t, offset_t> seg_size_range =
    GENERATE_COPY(table<offset_t, offset_t>({{1, 1}, {1, num_items}, {num_items, num_items}}));
  INFO("Test seg_size_range: [" << std::get<0>(seg_size_range) << ", " << std::get<1>(seg_size_range) << "]");

  // Generate input segments
  c2h::device_vector<offset_t> segment_offsets = c2h::gen_uniform_offsets<offset_t>(
    C2H_SEED(1), num_items, std::get<0>(seg_size_range), std::get<1>(seg_size_range));

  // Get array of keys from segment offsets
  const offset_t num_segments = static_cast<offset_t>(segment_offsets.size() - 1);
  c2h::device_vector<key_t> segment_keys(num_items);
  c2h::init_key_segments(segment_offsets, segment_keys);
  auto d_keys_it = segment_keys.cbegin();

  // Prepare input data
  value_t default_constant{};
  init_default_constant(default_constant);
  auto value_it = thrust::make_constant_iterator(default_constant);

  using op_t = hipcub::Sum;

  // Prepare verification data
  using accum_t = ::cuda::std::__accumulator_t<op_t, value_t, output_t>;
  c2h::host_vector<output_t> expected_result(num_segments);
  compute_segmented_problem_reference(value_it, segment_offsets, op_t{}, accum_t{}, expected_result.begin());
  c2h::host_vector<key_t> expected_keys = compute_unique_keys_reference(segment_keys);

  // Run test
  c2h::device_vector<offset_t> num_unique_keys(1);
  c2h::device_vector<key_t> out_unique_keys(num_segments);
  c2h::device_vector<output_t> out_result(num_segments);
  auto d_result_out_it = thrust::raw_pointer_cast(out_result.data());
  auto d_keys_out_it   = out_unique_keys.begin();
  device_reduce_by_key(
    d_keys_it,
    d_keys_out_it,
    value_it,
    d_result_out_it,
    thrust::raw_pointer_cast(num_unique_keys.data()),
    op_t{},
    num_items);

  // Verify result
  REQUIRE(expected_result == out_result);
}
