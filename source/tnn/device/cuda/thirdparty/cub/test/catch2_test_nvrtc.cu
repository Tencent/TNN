#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <hip/hip_runtime.h>

#include <string>

#include <c2h/catch2_test_helper.cuh>
#include <hip/hiprtc.h>

TEST_CASE("Test nvrtc", "[test][nvrtc]")
{
  hiprtcProgram prog{};

  const char* src =
    "#include <cub/warp/warp_reduce.cuh>                                                         \n"
    "#include <cub/warp/warp_scan.cuh>                                                           \n"
    "#include <cub/warp/warp_exchange.cuh>                                                       \n"
    "#include <cub/warp/warp_load.cuh>                                                           \n"
    "#include <cub/warp/warp_store.cuh>                                                          \n"
    "#include <cub/warp/warp_merge_sort.cuh>                                                     \n"
    "#include <cub/block/block_adjacent_difference.cuh>                                          \n"
    "#include <cub/block/block_discontinuity.cuh>                                                \n"
    "#include <cub/block/block_exchange.cuh>                                                     \n"
    "#include <cub/block/block_histogram.cuh>                                                    \n"
    "#include <cub/block/block_load.cuh>                                                         \n"
    "#include <cub/block/block_store.cuh>                                                        \n"
    "#include <cub/block/block_merge_sort.cuh>                                                   \n"
    "#include <cub/block/block_radix_rank.cuh>                                                   \n"
    "#include <cub/block/block_radix_sort.cuh>                                                   \n"
    "#include <cub/block/block_reduce.cuh>                                                       \n"
    "#include <cub/block/block_scan.cuh>                                                         \n"
    "#include <cub/device/dispatch/kernels/reduce.cuh>                                           \n"
    "#include <cub/device/dispatch/kernels/for_each.cuh>                                         \n"
    "                                                                                            \n"
    "extern \"C\" __global__ void kernel(int *ptr, int *errors)                                  \n"
    "{                                                                                           \n"
    "  constexpr int items_per_thread = 4;                                                       \n"
    "  constexpr int threads_per_block = 128;                                                    \n"
    "  using warp_load_t = hipcub::WarpLoad<int, items_per_thread>;                                 \n"
    "  using warp_load_storage_t = warp_load_t::TempStorage;                                     \n"
    "                                                                                            \n"
    "  using warp_exchange_t = hipcub::WarpExchange<int, items_per_thread>;                         \n"
    "  using warp_exchange_storage_t = warp_exchange_t::TempStorage;                             \n"
    "                                                                                            \n"
    "  using warp_reduce_t = hipcub::WarpReduce<int>;                                               \n"
    "  using warp_reduce_storage_t = warp_reduce_t::TempStorage;                                 \n"
    "                                                                                            \n"
    "  using warp_merge_sort_t = hipcub::WarpMergeSort<int, items_per_thread>;                      \n"
    "  using warp_merge_sort_storage_t = warp_merge_sort_t::TempStorage;                         \n"
    "                                                                                            \n"
    "  using warp_scan_t = hipcub::WarpScan<int>;                                                   \n"
    "  using warp_scan_storage_t = warp_scan_t::TempStorage;                                     \n"
    "                                                                                            \n"
    "  using warp_store_t = hipcub::WarpStore<int, items_per_thread>;                               \n"
    "  using warp_store_storage_t = warp_store_t::TempStorage;                                   \n"
    "                                                                                            \n"
    "  __shared__ warp_load_storage_t warp_load_storage;                                         \n"
    "  __shared__ warp_exchange_storage_t warp_exchange_storage;                                 \n"
    "  __shared__ warp_reduce_storage_t warp_reduce_storage;                                     \n"
    "  __shared__ warp_merge_sort_storage_t warp_merge_sort_storage;                             \n"
    "  __shared__ warp_scan_storage_t warp_scan_storage;                                         \n"
    "  __shared__ warp_store_storage_t warp_store_storage;                                       \n"
    "                                                                                            \n"
    "  int items[items_per_thread];                                                              \n"
    "  if (threadIdx.x < 32)                                                                     \n"
    "  {                                                                                         \n"
    "    // Test warp load                                                                       \n"
    "    warp_load_t(warp_load_storage).Load(ptr, items);                                        \n"
    "                                                                                            \n"
    "    for (int i = 0; i < items_per_thread; i++)                                              \n"
    "    {                                                                                       \n"
    "      if (items[i] != (i + threadIdx.x * items_per_thread))                                 \n"
    "      {                                                                                     \n"
    "        atomicAdd(errors, 1);                                                               \n"
    "      }                                                                                     \n"
    "    }                                                                                       \n"
    "                                                                                            \n"
    "    // Test warp exchange                                                                   \n"
    "    warp_exchange_t(warp_exchange_storage).BlockedToStriped(items, items);                  \n"
    "                                                                                            \n"
    "    for (int i = 0; i < items_per_thread; i++)                                              \n"
    "    {                                                                                       \n"
    "      if (items[i] != (i * 32 + threadIdx.x))                                               \n"
    "      {                                                                                     \n"
    "        atomicAdd(errors, 1);                                                               \n"
    "      }                                                                                     \n"
    "    }                                                                                       \n"
    "                                                                                            \n"
    "    // Test warp reduce                                                                     \n"
    "    const int sum = warp_reduce_t(warp_reduce_storage).Sum(items[0]);                       \n"
    "    if (threadIdx.x == 0)                                                                   \n"
    "    {                                                                                       \n"
    "      if (sum != (32 * (32 - 1) / 2))                                                       \n"
    "      {                                                                                     \n"
    "        atomicAdd(errors, 1);                                                               \n"
    "      }                                                                                     \n"
    "    }                                                                                       \n"
    "                                                                                            \n"
    "    // Test warp scan                                                                       \n"
    "    int prefix_sum{};                                                                       \n"
    "    warp_scan_t(warp_scan_storage).InclusiveSum(items[0], prefix_sum);                      \n"
    "    if (prefix_sum != (threadIdx.x * (threadIdx.x + 1) / 2))                                \n"
    "    {                                                                                       \n"
    "      atomicAdd(errors, 1);                                                                 \n"
    "    }                                                                                       \n"
    "                                                                                            \n"
    "    // Test warp merge sort                                                                 \n"
    "    warp_merge_sort_t(warp_merge_sort_storage).Sort(                                        \n"
    "      items,                                                                                \n"
    "      [](int a, int b) { return a < b; });                                                  \n"
    "                                                                                            \n"
    "    for (int i = 0; i < items_per_thread; i++)                                              \n"
    "    {                                                                                       \n"
    "      if (items[i] != (i + threadIdx.x * items_per_thread))                                 \n"
    "      {                                                                                     \n"
    "        atomicAdd(errors, 1);                                                               \n"
    "      }                                                                                     \n"
    "    }                                                                                       \n"
    "                                                                                            \n"
    "    // Test warp store                                                                      \n"
    "    warp_store_t(warp_store_storage).Store(ptr, items);                                     \n"
    "  }                                                                                         \n"
    "  __syncthreads();                                                                          \n"
    "                                                                                            \n"
    "  using block_load_t = hipcub::BlockLoad<int, threads_per_block, items_per_thread>;            \n"
    "  using block_load_storage_t = block_load_t::TempStorage;                                   \n"
    "                                                                                            \n"
    "  using block_exchange_t = hipcub::BlockExchange<int, threads_per_block, items_per_thread>;    \n"
    "  using block_exchange_storage_t = block_exchange_t::TempStorage;                           \n"
    "                                                                                            \n"
    "  using block_reduce_t = hipcub::BlockReduce<int, threads_per_block>;                          \n"
    "  using block_reduce_storage_t = block_reduce_t::TempStorage;                               \n"
    "                                                                                            \n"
    "  using block_scan_t = hipcub::BlockScan<int, threads_per_block>;                              \n"
    "  using block_scan_storage_t = block_scan_t::TempStorage;                                   \n"
    "                                                                                            \n"
    "  using block_radix_sort_t = hipcub::BlockRadixSort<int, threads_per_block, items_per_thread>; \n"
    "  using block_radix_sort_storage_t = block_radix_sort_t::TempStorage;                       \n"
    "                                                                                            \n"
    "  using block_store_t = hipcub::BlockStore<int, threads_per_block, items_per_thread>;          \n"
    "  using block_store_storage_t = block_store_t::TempStorage;                                 \n"
    "                                                                                            \n"
    "  __shared__ block_load_storage_t block_load_storage;                                       \n"
    "  __shared__ block_exchange_storage_t block_exchange_storage;                               \n"
    "  __shared__ block_reduce_storage_t block_reduce_storage;                                   \n"
    "  __shared__ block_scan_storage_t block_scan_storage;                                       \n"
    "  __shared__ block_radix_sort_storage_t block_radix_sort_storage;                           \n"
    "  __shared__ block_store_storage_t block_store_storage;                                     \n"
    "                                                                                            \n"
    "  // Test block load                                                                        \n"
    "  block_load_t(block_load_storage).Load(ptr, items);                                        \n"
    "                                                                                            \n"
    "  for (int i = 0; i < items_per_thread; i++)                                                \n"
    "  {                                                                                         \n"
    "    if (items[i] != (i + threadIdx.x * items_per_thread))                                   \n"
    "    {                                                                                       \n"
    "      atomicAdd(errors, 1);                                                                 \n"
    "    }                                                                                       \n"
    "  }                                                                                         \n"
    "                                                                                            \n"
    "  // Test block exchange                                                                    \n"
    "  block_exchange_t(block_exchange_storage).BlockedToStriped(items, items);                  \n"
    "                                                                                            \n"
    "  for (int i = 0; i < items_per_thread; i++)                                                \n"
    "  {                                                                                         \n"
    "    if (items[i] != (i * threads_per_block + threadIdx.x))                                  \n"
    "    {                                                                                       \n"
    "      atomicAdd(errors, 1);                                                                 \n"
    "    }                                                                                       \n"
    "  }                                                                                         \n"
    "                                                                                            \n"
    "  // Test block reduce                                                                      \n"
    "  const int sum = block_reduce_t(block_reduce_storage).Sum(items[0]);                       \n"
    "  if (threadIdx.x == 0)                                                                     \n"
    "  {                                                                                         \n"
    "    if (sum != (threads_per_block * (threads_per_block - 1) / 2))                           \n"
    "    {                                                                                       \n"
    "      atomicAdd(errors, 1);                                                                 \n"
    "    }                                                                                       \n"
    "  }                                                                                         \n"
    "                                                                                            \n"
    "  // Test block scan                                                                        \n"
    "  int prefix_sum{};                                                                         \n"
    "  block_scan_t(block_scan_storage).InclusiveSum(items[0], prefix_sum);                      \n"
    "  if (prefix_sum != (threadIdx.x * (threadIdx.x + 1) / 2))                                  \n"
    "  {                                                                                         \n"
    "    atomicAdd(errors, 1);                                                                   \n"
    "  }                                                                                         \n"
    "                                                                                            \n"
    "  // Test block radix sort                                                                  \n"
    "  block_radix_sort_t(block_radix_sort_storage).SortDescending(items);                       \n"
    "                                                                                            \n"
    "  // Test block store                                                                       \n"
    "  block_store_t(block_store_storage).Store(ptr, items);                                     \n"
    "}                                                                                           \n";

  const char* name = "test";

  REQUIRE(HIPRTC_SUCCESS == hiprtcCreateProgram(&prog, src, name, 0, nullptr, nullptr));

  int ptx_version{};
  hipcub::PtxVersion(ptx_version);
  const std::string arch = std::string("-arch=sm_") + std::to_string(ptx_version / 10);
  const std::string std  = std::string("-std=c++") + std::to_string(_CCCL_STD_VER - 2000);

  constexpr int num_includes         = 6;
  const char* includes[num_includes] = {
    NVRTC_CUB_PATH, NVRTC_THRUST_PATH, NVRTC_LIBCUDACXX_PATH, NVRTC_CTK_PATH, arch.c_str(), std.c_str()};

  std::size_t log_size{};
  hiprtcResult compile_result = hiprtcCompileProgram(prog, num_includes, includes);

  REQUIRE(HIPRTC_SUCCESS == hiprtcGetProgramLogSize(prog, &log_size));

  std::unique_ptr<char[]> log{new char[log_size]};
  REQUIRE(HIPRTC_SUCCESS == hiprtcGetProgramLog(prog, log.get()));
  INFO("nvrtc log = " << log.get());
  REQUIRE(HIPRTC_SUCCESS == compile_result);

  std::size_t code_size{};
  REQUIRE(HIPRTC_SUCCESS == hiprtcGetBitcodeSize(prog, &code_size));

  std::unique_ptr<char[]> code{new char[code_size]};
  REQUIRE(HIPRTC_SUCCESS == hiprtcGetBitcode(prog, code.get()));
  REQUIRE(HIPRTC_SUCCESS == hiprtcDestroyProgram(&prog));

  hipCtx_t context{};
  hipDevice_t device{};
  hipModule_t module{};
  hipFunction_t kernel{};

  REQUIRE(hipSuccess == hipInit(0));
  REQUIRE(hipSuccess == hipDeviceGet(&device, 0));
  REQUIRE(hipSuccess == hipCtxCreate(&context, 0, device));
  REQUIRE(hipSuccess == hipModuleLoadDataEx(&module, code.get(), 0, 0, 0));
  REQUIRE(hipSuccess == hipModuleGetFunction(&kernel, module, "kernel"));

  // Generate input for execution, and create output buffers.
  constexpr int threads_in_block = 128;
  constexpr int items_per_thread = 4;
  constexpr int tile_size        = threads_in_block * items_per_thread;

  hipDeviceptr_t d_ptr{};
  REQUIRE(hipSuccess == hipMalloc(&d_ptr, tile_size * sizeof(int)));

  hipDeviceptr_t d_err{};
  REQUIRE(hipSuccess == hipMalloc(&d_err, sizeof(int)));

  int h_ptr[tile_size];
  for (int i = 0; i < tile_size; i++)
  {
    h_ptr[i] = i;
  }
  REQUIRE(hipSuccess == hipMemcpyHtoD(d_ptr, h_ptr, tile_size * sizeof(int)));

  int h_err{0};
  REQUIRE(hipSuccess == hipMemcpyHtoD(d_err, &h_err, sizeof(int)));

  void* args[] = {&d_ptr, &d_err};

  REQUIRE(hipSuccess == hipModuleLaunchKernel(kernel, 1, 1, 1, threads_in_block, 1, 1, 0, nullptr, args, 0));
  REQUIRE(hipSuccess == hipCtxSynchronize());
  REQUIRE(hipSuccess == hipMemcpyDtoH(h_ptr, d_ptr, tile_size * sizeof(int)));
  REQUIRE(hipSuccess == hipMemcpyDtoH(&h_err, d_err, sizeof(int)));

  REQUIRE(h_err == 0);
  for (int i = 0; i < tile_size; i++)
  {
    const int actual   = h_ptr[i];
    const int expected = tile_size - i - 1;
    REQUIRE(actual == expected);
  }

  REQUIRE(hipSuccess == hipFree(d_ptr));
  REQUIRE(hipSuccess == hipFree(d_err));
  REQUIRE(hipSuccess == hipModuleUnload(module));
  REQUIRE(hipSuccess == hipCtxDestroy(context));
}
