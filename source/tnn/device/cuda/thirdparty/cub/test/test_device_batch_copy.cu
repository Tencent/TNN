#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_copy.cuh>
#include <cub/util_ptx.cuh>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>

#include <algorithm>
#include <cstdint>
#include <limits>
#include <numeric>
#include <random>
#include <string>
#include <type_traits>
#include <utility>
#include <vector>

#include "test_util.h"
#include <c2h/vector.cuh>

/**
 * @brief Host-side random data generation
 */
template <typename T>
void GenerateRandomData(
  T* rand_out,
  const std::size_t num_items,
  const T min_rand_val                                                           = std::numeric_limits<T>::min(),
  const T max_rand_val                                                           = std::numeric_limits<T>::max(),
  const std::uint_fast32_t seed                                                  = 320981U,
  typename std::enable_if<std::is_integral<T>::value && (sizeof(T) >= 2)>::type* = nullptr)
{
  // initialize random number generator
  std::mt19937 rng(seed);
  std::uniform_int_distribution<T> uni_dist(min_rand_val, max_rand_val);

  // generate random numbers
  for (std::size_t i = 0; i < num_items; ++i)
  {
    rand_out[i] = uni_dist(rng);
  }
}

/**
 * @brief Used for generating a shuffled but cohesive sequence of output-range offsets for the
 * sequence of input-ranges.
 */
template <typename RangeOffsetT, typename ByteOffsetT, typename RangeSizeT>
c2h::host_vector<ByteOffsetT>
GetShuffledRangeOffsets(const c2h::host_vector<RangeSizeT>& range_sizes, const std::uint_fast32_t seed = 320981U)
{
  RangeOffsetT num_ranges = static_cast<RangeOffsetT>(range_sizes.size());

  // We're remapping the i-th range to pmt_idxs[i]
  std::mt19937 rng(seed);
  c2h::host_vector<RangeOffsetT> pmt_idxs(num_ranges);
  std::iota(pmt_idxs.begin(), pmt_idxs.end(), static_cast<RangeOffsetT>(0));
  std::shuffle(std::begin(pmt_idxs), std::end(pmt_idxs), rng);

  // Compute the offsets using the new mapping
  ByteOffsetT running_offset = {};
  c2h::host_vector<ByteOffsetT> permuted_offsets;
  permuted_offsets.reserve(num_ranges);
  for (auto permuted_range_idx : pmt_idxs)
  {
    permuted_offsets.push_back(running_offset);
    running_offset += range_sizes[permuted_range_idx];
  }

  // Generate the scatter indexes that identify where each range was mapped to
  c2h::host_vector<RangeOffsetT> scatter_idxs(num_ranges);
  for (RangeOffsetT i = 0; i < num_ranges; i++)
  {
    scatter_idxs[pmt_idxs[i]] = i;
  }

  c2h::host_vector<ByteOffsetT> new_offsets(num_ranges);
  for (RangeOffsetT i = 0; i < num_ranges; i++)
  {
    new_offsets[i] = permuted_offsets[scatter_idxs[i]];
  }

  return new_offsets;
}

template <size_t n, typename... T>
typename std::enable_if<n >= thrust::tuple_size<thrust::tuple<T...>>::value>::type
print_tuple(std::ostream&, const thrust::tuple<T...>&)
{}

template <size_t n, typename... T>
typename std::enable_if<n + 1 <= thrust::tuple_size<thrust::tuple<T...>>::value>::type
print_tuple(std::ostream& os, const thrust::tuple<T...>& tup)
{
  _CCCL_IF_CONSTEXPR (n != 0)
  {
    os << ", ";
  }
  os << thrust::get<n>(tup);
  print_tuple<n + 1>(os, tup);
}

template <typename... T>
std::ostream& operator<<(std::ostream& os, const thrust::tuple<T...>& tup)
{
  os << "[";
  print_tuple<0>(os, tup);
  return os << "]";
}

struct Identity
{
  template <typename T>
  __host__ __device__ __forceinline__ T operator()(T x)
  {
    return x;
  }
};

/**
 * @brief Function object class template that takes an offset and returns an iterator at the given
 * offset relative to a fixed base iterator.
 *
 * @tparam IteratorT The random-access iterator type to be returned
 */
template <typename IteratorT>
struct OffsetToIteratorOp
{
  template <typename OffsetT>
  __host__ __device__ __forceinline__ thrust::transform_output_iterator<Identity, IteratorT>
  operator()(OffsetT offset) const
  {
    return thrust::make_transform_output_iterator(base_it + offset, Identity{});
  }
  IteratorT base_it;
};

template <typename AtomicT>
struct RepeatIndex
{
  template <typename OffsetT>
  __host__ __device__ __forceinline__ thrust::constant_iterator<AtomicT> operator()(OffsetT i)
  {
    return thrust::constant_iterator<AtomicT>(static_cast<AtomicT>(i));
  }
};

enum class TestDataGen
{
  // Random offsets into a data segment
  RANDOM,

  // Ranges cohesively reside next to each other
  CONSECUTIVE
};

std::string TestDataGenToString(TestDataGen gen)
{
  switch (gen)
  {
    case TestDataGen::RANDOM:
      return "TestDataGen::RANDOM";
    case TestDataGen::CONSECUTIVE:
      return "TestDataGen::CONSECUTIVE";
    default:
      return "Unknown";
  }
}

/**
 * @brief
 *
 * @tparam AtomicT The type of the elements being copied
 * @tparam RangeOffsetT Type used for indexing into the array of ranges
 * @tparam RangeSizeT Type used for indexing into individual elements of a range (large enough to
 * cover the max range size)
 * @tparam ByteOffsetT Type used for indexing into elements over *all* the ranges' sizes
 */
template <typename AtomicT, typename RangeOffsetT, typename RangeSizeT, typename ByteOffsetT>
void RunTest(RangeOffsetT num_ranges, RangeSizeT min_range_size, RangeSizeT max_range_size, TestDataGen output_gen)
try
{
  // Range segment data (their offsets and sizes)
  c2h::host_vector<RangeSizeT> h_range_sizes(num_ranges);
  thrust::counting_iterator<RangeOffsetT> iota(0);
  auto d_range_srcs = thrust::make_transform_iterator(iota, RepeatIndex<AtomicT>{});
  c2h::host_vector<ByteOffsetT> h_offsets(num_ranges + 1);

  // Generate the range sizes
  GenerateRandomData(h_range_sizes.data(), h_range_sizes.size(), min_range_size, max_range_size);

  // Compute the total bytes to be copied
  std::partial_sum(h_range_sizes.begin(), h_range_sizes.end(), h_offsets.begin() + 1);
  const ByteOffsetT num_total_items = h_offsets.back();
  h_offsets.pop_back();

  constexpr int32_t shuffle_seed = 123241;

  // Shuffle output range source-offsets
  if (output_gen == TestDataGen::RANDOM)
  {
    h_offsets = GetShuffledRangeOffsets<RangeOffsetT, ByteOffsetT>(h_range_sizes, shuffle_seed);
  }

  // Device-side resources
  c2h::device_vector<AtomicT> d_out(num_total_items);
  c2h::device_vector<ByteOffsetT> d_offsets(h_offsets);
  c2h::device_vector<RangeSizeT> d_range_sizes(h_range_sizes);

  // Prepare d_range_dsts
  using AtomicIterT = typename c2h::device_vector<AtomicT>::iterator;
  OffsetToIteratorOp<AtomicIterT> dst_transform_op{d_out.begin()};
  auto d_range_dsts = thrust::make_transform_iterator(d_offsets.begin(), dst_transform_op);

  // Get temporary storage requirements
  size_t temp_storage_bytes = 0;
  CubDebugExit(hipcub::DeviceCopy::Batched(
    nullptr, temp_storage_bytes, d_range_srcs, d_range_dsts, d_range_sizes.cbegin(), num_ranges));

  c2h::device_vector<std::uint8_t> d_temp_storage(temp_storage_bytes);

  c2h::host_vector<AtomicT> h_out(num_total_items);
  c2h::host_vector<AtomicT> h_gpu_results(num_total_items);

  // Invoke device-side algorithm being under test
  CubDebugExit(hipcub::DeviceCopy::Batched(
    thrust::raw_pointer_cast(d_temp_storage.data()),
    temp_storage_bytes,
    d_range_srcs,
    d_range_dsts,
    d_range_sizes.cbegin(),
    num_ranges));

  // Copy back the output range
  h_gpu_results = d_out;

  // CPU-side result generation for verification
  for (RangeOffsetT i = 0; i < num_ranges; i++)
  {
    std::copy(d_range_srcs[i], d_range_srcs[i] + h_range_sizes[i], h_out.begin() + h_offsets[i]);
  }

  const auto it_pair = std::mismatch(h_gpu_results.cbegin(), h_gpu_results.cend(), h_out.cbegin());

  if (it_pair.first != h_gpu_results.cend())
  {
    std::cout << "Mismatch at index " << std::distance(h_gpu_results.cbegin(), it_pair.first)
              << ", CPU vs. GPU: " << *it_pair.second << ", " << *it_pair.first << "\n";
  }
  AssertEquals(it_pair.first, h_gpu_results.cend());
}
catch (std::bad_alloc& e)
{
  (void) e;
#ifdef DEBUG_CHECKED_ALLOC_FAILURE
  std::cout
    << "Skipping test 'RunTest(" << num_ranges << ", " //
    << min_range_size << ", " //
    << max_range_size << ", " //
    << TestDataGenToString(output_gen) << ")" //
    << "' due to insufficient memory: " << e.what() << "\n";
#endif // DEBUG_CHECKED_ALLOC_FAILURE
}

struct object_with_non_trivial_ctor
{
  static constexpr int MAGIC = 923390;

  int field;
  int magic;

  __host__ __device__ object_with_non_trivial_ctor()
  {
    magic = MAGIC;
    field = 0;
  }
  __host__ __device__ object_with_non_trivial_ctor(int f)
  {
    magic = MAGIC;
    field = f;
  }

  object_with_non_trivial_ctor(const object_with_non_trivial_ctor& x) = default;

  __host__ __device__ object_with_non_trivial_ctor& operator=(const object_with_non_trivial_ctor& x)
  {
    if (magic == MAGIC)
    {
      field = x.field;
    }
    return *this;
  }
};

void nontrivial_constructor_test()
{
  constexpr int num_buffers = 3;
  c2h::device_vector<object_with_non_trivial_ctor> a(num_buffers, object_with_non_trivial_ctor(99));
  c2h::device_vector<object_with_non_trivial_ctor> b(num_buffers);
  using iterator = c2h::device_vector<object_with_non_trivial_ctor>::iterator;

  c2h::device_vector<iterator> a_iter{a.begin(), a.begin() + 1, a.begin() + 2};

  c2h::device_vector<iterator> b_iter{b.begin(), b.begin() + 1, b.begin() + 2};

  auto sizes = thrust::make_constant_iterator(1);

  std::uint8_t* d_temp_storage{};
  std::size_t temp_storage_bytes{};

  hipcub::DeviceCopy::Batched(d_temp_storage, temp_storage_bytes, a_iter.begin(), b_iter.begin(), sizes, num_buffers);

  c2h::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);
  d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

  hipcub::DeviceCopy::Batched(d_temp_storage, temp_storage_bytes, a_iter.begin(), b_iter.begin(), sizes, num_buffers);

  for (int i = 0; i < 10; i++)
  {
    object_with_non_trivial_ctor ha(a[i]);
    object_with_non_trivial_ctor hb(b[i]);
    int ia = ha.field;
    int ib = hb.field;

    if (ia != ib)
    {
      std::cerr << "error: " << ia << " != " << ib << "\n";
    }
  }
}

int main(int argc, char** argv)
{
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  //---------------------------------------------------------------------
  // DeviceCopy::Batched tests
  //---------------------------------------------------------------------
  // Run the nontrivial constructor test suggested by senior-zero
  nontrivial_constructor_test();
  // Type used for indexing into the array of ranges
  using RangeOffsetT = uint32_t;

  // Type used for indexing into individual elements of a range (large enough to cover the max range
  using RangeSizeT = uint32_t;

  // Type used for indexing into bytes over *all* the ranges' sizes
  using ByteOffsetT = uint32_t;

  // Total number of bytes that are targeted to be copied on each run
  constexpr RangeOffsetT target_copy_size = 64U << 20;

  // The number of randomly
  constexpr std::size_t num_rnd_range_tests = 32;

  // Each range's size will be random within this interval
  c2h::host_vector<std::pair<std::size_t, std::size_t>> size_ranges = {
    {0, 1},
    {1, 2},
    {0, 16},
    {1, 32},
    {1, 1024},
    {1, 32 * 1024},
    {128 * 1024, 256 * 1024},
    {target_copy_size, target_copy_size}};

  std::mt19937 rng(0);
  std::uniform_int_distribution<std::size_t> size_dist(1, 1000000);
  for (std::size_t i = 0; i < num_rnd_range_tests; i++)
  {
    auto range_begin = size_dist(rng);
    auto range_end   = size_dist(rng);
    if (range_begin > range_end)
    {
      std::swap(range_begin, range_end);
    }
    size_ranges.push_back({range_begin, range_end});
  }

  for (const auto& size_range : size_ranges)
  {
    // The most granular type being copied.
    using AtomicCopyT         = int64_t;
    RangeSizeT min_range_size = static_cast<RangeSizeT>(CUB_ROUND_UP_NEAREST(size_range.first, sizeof(AtomicCopyT)));
    RangeSizeT max_range_size =
      static_cast<RangeSizeT>(CUB_ROUND_UP_NEAREST(size_range.second, static_cast<RangeSizeT>(sizeof(AtomicCopyT))));
    double average_range_size      = (min_range_size + max_range_size) / 2.0;
    RangeOffsetT target_num_ranges = static_cast<RangeOffsetT>(target_copy_size / average_range_size);

    // Run tests with output ranges being consecutive
    RunTest<AtomicCopyT, RangeOffsetT, RangeSizeT, ByteOffsetT>(
      target_num_ranges, min_range_size, max_range_size, TestDataGen::CONSECUTIVE);

    // Run tests with output ranges being randomly shuffled
    RunTest<AtomicCopyT, RangeOffsetT, RangeSizeT, ByteOffsetT>(
      target_num_ranges, min_range_size, max_range_size, TestDataGen::RANDOM);
  }

  for (const auto& size_range : size_ranges)
  {
    // The most granular type being copied.
    using AtomicCopyT         = thrust::tuple<int64_t, int32_t, int16_t, char, char>;
    RangeSizeT min_range_size = static_cast<RangeSizeT>(CUB_ROUND_UP_NEAREST(size_range.first, sizeof(AtomicCopyT)));
    RangeSizeT max_range_size =
      static_cast<RangeSizeT>(CUB_ROUND_UP_NEAREST(size_range.second, static_cast<RangeSizeT>(sizeof(AtomicCopyT))));
    double average_range_size      = (min_range_size + max_range_size) / 2.0;
    RangeOffsetT target_num_ranges = static_cast<RangeOffsetT>(target_copy_size / average_range_size);

    // Run tests with output ranges being consecutive
    RunTest<AtomicCopyT, RangeOffsetT, RangeSizeT, ByteOffsetT>(
      target_num_ranges, min_range_size, max_range_size, TestDataGen::CONSECUTIVE);

    // Run tests with output ranges being randomly shuffled
    RunTest<AtomicCopyT, RangeOffsetT, RangeSizeT, ByteOffsetT>(
      target_num_ranges, min_range_size, max_range_size, TestDataGen::RANDOM);
  }

  //---------------------------------------------------------------------
  // DeviceCopy::Batched test with 64-bit offsets
  //---------------------------------------------------------------------
  using ByteOffset64T = uint64_t;
  using RangeSize64T  = uint64_t;
  ByteOffset64T large_target_copy_size =
    static_cast<ByteOffset64T>(std::numeric_limits<uint32_t>::max()) + (128ULL * 1024ULL * 1024ULL);
  // Make sure min_range_size is in fact smaller than max range size
  constexpr RangeOffsetT single_range = 1;

  // Run tests with output ranges being consecutive
  RunTest<uint8_t, RangeOffsetT, RangeSize64T, ByteOffset64T>(
    single_range, large_target_copy_size, large_target_copy_size, TestDataGen::CONSECUTIVE);
}
