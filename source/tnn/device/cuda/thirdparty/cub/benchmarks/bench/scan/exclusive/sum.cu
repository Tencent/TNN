/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <nvbench_helper.cuh>

// %RANGE% TUNE_ITEMS ipt 7:24:1
// %RANGE% TUNE_THREADS tpb 128:1024:32
// %RANGE% TUNE_MAGIC_NS ns 0:2048:4
// %RANGE% TUNE_DELAY_CONSTRUCTOR_ID dcid 0:7:1
// %RANGE% TUNE_L2_WRITE_LATENCY_NS l2w 0:1200:5
// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_LOAD ld 0:2:1

using op_t = hipcub::Sum;
#include "base.cuh"
