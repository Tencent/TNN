#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_partition.cuh>

#include <look_back_helper.cuh>
#include <nvbench_helper.cuh>

// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 128:1024:32
// %RANGE% TUNE_MAGIC_NS ns 0:2048:4
// %RANGE% TUNE_DELAY_CONSTRUCTOR_ID dcid 0:7:1
// %RANGE% TUNE_L2_WRITE_LATENCY_NS l2w 0:1200:5

#if !TUNE_BASE
#  if TUNE_TRANSPOSE == 0
#    define TUNE_LOAD_ALGORITHM hipcub::BLOCK_LOAD_DIRECT
#  else // TUNE_TRANSPOSE == 1
#    define TUNE_LOAD_ALGORITHM hipcub::BLOCK_LOAD_WARP_TRANSPOSE
#  endif // TUNE_TRANSPOSE

template <typename InputT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<350, policy_t, policy_t>
  {
    using ThreeWayPartitionPolicy = //
      hipcub::AgentThreeWayPartitionPolicy<TUNE_THREADS_PER_BLOCK,
                                        TUNE_ITEMS_PER_THREAD,
                                        TUNE_LOAD_ALGORITHM,
                                        hipcub::LOAD_DEFAULT,
                                        hipcub::BLOCK_SCAN_WARP_SCANS,
                                        delay_constructor_t>;
  };

  using MaxPolicy = policy_t;
};
#endif // !TUNE_BASE

template <class T>
struct less_then_t
{
  T m_val;

  __device__ bool operator()(const T& val) const
  {
    return val < m_val;
  }
};

template <typename T, typename OffsetT>
void partition(nvbench::state& state, nvbench::type_list<T, OffsetT>)
{
  using input_it_t        = const T*;
  using output_it_t       = T*;
  using num_selected_it_t = OffsetT*;
  using select_op_t       = less_then_t<T>;
  using offset_t          = OffsetT;

#if !TUNE_BASE
  using policy_t   = policy_hub_t<T>;
  using dispatch_t = hipcub::DispatchThreeWayPartitionIf<
    input_it_t,
    output_it_t,
    output_it_t,
    output_it_t,
    num_selected_it_t,
    select_op_t,
    select_op_t,
    offset_t,
    policy_t>;
#else // TUNE_BASE
  using dispatch_t = hipcub::DispatchThreeWayPartitionIf<
    input_it_t,
    output_it_t,
    output_it_t,
    output_it_t,
    num_selected_it_t,
    select_op_t,
    select_op_t,
    offset_t>;
#endif // !TUNE_BASE

  // Retrieve axis parameters
  const auto elements       = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  const bit_entropy entropy = str_to_entropy(state.get_string("Entropy"));

  T min_val{};
  T max_val = std::numeric_limits<T>::max();

  T left_border  = max_val / 3;
  T right_border = left_border * 2;

  select_op_t select_op_1{left_border};
  select_op_t select_op_2{right_border};

  thrust::device_vector<T> in = generate(elements, entropy, min_val, max_val);
  thrust::device_vector<offset_t> num_selected(1);
  thrust::device_vector<T> out_1(elements);
  thrust::device_vector<T> out_2(elements);
  thrust::device_vector<T> out_3(elements);

  input_it_t d_in                  = thrust::raw_pointer_cast(in.data());
  output_it_t d_out_1              = thrust::raw_pointer_cast(out_1.data());
  output_it_t d_out_2              = thrust::raw_pointer_cast(out_2.data());
  output_it_t d_out_3              = thrust::raw_pointer_cast(out_3.data());
  num_selected_it_t d_num_selected = thrust::raw_pointer_cast(num_selected.data());

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<T>(elements);
  state.add_global_memory_writes<offset_t>(1);

  std::size_t temp_size{};
  dispatch_t::Dispatch(
    nullptr, temp_size, d_in, d_out_1, d_out_2, d_out_3, d_num_selected, select_op_1, select_op_2, elements, 0);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(
      temp_storage,
      temp_size,
      d_in,
      d_out_1,
      d_out_2,
      d_out_3,
      d_num_selected,
      select_op_1,
      select_op_2,
      elements,
      launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(partition, NVBENCH_TYPE_AXES(fundamental_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.544", "0.000"});
