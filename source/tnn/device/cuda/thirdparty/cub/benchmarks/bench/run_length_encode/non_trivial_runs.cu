/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_run_length_encode.cuh>

#include <look_back_helper.cuh>
#include <nvbench_helper.cuh>

// %RANGE% TUNE_ITEMS ipt 7:24:1
// %RANGE% TUNE_THREADS tpb 128:1024:32
// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_TIME_SLICING ts 0:1:1
// %RANGE% TUNE_LOAD ld 0:1:1
// %RANGE% TUNE_MAGIC_NS ns 0:2048:4
// %RANGE% TUNE_DELAY_CONSTRUCTOR_ID dcid 0:7:1
// %RANGE% TUNE_L2_WRITE_LATENCY_NS l2w 0:1200:5

#if !TUNE_BASE
#  if TUNE_TRANSPOSE == 0
#    define TUNE_LOAD_ALGORITHM hipcub::BLOCK_LOAD_DIRECT
#  else // TUNE_TRANSPOSE == 1
#    define TUNE_LOAD_ALGORITHM hipcub::BLOCK_LOAD_WARP_TRANSPOSE
#  endif // TUNE_TRANSPOSE

#  if TUNE_LOAD == 0
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_DEFAULT
#  else // TUNE_LOAD == 1
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_CA
#  endif // TUNE_LOAD

struct device_rle_policy_hub
{
  struct Policy350 : hipcub::ChainedPolicy<350, Policy350, Policy350>
  {
    using RleSweepPolicyT =
      hipcub::AgentRlePolicy<TUNE_THREADS,
                          TUNE_ITEMS,
                          TUNE_LOAD_ALGORITHM,
                          TUNE_LOAD_MODIFIER,
                          TUNE_TIME_SLICING,
                          hipcub::BLOCK_SCAN_WARP_SCANS,
                          delay_constructor_t>;
  };

  using MaxPolicy = Policy350;
};
#endif // !TUNE_BASE

template <class T, class OffsetT>
static void rle(nvbench::state& state, nvbench::type_list<T, OffsetT>)
{
  using offset_t                   = OffsetT;
  using keys_input_it_t            = const T*;
  using offset_output_it_t         = offset_t*;
  using length_output_it_t         = offset_t*;
  using num_runs_output_iterator_t = offset_t*;
  using equality_op_t              = hipcub::Equality;
  using accum_t                    = offset_t;

#if !TUNE_BASE
  using dispatch_t =
    hipcub::DeviceRleDispatch<keys_input_it_t,
                           offset_output_it_t,
                           length_output_it_t,
                           num_runs_output_iterator_t,
                           equality_op_t,
                           offset_t,
                           device_rle_policy_hub>;
#else
  using dispatch_t =
    hipcub::DeviceRleDispatch<keys_input_it_t,
                           offset_output_it_t,
                           length_output_it_t,
                           num_runs_output_iterator_t,
                           equality_op_t,
                           offset_t>;
#endif

  const auto elements                    = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  constexpr std::size_t min_segment_size = 1;
  const std::size_t max_segment_size     = static_cast<std::size_t>(state.get_int64("MaxSegSize"));

  thrust::device_vector<offset_t> num_runs_out(1);
  thrust::device_vector<offset_t> out_offsets(elements);
  thrust::device_vector<offset_t> out_lengths(elements);
  thrust::device_vector<T> in_keys = generate.uniform.key_segments(elements, min_segment_size, max_segment_size);

  T* d_in_keys             = thrust::raw_pointer_cast(in_keys.data());
  offset_t* d_out_offsets  = thrust::raw_pointer_cast(out_offsets.data());
  offset_t* d_out_lengths  = thrust::raw_pointer_cast(out_lengths.data());
  offset_t* d_num_runs_out = thrust::raw_pointer_cast(num_runs_out.data());

  std::uint8_t* d_temp_storage{};
  std::size_t temp_storage_bytes{};

  dispatch_t::Dispatch(
    d_temp_storage,
    temp_storage_bytes,
    d_in_keys,
    d_out_offsets,
    d_out_lengths,
    d_num_runs_out,
    equality_op_t{},
    elements,
    0);

  thrust::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);
  d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

  dispatch_t::Dispatch(
    d_temp_storage,
    temp_storage_bytes,
    d_in_keys,
    d_out_offsets,
    d_out_lengths,
    d_num_runs_out,
    equality_op_t{},
    elements,
    0);
  hipDeviceSynchronize();
  const OffsetT num_runs = num_runs_out[0];

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<OffsetT>(num_runs);
  state.add_global_memory_writes<OffsetT>(num_runs);
  state.add_global_memory_writes<OffsetT>(1);

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(
      d_temp_storage,
      temp_storage_bytes,
      d_in_keys,
      d_out_offsets,
      d_out_lengths,
      d_num_runs_out,
      equality_op_t{},
      elements,
      launch.get_stream());
  });
}

using some_offset_types = nvbench::type_list<nvbench::int32_t>;

NVBENCH_BENCH_TYPES(rle, NVBENCH_TYPE_AXES(all_types, some_offset_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_int64_power_of_two_axis("MaxSegSize", {1, 4, 8});
