#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_radix_sort.cuh>
#include <cub/util_arch.cuh>

#include <cuda/std/type_traits>

#include <nvbench_helper.cuh>

// %//RANGE//% TUNE_RADIX_BITS bits 8:9:1
#define TUNE_RADIX_BITS 8

// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 128:1024:32

constexpr bool is_descending   = false;
constexpr bool is_overwrite_ok = false;

#if !TUNE_BASE
template <typename KeyT, typename ValueT, typename OffsetT>
struct policy_hub_t
{
  static constexpr bool KEYS_ONLY = std::is_same<ValueT, hipcub::NullType>::value;

  using DominantT = ::cuda::std::_If<(sizeof(ValueT) > sizeof(KeyT)), ValueT, KeyT>;

  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int ONESWEEP_RADIX_BITS = TUNE_RADIX_BITS;
    static constexpr bool ONESWEEP           = true;
    static constexpr bool OFFSET_64BIT       = sizeof(OffsetT) == 8;

    // Onesweep policy
    using OnesweepPolicy = hipcub::AgentRadixSortOnesweepPolicy<
      TUNE_THREADS_PER_BLOCK,
      TUNE_ITEMS_PER_THREAD,
      DominantT,
      1,
      hipcub::RADIX_RANK_MATCH_EARLY_COUNTS_ANY,
      hipcub::BLOCK_SCAN_RAKING_MEMOIZE,
      hipcub::RADIX_SORT_STORE_DIRECT,
      ONESWEEP_RADIX_BITS>;

    // These kernels are launched once, no point in tuning at the moment
    using HistogramPolicy    = hipcub::AgentRadixSortHistogramPolicy<128, 16, 1, KeyT, ONESWEEP_RADIX_BITS>;
    using ExclusiveSumPolicy = hipcub::AgentRadixSortExclusiveSumPolicy<256, ONESWEEP_RADIX_BITS>;
    using ScanPolicy =
      hipcub::AgentScanPolicy<512,
                           23,
                           OffsetT,
                           hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                           hipcub::LOAD_DEFAULT,
                           hipcub::BLOCK_STORE_WARP_TRANSPOSE,
                           hipcub::BLOCK_SCAN_RAKING_MEMOIZE>;

    // No point in tuning
    static constexpr int SINGLE_TILE_RADIX_BITS = (sizeof(KeyT) > 1) ? 6 : 5;

    // No point in tuning single-tile policy
    using SingleTilePolicy = hipcub::AgentRadixSortDownsweepPolicy<
      256,
      19,
      DominantT,
      hipcub::BLOCK_LOAD_DIRECT,
      hipcub::LOAD_LDG,
      hipcub::RADIX_RANK_MEMOIZE,
      hipcub::BLOCK_SCAN_WARP_SCANS,
      SINGLE_TILE_RADIX_BITS>;
  };

  using MaxPolicy = policy_t;
};

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_onesweep_temp_storage_size()
{
  using portion_offset  = int;
  using onesweep_policy = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::OnesweepPolicy;
  using agent_radix_sort_onesweep_t =
    hipcub::AgentRadixSortOnesweep<onesweep_policy, is_descending, KeyT, ValueT, OffsetT, portion_offset>;

  using hist_policy = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::HistogramPolicy;
  using hist_agent  = hipcub::AgentRadixSortHistogram<hist_policy, is_descending, KeyT, OffsetT>;

  return hipcub::max(sizeof(typename agent_radix_sort_onesweep_t::TempStorage), sizeof(typename hist_agent::TempStorage));
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_temp_storage_size()
{
  using policy_t = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t;

  static_assert(policy_t::ONESWEEP);
  return max_onesweep_temp_storage_size<KeyT, ValueT, OffsetT>();
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr bool fits_in_default_shared_memory()
{
  return max_temp_storage_size<KeyT, ValueT, OffsetT>() < hipcub::detail::max_smem_per_block;
}
#else // TUNE_BASE
template <typename, typename, typename>
constexpr bool fits_in_default_shared_memory()
{
  return true;
}
#endif // TUNE_BASE

template <typename KeyT, typename ValueT, typename OffsetT>
void radix_sort_values(
  std::integral_constant<bool, true>, nvbench::state& state, nvbench::type_list<KeyT, ValueT, OffsetT>)
{
  using offset_t = hipcub::detail::choose_offset_t<OffsetT>;

  using key_t   = KeyT;
  using value_t = ValueT;
#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t, value_t, offset_t>;
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t, policy_t>;
#else // TUNE_BASE
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t>;
#endif // TUNE_BASE

  constexpr int begin_bit = 0;
  constexpr int end_bit   = sizeof(key_t) * 8;

  // Retrieve axis parameters
  const auto elements       = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  const bit_entropy entropy = str_to_entropy(state.get_string("Entropy"));

  thrust::device_vector<key_t> keys_buffer_1     = generate(elements, entropy);
  thrust::device_vector<value_t> values_buffer_1 = generate(elements);
  thrust::device_vector<key_t> keys_buffer_2(elements);
  thrust::device_vector<value_t> values_buffer_2(elements);

  key_t* d_keys_buffer_1     = thrust::raw_pointer_cast(keys_buffer_1.data());
  key_t* d_keys_buffer_2     = thrust::raw_pointer_cast(keys_buffer_2.data());
  value_t* d_values_buffer_1 = thrust::raw_pointer_cast(values_buffer_1.data());
  value_t* d_values_buffer_2 = thrust::raw_pointer_cast(values_buffer_2.data());

  hipcub::DoubleBuffer<key_t> d_keys(d_keys_buffer_1, d_keys_buffer_2);
  hipcub::DoubleBuffer<value_t> d_values(d_values_buffer_1, d_values_buffer_2);

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<KeyT>(elements);
  state.add_global_memory_reads<ValueT>(elements);
  state.add_global_memory_writes<KeyT>(elements);
  state.add_global_memory_writes<ValueT>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::Dispatch(
    nullptr,
    temp_size,
    d_keys,
    d_values,
    static_cast<offset_t>(elements),
    begin_bit,
    end_bit,
    is_overwrite_ok,
    0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    hipcub::DoubleBuffer<key_t> keys     = d_keys;
    hipcub::DoubleBuffer<value_t> values = d_values;

    dispatch_t::Dispatch(
      temp_storage,
      temp_size,
      keys,
      values,
      static_cast<offset_t>(elements),
      begin_bit,
      end_bit,
      is_overwrite_ok,
      launch.get_stream());
  });
}

template <typename KeyT, typename ValueT, typename OffsetT>
void radix_sort_values(std::integral_constant<bool, false>, nvbench::state&, nvbench::type_list<KeyT, ValueT, OffsetT>)
{
  (void) is_descending;
  (void) is_overwrite_ok;
}

template <typename KeyT, typename ValueT, typename OffsetT>
void radix_sort_values(nvbench::state& state, nvbench::type_list<KeyT, ValueT, OffsetT> tl)
{
  using offset_t = hipcub::detail::choose_offset_t<OffsetT>;

  radix_sort_values(std::integral_constant<bool, fits_in_default_shared_memory<KeyT, ValueT, offset_t>()>{}, state, tl);
}

#ifdef TUNE_KeyT
using key_types = nvbench::type_list<TUNE_KeyT>;
#else // !defined(TUNE_KeyT)
using key_types = integral_types;
#endif // TUNE_KeyT

#ifdef TUNE_ValueT
using value_types = nvbench::type_list<TUNE_ValueT>;
#else // !defined(Tune_ValueT)
using value_types =
  nvbench::type_list<int8_t,
                     int16_t,
                     int32_t,
                     int64_t
#  if NVBENCH_HELPER_HAS_I128
                     ,
                     int128_t
#  endif
                     >;
#endif // TUNE_ValueT

NVBENCH_BENCH_TYPES(radix_sort_values, NVBENCH_TYPE_AXES(key_types, value_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"KeyT{ct}", "ValueT{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"});
