// SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: BSD-3-Clause

#include <cub/device/device_select.cuh>

#include <limits>

#include <look_back_helper.cuh>
#include <nvbench_helper.cuh>

// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_LOAD ld 0:1:1
// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK tpb 128:1024:32
// %RANGE% TUNE_MAGIC_NS ns 0:2048:4
// %RANGE% TUNE_DELAY_CONSTRUCTOR_ID dcid 0:7:1
// %RANGE% TUNE_L2_WRITE_LATENCY_NS l2w 0:1200:5

constexpr bool keep_rejects = false;

#if !TUNE_BASE
#  if TUNE_TRANSPOSE == 0
#    define TUNE_LOAD_ALGORITHM hipcub::BLOCK_LOAD_DIRECT
#  else // TUNE_TRANSPOSE == 1
#    define TUNE_LOAD_ALGORITHM hipcub::BLOCK_LOAD_WARP_TRANSPOSE
#  endif // TUNE_TRANSPOSE

#  if TUNE_LOAD == 0
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_DEFAULT
#  else // TUNE_LOAD == 1
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_CA
#  endif // TUNE_LOAD

template <typename InputT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int NOMINAL_4B_ITEMS_PER_THREAD = TUNE_ITEMS_PER_THREAD;

    static constexpr int ITEMS_PER_THREAD =
      CUB_MIN(NOMINAL_4B_ITEMS_PER_THREAD, CUB_MAX(1, (NOMINAL_4B_ITEMS_PER_THREAD * 4 / sizeof(InputT))));

    using SelectIfPolicyT =
      hipcub::AgentSelectIfPolicy<TUNE_THREADS_PER_BLOCK,
                               ITEMS_PER_THREAD,
                               TUNE_LOAD_ALGORITHM,
                               TUNE_LOAD_MODIFIER,
                               hipcub::BLOCK_SCAN_WARP_SCANS,
                               delay_constructor_t>;
  };

  using MaxPolicy = policy_t;
};
#endif // !TUNE_BASE

template <typename T, typename OffsetT, typename InPlaceAlgT>
static void unique(nvbench::state& state, nvbench::type_list<T, OffsetT, InPlaceAlgT>)
{
  using input_it_t         = const T*;
  using flag_it_t          = hipcub::NullType*;
  using output_it_t        = T*;
  using num_selected_it_t  = OffsetT*;
  using select_op_t        = hipcub::NullType;
  using equality_op_t      = hipcub::Equality;
  using offset_t           = OffsetT;
  constexpr bool may_alias = InPlaceAlgT::value;

#if !TUNE_BASE
  using policy_t   = policy_hub_t<T>;
  using dispatch_t = hipcub::DispatchSelectIf<
    input_it_t,
    flag_it_t,
    output_it_t,
    num_selected_it_t,
    select_op_t,
    equality_op_t,
    offset_t,
    keep_rejects,
    may_alias,
    policy_t>;
#else // TUNE_BASE
  using dispatch_t = hipcub::DispatchSelectIf<
    input_it_t,
    flag_it_t,
    output_it_t,
    num_selected_it_t,
    select_op_t,
    equality_op_t,
    offset_t,
    keep_rejects,
    may_alias>;
#endif // TUNE_BASE

  // Retrieve axis parameters
  const auto elements                    = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  constexpr std::size_t min_segment_size = 1;
  const std::size_t max_segment_size     = static_cast<std::size_t>(state.get_int64("MaxSegSize"));

  thrust::device_vector<T> in = generate.uniform.key_segments(elements, min_segment_size, max_segment_size);
  thrust::device_vector<T> out(elements);
  thrust::device_vector<offset_t> num_unique_out(1);

  input_it_t d_in                = thrust::raw_pointer_cast(in.data());
  output_it_t d_out              = thrust::raw_pointer_cast(out.data());
  flag_it_t d_flags              = nullptr;
  num_selected_it_t d_num_unique = thrust::raw_pointer_cast(num_unique_out.data());

  // Get temporary storage requirements
  std::size_t temp_size{};
  dispatch_t::Dispatch(
    nullptr, temp_size, d_in, d_flags, d_out, d_num_unique, select_op_t{}, equality_op_t{}, elements, 0);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  // Get number of unique elements
  dispatch_t::Dispatch(
    temp_storage, temp_size, d_in, d_flags, d_out, d_num_unique, select_op_t{}, equality_op_t{}, elements, 0);

  hipDeviceSynchronize();
  const OffsetT num_unique = num_unique_out[0];

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);
  state.add_global_memory_writes<T>(num_unique);
  state.add_global_memory_writes<offset_t>(1);

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(
      temp_storage,
      temp_size,
      d_in,
      d_flags,
      d_out,
      d_num_unique,
      select_op_t{},
      equality_op_t{},
      elements,
      launch.get_stream());
  });
}

using in_place_alg = nvbench::type_list<::cuda::std::false_type, ::cuda::std::true_type>;

NVBENCH_BENCH_TYPES(unique, NVBENCH_TYPE_AXES(fundamental_types, offset_types, in_place_alg))
  .set_name("base")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}", "IsInPlace{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_int64_power_of_two_axis("MaxSegSize", {1, 4, 8});
