#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_for.cuh>

#include <nvbench_helper.cuh>

template <class T>
struct op_t
{
  int* d_count{};

  __device__ void operator()(T val) const
  {
    if (val == T{})
    {
      atomicAdd(d_count, 1);
    }
  }
};

template <class T, class OffsetT>
void for_each(nvbench::state& state, nvbench::type_list<T, OffsetT>)
{
  using input_it_t  = const T*;
  using output_it_t = int*;
  using offset_t    = OffsetT;

  const auto elements = static_cast<offset_t>(state.get_int64("Elements{io}"));

  thrust::device_vector<T> in(elements, T{42});

  input_it_t d_in   = thrust::raw_pointer_cast(in.data());
  output_it_t d_out = nullptr;

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements);

  op_t<T> op{d_out};

  std::size_t temp_size{};
  hipcub::DeviceFor::ForEachCopyN(nullptr, temp_size, d_in, elements, op);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    hipcub::DeviceFor::ForEachCopyN(temp_storage, temp_size, d_in, elements, op, launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(for_each, NVBENCH_TYPE_AXES(fundamental_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4));
