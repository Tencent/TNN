/******************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/device/device_merge_sort.cuh>

#include <nvbench_helper.cuh>

// %RANGE% TUNE_TRANSPOSE trp 0:1:1
// %RANGE% TUNE_LOAD ld 0:2:1
// %RANGE% TUNE_ITEMS_PER_THREAD ipt 7:24:1
// %RANGE% TUNE_THREADS_PER_BLOCK_POW2 tpb 6:10:1

#ifndef TUNE_BASE
#  define TUNE_THREADS_PER_BLOCK (1 << TUNE_THREADS_PER_BLOCK_POW2)
#endif // TUNE_BASE

using value_t = hipcub::NullType;

#if !TUNE_BASE

#  if TUNE_TRANSPOSE == 0
#    define TUNE_LOAD_ALGORITHM  hipcub::BLOCK_LOAD_DIRECT
#    define TUNE_STORE_ALGORITHM hipcub::BLOCK_STORE_DIRECT
#  else // TUNE_TRANSPOSE == 1
#    define TUNE_LOAD_ALGORITHM  hipcub::BLOCK_LOAD_WARP_TRANSPOSE
#    define TUNE_STORE_ALGORITHM hipcub::BLOCK_STORE_WARP_TRANSPOSE
#  endif // TUNE_TRANSPOSE

#  if TUNE_LOAD == 0
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_DEFAULT
#  elif TUNE_LOAD == 1
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_LDG
#  else // TUNE_LOAD == 2
#    define TUNE_LOAD_MODIFIER hipcub::LOAD_CA
#  endif // TUNE_LOAD

template <typename KeyT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using MergeSortPolicy =
      hipcub::AgentMergeSortPolicy<TUNE_THREADS_PER_BLOCK,
                                hipcub::Nominal4BItemsToItems<KeyT>(TUNE_ITEMS_PER_THREAD),
                                TUNE_LOAD_ALGORITHM,
                                TUNE_LOAD_MODIFIER,
                                TUNE_STORE_ALGORITHM>;
  };

  using MaxPolicy = policy_t;
};
#endif // !TUNE_BASE

template <typename T, typename OffsetT>
void keys(nvbench::state& state, nvbench::type_list<T, OffsetT>)
{
  using key_t            = T;
  using value_t          = hipcub::NullType;
  using key_input_it_t   = key_t*;
  using value_input_it_t = value_t*;
  using key_it_t         = key_t*;
  using value_it_t       = value_t*;
  using offset_t         = OffsetT;
  using compare_op_t     = less_t;

#if !TUNE_BASE
  using policy_t = policy_hub_t<key_t>;
  using dispatch_t =
    hipcub::DispatchMergeSort<key_input_it_t, value_input_it_t, key_it_t, value_it_t, offset_t, compare_op_t, policy_t>;
#else // TUNE_BASE
  using dispatch_t =
    hipcub::DispatchMergeSort<key_input_it_t, value_input_it_t, key_it_t, value_it_t, offset_t, compare_op_t>;
#endif // TUNE_BASE

  // Retrieve axis parameters
  const auto elements       = static_cast<std::size_t>(state.get_int64("Elements{io}"));
  const bit_entropy entropy = str_to_entropy(state.get_string("Entropy"));

  thrust::device_vector<T> buffer_1 = generate(elements, entropy);
  thrust::device_vector<T> buffer_2(elements);

  key_t* d_buffer_1 = thrust::raw_pointer_cast(buffer_1.data());
  key_t* d_buffer_2 = thrust::raw_pointer_cast(buffer_2.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::Dispatch(
    nullptr,
    temp_size,
    d_buffer_1,
    nullptr,
    d_buffer_2,
    nullptr,
    static_cast<offset_t>(elements),
    compare_op_t{},
    0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto* temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec(nvbench::exec_tag::no_batch, [&](nvbench::launch& launch) {
    dispatch_t::Dispatch(
      temp_storage,
      temp_size,
      d_buffer_1,
      nullptr,
      d_buffer_2,
      nullptr,
      static_cast<offset_t>(elements),
      compare_op_t{},
      launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(keys, NVBENCH_TYPE_AXES(all_types, offset_types))
  .set_name("base")
  .set_type_axes_names({"T{ct}", "OffsetT{ct}"})
  .add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4))
  .add_string_axis("Entropy", {"1.000", "0.201"});
